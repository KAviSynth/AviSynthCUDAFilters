#include "hip/hip_runtime.h"

#include <stdint.h>
#include <avisynth.h>

#include <algorithm>

#include "CommonFunctions.h"
#include "KFM.h"
#include "TextOut.h"

#include "VectorFunctions.cuh"
#include "ReduceKernel.cuh"
#include "KFMFilterBase.cuh"


__device__ __host__ void CountFlag(int cnt[3], int flag)
{
  if (flag & MOVE) cnt[0]++;
  if (flag & SHIMA) cnt[1]++;
  if (flag & LSHIMA) cnt[2]++;
}

void cpu_count_fmflags(FMCount* dst, const uchar4* flagp, int width, int height, int pitch)
{
  int cnt[3] = { 0 };
  for (int y = 0; y < height; ++y) {
    for (int x = 0; x < width; ++x) {
      uchar4 flags = flagp[x + y * pitch];
      CountFlag(cnt, flags.x);
      CountFlag(cnt, flags.y);
      CountFlag(cnt, flags.z);
      CountFlag(cnt, flags.w);
    }
  }
  dst->move = cnt[0];
  dst->shima = cnt[1];
  dst->lshima = cnt[2];
}

__global__ void kl_init_fmcount(FMCount* dst)
{
  int tx = threadIdx.x;
  dst[tx].move = dst[tx].shima = dst[tx].lshima = 0;
}

enum {
  FM_COUNT_TH_W = 32,
  FM_COUNT_TH_H = 16,
  FM_COUNT_THREADS = FM_COUNT_TH_W * FM_COUNT_TH_H,
};

__global__ void kl_count_fmflags(FMCount* dst, const uchar4* flagp, int width, int height, int pitch)
{
  int x = threadIdx.x + blockIdx.x * FM_COUNT_TH_W;
  int y = threadIdx.y + blockIdx.y * FM_COUNT_TH_H;
  int tid = threadIdx.x + threadIdx.y * FM_COUNT_TH_W;

  int cnt[3] = { 0 };

  if (x < width && y < height) {
    uchar4 flags = flagp[x + y * pitch];
    CountFlag(cnt, flags.x);
    CountFlag(cnt, flags.y);
    CountFlag(cnt, flags.z);
    CountFlag(cnt, flags.w);
  }

  __shared__ int sbuf[FM_COUNT_THREADS * 3];
  dev_reduceN<int, 3, FM_COUNT_THREADS, AddReducer<int>>(tid, cnt, sbuf);

  if (tid == 0) {
    atomicAdd(&dst->move, cnt[0]);
    atomicAdd(&dst->shima, cnt[1]);
    atomicAdd(&dst->lshima, cnt[2]);
  }
}

class KFMFrameAnalyze : public KFMFilterBase
{
  VideoInfo padvi;
  VideoInfo flagvi;

  FrameAnalyzeParam prmY;
  FrameAnalyzeParam prmC;

  PClip superclip;

  void CountFlags(Frame& flag, Frame& dst, int parity, PNeoEnv env)
  {
    const uchar4* flagp = flag.GetReadPtr<uchar4>(PLANAR_Y);
    FMCount* fmcnt = dst.GetWritePtr<FMCount>();
    int width4 = srcvi.width >> 2;
    int flagPitch = flag.GetPitch<uchar4>(PLANAR_Y);

    FMCount* fmcnt0 = &fmcnt[0];
    FMCount* fmcnt1 = &fmcnt[1];
    if (!parity) {
      std::swap(fmcnt0, fmcnt1);
    }

    if (IS_CUDA) {
      dim3 threads(FM_COUNT_TH_W, FM_COUNT_TH_H);
      dim3 blocks(nblocks(srcvi.width, threads.x), nblocks(srcvi.height / 2, threads.y));
      kl_init_fmcount << <1, 2 >> > (fmcnt);
      DEBUG_SYNC;
      kl_count_fmflags << <blocks, threads >> >(
        fmcnt0, flagp, width4, srcvi.height / 2, flagPitch * 2);
      DEBUG_SYNC;
      kl_count_fmflags << <blocks, threads >> >(
        fmcnt1, flagp + flagPitch, width4, srcvi.height / 2, flagPitch * 2);
      DEBUG_SYNC;
    }
    else {
      cpu_count_fmflags(fmcnt0, flagp, width4, srcvi.height / 2, flagPitch * 2);
      cpu_count_fmflags(fmcnt1, flagp + flagPitch, width4, srcvi.height / 2, flagPitch * 2);
    }
  }

  template <typename pixel_t>
  PVideoFrame GetFrameT(int n, PNeoEnv env)
  {
    int parity = child->GetParity(n);

    Frame f0padded;
    Frame f1padded;

    if (superclip) {
      f0padded = Frame(superclip->GetFrame(n, env), VPAD);
      f1padded = Frame(superclip->GetFrame(n + 1, env), VPAD);
    }
    else {
      Frame f0 = child->GetFrame(n, env);
      Frame f1 = child->GetFrame(n + 1, env);
      f0padded = Frame(env->NewVideoFrame(padvi), VPAD);
      f1padded = Frame(env->NewVideoFrame(padvi), VPAD);
      CopyFrame<pixel_t>(f0, f0padded, env);
      PadFrame<pixel_t>(f0padded, env);
      CopyFrame<pixel_t>(f1, f1padded, env);
      PadFrame<pixel_t>(f1padded, env);
    }

    Frame fflag = env->NewVideoFrame(flagvi);
    Frame dst = env->NewVideoFrame(vi);

    AnalyzeFrame<pixel_t>(f0padded, f1padded, fflag, &prmY, &prmC, env);
    MergeUVFlags(fflag, env); // UV���茋�ʂ�Y�Ƀ}�[�W
    CountFlags(fflag, dst, parity, env);

    return dst.frame;
  }

public:
  KFMFrameAnalyze(PClip clip, int threshMY, int threshSY, int threshMC, int threshSC, PClip super, IScriptEnvironment* env)
    : KFMFilterBase(clip)
    , prmY(threshMY, threshSY, threshSY * 3)
    , prmC(threshMC, threshSC, threshSC * 3)
    , padvi(vi)
    , flagvi()
    , superclip(super)
  {
    padvi.height += VPAD * 2;

    int out_bytes = sizeof(FMCount) * 2;
    vi.pixel_type = VideoInfo::CS_BGR32;
    vi.width = 16;
    vi.height = nblocks(out_bytes, vi.width * 4);

    flagvi.pixel_type = Get8BitType(srcvi);
    flagvi.width = srcvi.width;
    flagvi.height = srcvi.height;
  }

  PVideoFrame __stdcall GetFrame(int n, IScriptEnvironment* env_)
  {
    PNeoEnv env = env_;

    int pixelSize = vi.ComponentSize();
    switch (pixelSize) {
    case 1:
      return GetFrameT<uint8_t>(n, env);
    case 2:
      return GetFrameT<uint16_t>(n, env);
    default:
      env->ThrowError("[KFMFrameAnalyze] Unsupported pixel format");
    }

    return PVideoFrame();
  }

  static AVSValue __cdecl Create(AVSValue args, void* user_data, IScriptEnvironment* env)
  {
    return new KFMFrameAnalyze(
      args[0].AsClip(),       // clip
      args[1].AsInt(15),      // threshMY
      args[2].AsInt(7),       // threshSY
      args[3].AsInt(20),      // threshMC
      args[4].AsInt(8),       // threshSC
      args[5].AsClip(),       // super
      env
    );
  }
};

class KFMFrameAnalyzeCheck : public GenericVideoFilter
{
  PClip clipB;
public:
  KFMFrameAnalyzeCheck(PClip clipA, PClip clipB, IScriptEnvironment* env)
    : GenericVideoFilter(clipA)
    , clipB(clipB)
  {}

  PVideoFrame __stdcall GetFrame(int n, IScriptEnvironment* env_)
  {
    PNeoEnv env = env_;

    Frame frameA = child->GetFrame(n, env);
    Frame frameB = clipB->GetFrame(n, env);

    const FMCount* fmcntA = frameA.GetReadPtr<FMCount>();
    const FMCount* fmcntB = frameB.GetReadPtr<FMCount>();

    if (memcmp(fmcntA, fmcntB, sizeof(FMCount) * 2)) {
      env->ThrowError("[KFMFrameAnalyzeCheck] Unmatch !!!");
    }

    return frameA.frame;
  }

  static AVSValue __cdecl Create(AVSValue args, void* user_data, IScriptEnvironment* env)
  {
    return new KFMFrameAnalyzeCheck(
      args[0].AsClip(),       // clipA
      args[1].AsClip(),       // clipB
      env
    );
  }
};

class KFMFrameAnalyzeShow : public KFMFilterBase
{
  typedef uint8_t pixel_t;

  VideoInfo padvi;
  VideoInfo flagvi;

  FrameAnalyzeParam prmY;
  FrameAnalyzeParam prmC;

  PClip superclip;

  int threshMY;
  int threshSY;
  int threshLSY;
  int threshMC;
  int threshSC;
  int threshLSC;

  int logUVx;
  int logUVy;

  void VisualizeFlags(Frame& dst, Frame& fflag, PNeoEnv env)
  {
    // ���茋�ʂ�\��
    int black[] = { 0, 128, 128 };
    int blue[] = { 73, 230, 111 };
    int gray[] = { 140, 128, 128 };
    int purple[] = { 197, 160, 122 };

    const pixel_t* fflagp = fflag.GetReadPtr<pixel_t>(PLANAR_Y);
    pixel_t* dstY = dst.GetWritePtr<pixel_t>(PLANAR_Y);
    pixel_t* dstU = dst.GetWritePtr<pixel_t>(PLANAR_U);
    pixel_t* dstV = dst.GetWritePtr<pixel_t>(PLANAR_V);

    int flagPitch = fflag.GetPitch<pixel_t>(PLANAR_Y);
    int dstPitchY = dst.GetPitch<pixel_t>(PLANAR_Y);
    int dstPitchUV = dst.GetPitch<pixel_t>(PLANAR_U);

    // ���ŏ��������Ă���
    for (int y = 0; y < vi.height; ++y) {
      for (int x = 0; x < vi.width; ++x) {
        int offY = x + y * dstPitchY;
        int offUV = (x >> logUVx) + (y >> logUVy) * dstPitchUV;
        dstY[offY] = black[0];
        dstU[offUV] = black[1];
        dstV[offUV] = black[2];
      }
    }

    // �F��t����
    for (int y = 0; y < vi.height; ++y) {
      for (int x = 0; x < vi.width; ++x) {
        int flag = fflagp[x + y * flagPitch];
        flag |= (flag >> 4);

        int* color = nullptr;
        if ((flag & MOVE) && (flag & SHIMA)) {
          color = purple;
        }
        else if (flag & MOVE) {
          color = blue;
        }
        else if (flag & SHIMA) {
          color = gray;
        }

        if (color) {
          int offY = x + y * dstPitchY;
          int offUV = (x >> logUVx) + (y >> logUVy) * dstPitchUV;
          dstY[offY] = color[0];
          dstU[offUV] = color[1];
          dstV[offUV] = color[2];
        }
      }
    }
  }

  template <typename pixel_t>
  PVideoFrame GetFrameT(int n, PNeoEnv env)
  {
    Frame f0padded;
    Frame f1padded;

    if (superclip) {
      f0padded = Frame(superclip->GetFrame(n, env), VPAD);
      f1padded = Frame(superclip->GetFrame(n + 1, env), VPAD);
    }
    else {
      Frame f0 = child->GetFrame(n, env);
      Frame f1 = child->GetFrame(n + 1, env);
      f0padded = Frame(env->NewVideoFrame(padvi), VPAD);
      f1padded = Frame(env->NewVideoFrame(padvi), VPAD);
      CopyFrame<pixel_t>(f0, f0padded, env);
      PadFrame<pixel_t>(f0padded, env);
      CopyFrame<pixel_t>(f1, f1padded, env);
      PadFrame<pixel_t>(f1padded, env);
    }

    Frame fflag = env->NewVideoFrame(flagvi);
    Frame dst = env->NewVideoFrame(vi);

    AnalyzeFrame<pixel_t>(f0padded, f1padded, fflag, &prmY, &prmC, env);
    MergeUVFlags(fflag, env); // UV���茋�ʂ�Y�Ƀ}�[�W
    VisualizeFlags(dst, fflag, env);

    return dst.frame;
  }

public:
  KFMFrameAnalyzeShow(PClip clip, int threshMY, int threshSY, int threshMC, int threshSC, PClip super, IScriptEnvironment* env)
    : KFMFilterBase(clip)
    , prmY(threshMY, threshSY, threshSY * 3)
    , prmC(threshMC, threshSC, threshSC * 3)
    , logUVx(vi.GetPlaneWidthSubsampling(PLANAR_U))
    , logUVy(vi.GetPlaneHeightSubsampling(PLANAR_U))
    , padvi(vi)
    , flagvi()
    , superclip(super)
  {
    padvi.height += VPAD * 2;

    flagvi.pixel_type = Get8BitType(srcvi);
    flagvi.width = srcvi.width;
    flagvi.height = srcvi.height;
  }

  PVideoFrame __stdcall GetFrame(int n, IScriptEnvironment* env_)
  {
    PNeoEnv env = env_;

    int pixelSize = vi.ComponentSize();
    switch (pixelSize) {
    case 1:
      return GetFrameT<uint8_t>(n, env);
    case 2:
      return GetFrameT<uint16_t>(n, env);
    default:
      env->ThrowError("[KFMFrameDev] Unsupported pixel format");
    }

    return PVideoFrame();
  }

  // CUDA��Ή�
  int __stdcall SetCacheHints(int cachehints, int frame_range) { return 0; }

  static AVSValue __cdecl Create(AVSValue args, void* user_data, IScriptEnvironment* env)
  {
    return new KFMFrameAnalyzeShow(
      args[0].AsClip(),       // clip
      args[1].AsInt(10),       // threshMY
      args[2].AsInt(10),       // threshSY
      args[3].AsInt(10),       // threshMC
      args[4].AsInt(10),       // threshSC
      args[5].AsClip(),        // super
      env
    );
  }
};

class KTelecine : public KFMFilterBase
{
  PClip fmclip;
  bool show;

  PulldownPatterns patterns;

  template <typename pixel_t>
  void CopyField(bool top, Frame* const * frames, Frame& dst, PNeoEnv env)
  {
    typedef typename VectorType<pixel_t>::type vpixel_t;
    Frame& frame0 = *frames[0];
    const vpixel_t* src0Y = frame0.GetReadPtr<vpixel_t>(PLANAR_Y);
    const vpixel_t* src0U = frame0.GetReadPtr<vpixel_t>(PLANAR_U);
    const vpixel_t* src0V = frame0.GetReadPtr<vpixel_t>(PLANAR_V);
    vpixel_t* dstY = dst.GetWritePtr<vpixel_t>(PLANAR_Y);
    vpixel_t* dstU = dst.GetWritePtr<vpixel_t>(PLANAR_U);
    vpixel_t* dstV = dst.GetWritePtr<vpixel_t>(PLANAR_V);

    int pitchY = frame0.GetPitch<vpixel_t>(PLANAR_Y);
    int pitchUV = frame0.GetPitch<vpixel_t>(PLANAR_U);
    int width4 = vi.width >> 2;
    int width4UV = width4 >> logUVx;
    int heightUV = vi.height >> logUVy;

    if (!top) {
      src0Y += pitchY;
      src0U += pitchUV;
      src0V += pitchUV;
      dstY += pitchY;
      dstU += pitchUV;
      dstV += pitchUV;
    }

    if (frames[1] == nullptr) {
      if (IS_CUDA) {
        dim3 threads(32, 16);
        dim3 blocks(nblocks(width4, threads.x), nblocks(srcvi.height / 2, threads.y));
        dim3 blocksUV(nblocks(width4UV, threads.x), nblocks(heightUV / 2, threads.y));
        kl_copy << <blocks, threads >> >(dstY, src0Y, width4, vi.height / 2, pitchY * 2);
        DEBUG_SYNC;
        kl_copy << <blocksUV, threads >> >(dstU, src0U, width4UV, heightUV / 2, pitchUV * 2);
        DEBUG_SYNC;
        kl_copy << <blocksUV, threads >> >(dstV, src0V, width4UV, heightUV / 2, pitchUV * 2);
        DEBUG_SYNC;
      }
      else {
        cpu_copy(dstY, src0Y, width4, vi.height / 2, pitchY * 2);
        cpu_copy(dstU, src0U, width4UV, heightUV / 2, pitchUV * 2);
        cpu_copy(dstV, src0V, width4UV, heightUV / 2, pitchUV * 2);
      }
    }
    else {
      Frame& frame1 = *frames[1];
      const vpixel_t* src1Y = frame1.GetReadPtr<vpixel_t>(PLANAR_Y);
      const vpixel_t* src1U = frame1.GetReadPtr<vpixel_t>(PLANAR_U);
      const vpixel_t* src1V = frame1.GetReadPtr<vpixel_t>(PLANAR_V);

      if (!top) {
        src1Y += pitchY;
        src1U += pitchUV;
        src1V += pitchUV;
      }

      if (IS_CUDA) {
        dim3 threads(32, 16);
        dim3 blocks(nblocks(width4, threads.x), nblocks(srcvi.height / 2, threads.y));
        dim3 blocksUV(nblocks(width4UV, threads.x), nblocks(heightUV / 2, threads.y));
        kl_average << <blocks, threads >> >(dstY, src0Y, src1Y, width4, vi.height / 2, pitchY * 2);
        DEBUG_SYNC;
        kl_average << <blocksUV, threads >> >(dstU, src0U, src1U, width4UV, heightUV / 2, pitchUV * 2);
        DEBUG_SYNC;
        kl_average << <blocksUV, threads >> >(dstV, src0V, src1V, width4UV, heightUV / 2, pitchUV * 2);
        DEBUG_SYNC;
      }
      else {
        cpu_average(dstY, src0Y, src1Y, width4, vi.height / 2, pitchY * 2);
        cpu_average(dstU, src0U, src1U, width4UV, heightUV / 2, pitchUV * 2);
        cpu_average(dstV, src0V, src1V, width4UV, heightUV / 2, pitchUV * 2);
      }
    }

  }

  template <typename pixel_t>
  Frame CreateWeaveFrame(PClip clip, int n, int fstart, int fnum, int parity, PNeoEnv env)
  {
    // fstart��0or1�ɂ���
    if (fstart < 0 || fstart >= 2) {
      n += fstart / 2;
      fstart &= 1;
    }

    assert(fstart == 0 || fstart == 1);
    assert(fnum == 2 || fnum == 3 || fnum == 4);

    if (fstart == 0 && fnum == 2) {
      return clip->GetFrame(n, env);
    }
    else {
      Frame cur = clip->GetFrame(n, env);
      Frame nxt = clip->GetFrame(n + 1, env);
      Frame dst = env->NewVideoFrame(vi);

      // 3�t�B�[���h�̂Ƃ��͏d���t�B�[���h�𕽋ω�����

      Frame* srct[2] = { 0 };
      Frame* srcb[2] = { 0 };

      if (parity) {
        srct[0] = &nxt;
        srcb[0] = &cur;
        if (fnum >= 3) {
          if (fstart == 0) {
            srct[1] = &cur;
          }
          else {
            srcb[1] = &nxt;
          }
        }
      }
      else {
        srct[0] = &cur;
        srcb[0] = &nxt;
        if (fnum >= 3) {
          if (fstart == 0) {
            srcb[1] = &cur;
          }
          else {
            srct[1] = &nxt;
          }
        }
      }

      CopyField<pixel_t>(true, srct, dst, env);
      CopyField<pixel_t>(false, srcb, dst, env);

      return dst;
    }
  }

  template <typename pixel_t>
  void DrawInfo(Frame& dst, int pattern, float cost, int fnum, PNeoEnv env) {
    int number;
    const char* patternName = patterns.PatternToString(pattern, number);
    char buf[100]; sprintf(buf, "KFM: %s-%d (%d) (%.1f) - %d", patternName, number, pattern, cost, fnum);
    DrawText<pixel_t>(dst.frame, vi.BitsPerComponent(), 0, 0, buf, env);
  }

  template <typename pixel_t>
  PVideoFrame GetFrameT(int n, PNeoEnv env)
  {
    int cycleIndex = n / 4;
    int parity = child->GetParity(cycleIndex * 5);
    Frame fm = fmclip->GetFrame(cycleIndex, env);
    int pattern = fm.GetProperty("KFM_Pattern", -1);
    if (pattern == -1) {
      env->ThrowError("[KTelecine] Failed to get frame info. Check fmclip");
    }
    float cost = (float)fm.GetProperty("KFM_Cost", 1.0);
    Frame24Info frameInfo = patterns.GetFrame24(pattern, n);

    int fstart = frameInfo.cycleIndex * 10 + frameInfo.fieldStartIndex;
    Frame out = CreateWeaveFrame<pixel_t>(child, 0, fstart, frameInfo.numFields, parity, env);

    if (show) {
      DrawInfo<pixel_t>(out, pattern, cost, frameInfo.numFields, env);
    }

    return out.frame;
  }

public:
  KTelecine(PClip child, PClip fmclip, bool show, IScriptEnvironment* env)
    : KFMFilterBase(child)
    , fmclip(fmclip)
    , show(show)
  {
    // �t���[�����[�g
    vi.MulDivFPS(4, 5);
    vi.num_frames = (vi.num_frames / 5 * 4) + (vi.num_frames % 5);
  }

  PVideoFrame __stdcall GetFrame(int n, IScriptEnvironment* env_)
  {
    PNeoEnv env = env_;

    int pixelSize = vi.ComponentSize();
    switch (pixelSize) {
    case 1:
      return GetFrameT<uint8_t>(n, env);
    case 2:
      return GetFrameT<uint16_t>(n, env);
    default:
      env->ThrowError("[KTelecine] Unsupported pixel format");
    }

    return PVideoFrame();
  }

  int __stdcall SetCacheHints(int cachehints, int frame_range) {
    if (cachehints == CACHE_GET_DEV_TYPE) {
      return GetDeviceTypes(child) &
        (DEV_TYPE_CPU | DEV_TYPE_CUDA);
    }
    return 0;
  };

  static AVSValue __cdecl Create(AVSValue args, void* user_data, IScriptEnvironment* env)
  {
    return new KTelecine(
      args[0].AsClip(),       // source
      args[1].AsClip(),       // fmclip
      args[2].AsBool(false),  // show
      env
    );
  }
};

enum {
  DC_OVERLAP = 4,
  DC_BLOCK_SIZE = 8,
  DC_BLOCK_TH_W = 8,
  DC_BLOCK_TH_H = 8,
};

template <typename pixel_t, typename vpixel_t>
void cpu_detect_combe(pixel_t* flagp, int fpitch,
  const vpixel_t* srcp, int pitch, int nBlkX, int nBlkY, int shift)
{
  for (int by = 0; by < nBlkY - 1; ++by) {
    for (int bx = 0; bx < nBlkX - 1; ++bx) {
      int sum = 0;
      for (int tx = 0; tx < 2; ++tx) {
        int x = bx * DC_OVERLAP / 4 + tx;
        int y = by * DC_OVERLAP;
        auto L0 = srcp[x + (y + 0) * pitch];
        auto L1 = srcp[x + (y + 1) * pitch];
        auto L2 = srcp[x + (y + 2) * pitch];
        auto L3 = srcp[x + (y + 3) * pitch];
        auto L4 = srcp[x + (y + 4) * pitch];
        auto L5 = srcp[x + (y + 5) * pitch];
        auto L6 = srcp[x + (y + 6) * pitch];
        auto L7 = srcp[x + (y + 7) * pitch];
        int4 diff8 = absdiff(L0, L7);
        int4 diffT = absdiff(L0, L1) + absdiff(L1, L2) + absdiff(L2, L3) + absdiff(L3, L4) + absdiff(L4, L5) + absdiff(L5, L6) + absdiff(L6, L7) - diff8;
        int4 diffE = absdiff(L0, L2) + absdiff(L2, L4) + absdiff(L4, L6) + absdiff(L6, L7) - diff8;
        int4 diffO = absdiff(L0, L1) + absdiff(L1, L3) + absdiff(L3, L5) + absdiff(L5, L7) - diff8;
        int4 score = diffT - diffE - diffO;
        sum += score.x + score.y + score.z + score.w;
      }
      flagp[(bx + 1) + (by + 1) * fpitch] = clamp(sum >> shift, 0, 255);
    }
  }
}

template <typename pixel_t, typename vpixel_t>
__global__ void kl_detect_combe(pixel_t* flagp, int fpitch,
  const vpixel_t* srcp, int pitch, int nBlkX, int nBlkY, int shift)
{
  int tx = threadIdx.x;
  int bx = blockIdx.x * DC_BLOCK_TH_W + threadIdx.y;
  int by = blockIdx.y * DC_BLOCK_TH_H + threadIdx.z;

  if (bx < nBlkX - 1 && by < nBlkY - 1) {
    int x = bx * DC_OVERLAP / 4 + tx;
    int y = by * DC_OVERLAP;
    auto L0 = srcp[x + (y + 0) * pitch];
    auto L1 = srcp[x + (y + 1) * pitch];
    auto L2 = srcp[x + (y + 2) * pitch];
    auto L3 = srcp[x + (y + 3) * pitch];
    auto L4 = srcp[x + (y + 4) * pitch];
    auto L5 = srcp[x + (y + 5) * pitch];
    auto L6 = srcp[x + (y + 6) * pitch];
    auto L7 = srcp[x + (y + 7) * pitch];
    int4 diff8 = absdiff(L0, L7);
    int4 diffT = absdiff(L0, L1) + absdiff(L1, L2) + absdiff(L2, L3) + absdiff(L3, L4) + absdiff(L4, L5) + absdiff(L5, L6) + absdiff(L6, L7) - diff8;
    int4 diffE = absdiff(L0, L2) + absdiff(L2, L4) + absdiff(L4, L6) + absdiff(L6, L7) - diff8;
    int4 diffO = absdiff(L0, L1) + absdiff(L1, L3) + absdiff(L3, L5) + absdiff(L5, L7) - diff8;
    int4 score = diffT - diffE - diffO;
    int sum = score.x + score.y + score.z + score.w;
#if CUDART_VERSION >= 9000
    sum += __shfl_down_sync(0xffffffff, sum, 1);
#else
    sum += __shfl_down(sum, 1);
#endif
    if (tx == 0) {
      flagp[(bx + 1) + (by + 1) * fpitch] = clamp(sum >> shift, 0, 255);
    }
  }
}

__device__ __host__ int BinomialMerge(int a, int b, int c, int d, int e, int thresh)
{
  int minv = min(a, min(b, min(c, min(d, e))));
  int maxv = max(a, max(b, max(c, max(d, e))));
  if (maxv - minv < thresh) {
    return (b + 2 * c + d + 2) >> 2;
  }
  return c;
}

template <typename pixel_t>
void cpu_remove_combe2(pixel_t* dst,
  const pixel_t* src, int width, int height, int pitch,
  const pixel_t* combe, int c_pitch, int thcombe, int thdiff)
{
  for (int y = 0; y < height; ++y) {
    for (int x = 0; x < width; ++x) {
      int score = combe[(x >> 2) + (y >> 2) * c_pitch];
      if (score >= thcombe) {
        dst[x + y * pitch] = BinomialMerge(
          src[x + (y - 2) * pitch],
          src[x + (y - 1) * pitch],
          src[x + y * pitch],
          src[x + (y + 1) * pitch],
          src[x + (y + 2) * pitch],
          thdiff);
      }
      else {
        dst[x + y * pitch] = src[x + y * pitch];
      }
    }
  }
}

template <typename pixel_t>
__global__ void kl_remove_combe2(pixel_t* dst,
  const pixel_t* src, int width, int height, int pitch,
  const pixel_t* combe, int c_pitch, int thcombe, int thdiff)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < width && y < height) {
    int score = combe[(x >> 2) + (y >> 2) * c_pitch];
    if (score >= thcombe) {
      dst[x + y * pitch] = BinomialMerge(
        src[x + (y - 2) * pitch],
        src[x + (y - 1) * pitch],
        src[x + y * pitch],
        src[x + (y + 1) * pitch],
        src[x + (y + 2) * pitch],
        thdiff);
    }
    else {
      dst[x + y * pitch] = src[x + y * pitch];
    }
  }
}

void cpu_max_extend_blocks(uint8_t* dstp, int pitch, int nBlkX, int nBlkY)
{
  for (int by = 1; by < nBlkY; ++by) {
    dstp[0 + by * pitch] = dstp[0 + 1 + (by + 0) * pitch];
    for (int bx = 1; bx < nBlkX - 1; ++bx) {
      dstp[bx + by * pitch] = max(
        dstp[bx + by * pitch], dstp[bx + 1 + (by + 0) * pitch]);
    }
  }
  for (int bx = 0; bx < nBlkX; ++bx) {
    dstp[bx] = dstp[bx + pitch];
  }
  for (int by = 1; by < nBlkY - 1; ++by) {
    for (int bx = 0; bx < nBlkX; ++bx) {
      dstp[bx + by * pitch] = max(
        dstp[bx + by * pitch], dstp[bx + 0 + (by + 1) * pitch]);
    }
  }
}

__global__ void kl_max_extend_blocks_h(uint8_t* dstp, const uint8_t* srcp, int pitch, int nBlkX, int nBlkY)
{
  int bx = threadIdx.x + blockIdx.x * blockDim.x;
  int by = threadIdx.y + blockIdx.y * blockDim.y;

  if (bx < nBlkX && by < nBlkY) {
    if (bx == nBlkX - 1) {
      // �������ޗ\�肪�Ȃ��Ƃ���Ƀ\�[�X���R�s�[����
      dstp[bx + by * pitch] = srcp[bx + by * pitch];
    }
    else if (bx == 0) {
      dstp[bx + by * pitch] = srcp[bx + 1 + (by + 0) * pitch];
    }
    else {
      dstp[bx + by * pitch] = max(
        srcp[bx + 0 + (by + 0) * pitch], srcp[bx + 1 + (by + 0) * pitch]);
    }
  }
}

__global__ void kl_max_extend_blocks_v(uint8_t* dstp, const uint8_t* srcp, int pitch, int nBlkX, int nBlkY)
{
  int bx = threadIdx.x + blockIdx.x * blockDim.x;
  int by = threadIdx.y + blockIdx.y * blockDim.y;

  if (bx < nBlkX && by < nBlkY) {
    if (by == nBlkY - 1) {
      // �������ޗ\�肪�Ȃ��Ƃ���Ƀ\�[�X���R�s�[����
      dstp[bx + by * pitch] = srcp[bx + by * pitch];
    }
    else if (by == 0) {
      dstp[bx + by * pitch] = srcp[bx + 0 + (by + 1) * pitch];
    }
    else {
      dstp[bx + by * pitch] = max(
        srcp[bx + 0 + (by + 0) * pitch], srcp[bx + 0 + (by + 1) * pitch]);
    }
  }
}

template <typename pixel_t>
void cpu_combe_to_flag(pixel_t* flag, int nBlkX, int nBlkY, int fpitch, const pixel_t* combe, int cpitch)
{
  for (int y = 0; y < nBlkY; ++y) {
    for (int x = 0; x < nBlkX; ++x) {
      flag[x + y * fpitch] =
        (combe[(2 * x + 0) + (2 * y + 0) * cpitch] +
          combe[(2 * x + 1) + (2 * y + 0) * cpitch] +
          combe[(2 * x + 0) + (2 * y + 1) * cpitch] +
          combe[(2 * x + 1) + (2 * y + 1) * cpitch] + 2) >> 2;
    }
  }
}

template <typename pixel_t>
__global__ void kl_combe_to_flag(pixel_t* flag, int nBlkX, int nBlkY, int fpitch, const pixel_t* combe, int cpitch)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < nBlkX && y < nBlkY) {
    flag[x + y * fpitch] =
      (combe[(2 * x + 0) + (2 * y + 0) * cpitch] +
        combe[(2 * x + 1) + (2 * y + 0) * cpitch] +
        combe[(2 * x + 0) + (2 * y + 1) * cpitch] +
        combe[(2 * x + 1) + (2 * y + 1) * cpitch] + 2) >> 2;
  }
}

template <typename pixel_t>
void cpu_sum_box3x3(pixel_t* dst, pixel_t* src, int width, int height, int pitch, int maxv)
{
  for (int y = 0; y < height; ++y) {
    for (int x = 0; x < width; ++x) {
      auto sumv = (src[(x - 1) + (y - 1)*pitch] + src[(x + 0) + (y - 1)*pitch] + src[(x + 1) + (y - 1)*pitch] +
        src[(x - 1) + (y + 0)*pitch] + src[(x + 0) + (y + 0)*pitch] + src[(x + 1) + (y + 0)*pitch] +
        src[(x - 1) + (y + 1)*pitch] + src[(x + 0) + (y + 1)*pitch] + src[(x + 1) + (y + 1)*pitch]);
      dst[x + y * pitch] = min(sumv >> 2, maxv); // �K����1/4����
    }
  }
}

template <typename pixel_t>
__global__ void kl_sum_box3x3(pixel_t* dst, pixel_t* src, int width, int height, int pitch, int maxv)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < width && y < height) {
    auto sumv = (src[(x - 1) + (y - 1)*pitch] + src[(x + 0) + (y - 1)*pitch] + src[(x + 1) + (y - 1)*pitch] +
      src[(x - 1) + (y + 0)*pitch] + src[(x + 0) + (y + 0)*pitch] + src[(x + 1) + (y + 0)*pitch] +
      src[(x - 1) + (y + 1)*pitch] + src[(x + 0) + (y + 1)*pitch] + src[(x + 1) + (y + 1)*pitch]);
    dst[x + y * pitch] = min(sumv >> 2, maxv); // �K����1/4����
  }
}

class KRemoveCombeCheck : public GenericVideoFilter
{
  PClip clipB;
  int nBlkX, nBlkY;
public:
  KRemoveCombeCheck(PClip clipA, PClip clipB, IScriptEnvironment* env)
    : GenericVideoFilter(clipA)
    , clipB(clipB)
  {
    nBlkX = nblocks(vi.width, OVERLAP);
    nBlkY = nblocks(vi.height, OVERLAP);
  }

  PVideoFrame __stdcall GetFrame(int n, IScriptEnvironment* env_)
  {
    PNeoEnv env = env_;

    Frame frameA = WrapSwitchFragFrame(
      child->GetFrame(n, env)->GetProperty(COMBE_FLAG_STR)->GetFrame());
    Frame frameB = WrapSwitchFragFrame(
      clipB->GetFrame(n, env)->GetProperty(COMBE_FLAG_STR)->GetFrame());

    const uint8_t* fmcntA = frameA.GetReadPtr<uint8_t>();
    const uint8_t* fmcntB = frameB.GetReadPtr<uint8_t>();

    for (int by = 0; by < nBlkY; ++by) {
      for (int bx = 0; bx < nBlkX; ++bx) {
        if (fmcntA[bx + by * nBlkX] != fmcntB[bx + by * nBlkX]) {
          env->ThrowError("[KRemoveCombeCheck] Unmatch !!!");
        }
      }
    }

    return frameA.frame;
  }

  static AVSValue __cdecl Create(AVSValue args, void* user_data, IScriptEnvironment* env)
  {
    return new KRemoveCombeCheck(
      args[0].AsClip(),       // clipA
      args[1].AsClip(),       // clipB
      env
    );
  }
};

class KRemoveCombe : public KFMFilterBase
{
  VideoInfo padvi;
  VideoInfo combvi;
  VideoInfo blockvi;

  float thsmooth;
  float smooth;
  bool detect_uv;
  bool show;
  float thcombe;

  template <typename pixel_t>
  void DetectCombe(Frame& src, Frame& combe, PNeoEnv env)
  {
    typedef typename VectorType<pixel_t>::type vpixel_t;
    const vpixel_t* srcY = src.GetReadPtr<vpixel_t>(PLANAR_Y);
    const vpixel_t* srcU = src.GetReadPtr<vpixel_t>(PLANAR_U);
    const vpixel_t* srcV = src.GetReadPtr<vpixel_t>(PLANAR_V);
    uint8_t* combeY = combe.GetWritePtr<uint8_t>(PLANAR_Y);
    uint8_t* combeU = combe.GetWritePtr<uint8_t>(PLANAR_U);
    uint8_t* combeV = combe.GetWritePtr<uint8_t>(PLANAR_V);

    int pitchY = src.GetPitch<vpixel_t>(PLANAR_Y);
    int pitchUV = src.GetPitch<vpixel_t>(PLANAR_U);
    int fpitchY = combe.GetPitch<uint8_t>(PLANAR_Y);
    int fpitchUV = combe.GetPitch<uint8_t>(PLANAR_U);
    int widthUV = combvi.width >> logUVx;
    int heightUV = combvi.height >> logUVy;

    int shift = vi.BitsPerComponent() - 8 + 4;

    if (IS_CUDA) {
      dim3 threads(2, DC_BLOCK_TH_W, DC_BLOCK_TH_H);
      dim3 blocks(nblocks(combvi.width, DC_BLOCK_TH_W), nblocks(combvi.height, DC_BLOCK_TH_H));
      dim3 blocksUV(nblocks(widthUV, DC_BLOCK_TH_W), nblocks(heightUV, DC_BLOCK_TH_H));
      kl_detect_combe << <blocks, threads >> >(combeY, fpitchY, srcY, pitchY, combvi.width, combvi.height, shift);
      DEBUG_SYNC;
      if (detect_uv) {
        kl_detect_combe << <blocksUV, threads >> >(combeU, fpitchUV, srcU, pitchUV, widthUV, heightUV, shift);
        DEBUG_SYNC;
        kl_detect_combe << <blocksUV, threads >> >(combeV, fpitchUV, srcV, pitchUV, widthUV, heightUV, shift);
        DEBUG_SYNC;
      }
    }
    else {
      cpu_detect_combe(combeY, fpitchY, srcY, pitchY, combvi.width, combvi.height, shift);
      if (detect_uv) {
        cpu_detect_combe(combeU, fpitchUV, srcU, pitchUV, widthUV, heightUV, shift);
        cpu_detect_combe(combeV, fpitchUV, srcV, pitchUV, widthUV, heightUV, shift);
      }
    }
  }

  void ExtendBlocks(Frame& dst, Frame& tmp, PNeoEnv env)
  {
    uint8_t* tmpY = tmp.GetWritePtr<uint8_t>(PLANAR_Y);
    uint8_t* tmpU = tmp.GetWritePtr<uint8_t>(PLANAR_U);
    uint8_t* tmpV = tmp.GetWritePtr<uint8_t>(PLANAR_V);
    uint8_t* dstY = dst.GetWritePtr<uint8_t>(PLANAR_Y);
    uint8_t* dstU = dst.GetWritePtr<uint8_t>(PLANAR_U);
    uint8_t* dstV = dst.GetWritePtr<uint8_t>(PLANAR_V);

    int pitchY = tmp.GetPitch<uint8_t>(PLANAR_Y);
    int pitchUV = tmp.GetPitch<uint8_t>(PLANAR_U);
    int widthUV = combvi.width >> logUVx;
    int heightUV = combvi.height >> logUVy;

    if (IS_CUDA) {
      dim3 threads(32, 16);
      dim3 blocks(nblocks(combvi.width, threads.x), nblocks(combvi.height, threads.y));
      dim3 blocksUV(nblocks(widthUV, threads.x), nblocks(heightUV, threads.y));
      kl_max_extend_blocks_h << <blocks, threads >> >(tmpY, dstY, pitchY, combvi.width, combvi.height);
      kl_max_extend_blocks_v << <blocks, threads >> >(dstY, tmpY, pitchY, combvi.width, combvi.height);
      DEBUG_SYNC;
      if (detect_uv) {
        kl_max_extend_blocks_h << <blocksUV, threads >> > (tmpU, dstU, pitchUV, widthUV, heightUV);
        kl_max_extend_blocks_v << <blocksUV, threads >> > (dstU, tmpU, pitchUV, widthUV, heightUV);
        DEBUG_SYNC;
        kl_max_extend_blocks_h << <blocksUV, threads >> > (tmpV, dstV, pitchUV, widthUV, heightUV);
        kl_max_extend_blocks_v << <blocksUV, threads >> > (dstV, tmpV, pitchUV, widthUV, heightUV);
        DEBUG_SYNC;
      }
    }
    else {
      cpu_max_extend_blocks(dstY, pitchY, combvi.width, combvi.height);
      if (detect_uv) {
        cpu_max_extend_blocks(dstU, pitchUV, widthUV, heightUV);
        cpu_max_extend_blocks(dstV, pitchUV, widthUV, heightUV);
      }
    }
  }

  void MergeUVCoefs(Frame& combe, PNeoEnv env)
  {
    uint8_t* fY = combe.GetWritePtr<uint8_t>(PLANAR_Y);
    uint8_t* fU = combe.GetWritePtr<uint8_t>(PLANAR_U);
    uint8_t* fV = combe.GetWritePtr<uint8_t>(PLANAR_V);
    int pitchY = combe.GetPitch<uint8_t>(PLANAR_Y);
    int pitchUV = combe.GetPitch<uint8_t>(PLANAR_U);

    if (IS_CUDA) {
      dim3 threads(32, 16);
      dim3 blocks(nblocks(combvi.width, threads.x), nblocks(combvi.height, threads.y));
      kl_merge_uvcoefs << <blocks, threads >> >(fY,
        fU, fV, combvi.width, combvi.height, pitchY, pitchUV, logUVx, logUVy);
      DEBUG_SYNC;
    }
    else {
      cpu_merge_uvcoefs(fY,
        fU, fV, combvi.width, combvi.height, pitchY, pitchUV, logUVx, logUVy);
    }
  }

  void ApplyUVCoefs(Frame& combe, PNeoEnv env)
  {
    uint8_t* fY = combe.GetWritePtr<uint8_t>(PLANAR_Y);
    uint8_t* fU = combe.GetWritePtr<uint8_t>(PLANAR_U);
    uint8_t* fV = combe.GetWritePtr<uint8_t>(PLANAR_V);
    int pitchY = combe.GetPitch<uint8_t>(PLANAR_Y);
    int pitchUV = combe.GetPitch<uint8_t>(PLANAR_U);
    int widthUV = combvi.width >> logUVx;
    int heightUV = combvi.height >> logUVy;

    if (IS_CUDA) {
      dim3 threads(32, 16);
      dim3 blocks(nblocks(widthUV, threads.x), nblocks(heightUV, threads.y));
      kl_apply_uvcoefs_420 << <blocks, threads >> >(fY,
        fU, fV, widthUV, heightUV, pitchY, pitchUV);
      DEBUG_SYNC;
    }
    else {
      cpu_apply_uvcoefs_420(fY, fU, fV, widthUV, heightUV, pitchY, pitchUV);
    }
  }

  template <typename pixel_t>
  void RemoveCombe(Frame& dst, Frame& src, Frame& combe, int thcombe, int thdiff, PNeoEnv env)
  {
    const uint8_t* combeY = combe.GetReadPtr<uint8_t>(PLANAR_Y);
    const uint8_t* combeU = combe.GetReadPtr<uint8_t>(PLANAR_U);
    const uint8_t* combeV = combe.GetReadPtr<uint8_t>(PLANAR_V);
    const pixel_t* srcY = src.GetReadPtr<pixel_t>(PLANAR_Y);
    const pixel_t* srcU = src.GetReadPtr<pixel_t>(PLANAR_U);
    const pixel_t* srcV = src.GetReadPtr<pixel_t>(PLANAR_V);
    pixel_t* dstY = dst.GetWritePtr<pixel_t>(PLANAR_Y);
    pixel_t* dstU = dst.GetWritePtr<pixel_t>(PLANAR_U);
    pixel_t* dstV = dst.GetWritePtr<pixel_t>(PLANAR_V);

    int pitchY = src.GetPitch<pixel_t>(PLANAR_Y);
    int pitchUV = src.GetPitch<pixel_t>(PLANAR_U);
    int fpitchY = combe.GetPitch<uint8_t>(PLANAR_Y);
    int fpitchUV = combe.GetPitch<uint8_t>(PLANAR_U);
    int widthUV = vi.width >> logUVx;
    int heightUV = vi.height >> logUVy;

    if (IS_CUDA) {
      dim3 threads(32, 16);
      dim3 blocks(nblocks(vi.width, threads.x), nblocks(vi.height, threads.y));
      dim3 blocksUV(nblocks(widthUV, threads.x), nblocks(heightUV, threads.y));
      kl_remove_combe2 << <blocks, threads >> >(dstY, srcY, vi.width, vi.height, pitchY, combeY, fpitchY, thcombe, thdiff);
      DEBUG_SYNC;
      kl_remove_combe2 << <blocksUV, threads >> >(dstU, srcU, widthUV, heightUV, pitchUV, combeU, fpitchUV, thcombe, thdiff);
      DEBUG_SYNC;
      kl_remove_combe2 << <blocksUV, threads >> >(dstV, srcV, widthUV, heightUV, pitchUV, combeV, fpitchUV, thcombe, thdiff);
      DEBUG_SYNC;
    }
    else {
      cpu_remove_combe2(dstY, srcY, vi.width, vi.height, pitchY, combeY, fpitchY, thcombe, thdiff);
      cpu_remove_combe2(dstU, srcU, widthUV, heightUV, pitchUV, combeU, fpitchUV, thcombe, thdiff);
      cpu_remove_combe2(dstV, srcV, widthUV, heightUV, pitchUV, combeV, fpitchUV, thcombe, thdiff);
    }
  }

  template <typename pixel_t>
  void VisualizeCombe(Frame& dst, Frame& combe, int thresh, PNeoEnv env)
  {
    // ���茋�ʂ�\��
    int blue[] = { 73, 230, 111 };

    const uint8_t* combep = combe.GetReadPtr<uint8_t>(PLANAR_Y);
    pixel_t* dstY = dst.GetWritePtr<pixel_t>(PLANAR_Y);
    pixel_t* dstU = dst.GetWritePtr<pixel_t>(PLANAR_U);
    pixel_t* dstV = dst.GetWritePtr<pixel_t>(PLANAR_V);

    int combePitch = combe.GetPitch<uint8_t>(PLANAR_Y);
    int dstPitchY = dst.GetPitch<pixel_t>(PLANAR_Y);
    int dstPitchUV = dst.GetPitch<pixel_t>(PLANAR_U);

    // �F��t����
    for (int y = 0; y < vi.height; ++y) {
      for (int x = 0; x < vi.width; ++x) {
        int score = combep[(x >> 2) + (y >> 2) * combePitch];

        int* color = nullptr;
        if (score >= thresh) {
          color = blue;
        }

        if (color) {
          int offY = x + y * dstPitchY;
          int offUV = (x >> logUVx) + (y >> logUVy) * dstPitchUV;
          dstY[offY] = color[0];
          dstU[offUV] = color[1];
          dstV[offUV] = color[2];
        }
      }
    }
  }

  void MakeSwitchFlag(Frame& flag, Frame& flagtmp, Frame& combe, PNeoEnv env)
  {
    const uint8_t* srcp = combe.GetReadPtr<uint8_t>(PLANAR_Y);
    uint8_t* flagp = flag.GetWritePtr<uint8_t>();
    uint8_t* flagtmpp = flagtmp.GetWritePtr<uint8_t>();

    int height = flag.GetHeight();
    int width = flag.GetWidth<uint8_t>();
    int fpitch = flag.GetPitch<uint8_t>();
    int cpitch = combe.GetPitch<uint8_t>();

    if (IS_CUDA) {
      dim3 threads(16, 8);
      dim3 blocks(nblocks(width, threads.x), nblocks(height, threads.y));
      kl_combe_to_flag << <blocks, threads >> >(
        flagp, width, height, fpitch, srcp, cpitch);
      DEBUG_SYNC;
      kl_sum_box3x3 << <blocks, threads >> >(
        flagtmpp, flagp, width, height, fpitch, 255);
      DEBUG_SYNC;
      kl_sum_box3x3 << <blocks, threads >> >(
        flagp, flagtmpp, width, height, fpitch, 255);
      DEBUG_SYNC;
    }
    else {
      cpu_combe_to_flag(flagp, width, height, fpitch, srcp, cpitch);
      cpu_sum_box3x3(flagtmpp, flagp, width, height, fpitch, 255);
      cpu_sum_box3x3(flagp, flagtmpp, width, height, fpitch, 255);
    }
  }

  PVideoFrame GetFrameT(int n, PNeoEnv env)
  {
    typedef uint8_t pixel_t;

    Frame src = child->GetFrame(n, env);
    Frame padded = Frame(env->NewVideoFrame(padvi), VPAD);
    Frame dst = env->NewVideoFrame(vi);
    Frame combe = env->NewVideoFrame(combvi);
    Frame combetmp = env->NewVideoFrame(combvi);
    Frame flag = NewSwitchFlagFrame(vi, env);
    Frame flagtmp = NewSwitchFlagFrame(vi, env);

    CopyFrame<pixel_t>(src, padded, env);
    PadFrame<pixel_t>(padded, env);
    DetectCombe<pixel_t>(padded, combe, env);
    ExtendBlocks(combe, combetmp, env);
    if (detect_uv) {
      MergeUVCoefs(combe, env);
    }
    ApplyUVCoefs(combe, env);
    RemoveCombe<pixel_t>(dst, padded, combe, (int)thsmooth, (int)smooth, env);
    DetectCombe<pixel_t>(dst, combe, env);
    ExtendBlocks(combe, combetmp, env);
    if (detect_uv) {
      MergeUVCoefs(combe, env);
    }
    MakeSwitchFlag(flag, flagtmp, combe, env);
    dst.SetProperty(COMBE_FLAG_STR, flag.frame);

    if (!IS_CUDA && show) {
      VisualizeCombe<pixel_t>(dst, combe, (int)thcombe, env);
      return dst.frame;
    }

    return dst.frame;
  }

public:
  KRemoveCombe(PClip clip, float thsmooth, float smooth, bool uv, bool show, float thcombe, IScriptEnvironment* env)
    : KFMFilterBase(clip)
    , padvi(vi)
    , blockvi(vi)
    , thsmooth(thsmooth)
    , smooth(smooth)
    , detect_uv(uv)
    , show(show)
    , thcombe(thcombe)
  {
    if (vi.width & 7) env->ThrowError("[KRemoveCombe]: width must be multiple of 8");
    if (vi.height & 7) env->ThrowError("[KRemoveCombe]: height must be multiple of 8");

    padvi.height += VPAD * 2;

    combvi.width = vi.width / DC_OVERLAP;
    combvi.height = vi.height / DC_OVERLAP;
    combvi.pixel_type = Get8BitType(vi);

    blockvi.width = nblocks(vi.width, OVERLAP);
    blockvi.height = nblocks(vi.height, OVERLAP);
    blockvi.pixel_type = VideoInfo::CS_Y8;
  }

  PVideoFrame __stdcall GetFrame(int n, IScriptEnvironment* env_)
  {
    PNeoEnv env = env_;

    int pixelSize = vi.ComponentSize();
    switch (pixelSize) {
    case 1:
      return GetFrameT(n, env);
    case 2:
      return GetFrameT(n, env);
    default:
      env->ThrowError("[KRemoveCombe] Unsupported pixel format");
    }

    return PVideoFrame();
  }

  static AVSValue __cdecl Create(AVSValue args, void* user_data, IScriptEnvironment* env)
  {
    return new KRemoveCombe(
      args[0].AsClip(),       // source
      (float)args[1].AsFloat(30), // thsmooth
      (float)args[2].AsFloat(100), // smooth
      args[3].AsBool(false), // uv
      args[4].AsBool(false), // show
      (float)args[5].AsFloat(100), // thcombe
      env
    );
  }
};

void AddFuncCombingAnalyze(IScriptEnvironment* env)
{
  env->AddFunction("KFMFrameAnalyzeShow", "c[threshMY]i[threshSY]i[threshMC]i[threshSC]i[super]c", KFMFrameAnalyzeShow::Create, 0);
  env->AddFunction("KFMFrameAnalyze", "c[threshMY]i[threshSY]i[threshMC]i[threshSC]i[super]c", KFMFrameAnalyze::Create, 0);

  env->AddFunction("KFMFrameAnalyzeCheck", "cc", KFMFrameAnalyzeCheck::Create, 0);

  env->AddFunction("KTelecine", "cc[show]b", KTelecine::Create, 0);
  env->AddFunction("KRemoveCombe", "c[thsmooth]f[smooth]f[uv]b[show]b[thcombe]f", KRemoveCombe::Create, 0);
  env->AddFunction("KRemoveCombeCheck", "cc", KRemoveCombeCheck::Create, 0);
}
