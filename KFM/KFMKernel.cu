
#include <stdint.h>
#include <avisynth.h>

#include <algorithm>
#include <memory>

#include "CommonFunctions.h"
#include "KFM.h"
#include "TextOut.h"

#include "VectorFunctions.cuh"
#include "ReduceKernel.cuh"
#include "KFMFilterBase.cuh"

class KPatchCombe : public KFMFilterBase
{
  PClip clip60;
  PClip combemaskclip;
  PClip containscombeclip;
  PClip fmclip;

  PulldownPatterns patterns;

  template <typename pixel_t>
  PVideoFrame GetFrameT(int n, PNeoEnv env)
  {
    PDevice cpuDevice = env->GetDevice(DEV_TYPE_CPU, 0);

    {
      Frame containsframe = env->GetFrame(containscombeclip, n, cpuDevice);
      if (*containsframe.GetReadPtr<int>() == 0) {
        // �_���ȃu���b�N�͂Ȃ��̂ł��̂܂ܕԂ�
        return child->GetFrame(n, env);
      }
    }

    int cycleIndex = n / 4;
    Frame fmframe = env->GetFrame(fmclip, cycleIndex, cpuDevice);
    int kfmPattern = fmframe.GetProperty("KFM_Pattern", -1);
    if (kfmPattern == -1) {
      env->ThrowError("[KPatchCombe] Failed to get frame info. Check fmclip");
    }
    Frame24Info frameInfo = patterns.GetFrame24(kfmPattern, n);

    int fieldIndex[] = { 1, 3, 6, 8 };
    // �W���ʒu
    int n60 = fieldIndex[n % 4];
    // �t�B�[���h�Ώ۔͈͂ɕ␳
    n60 = clamp(n60, frameInfo.fieldStartIndex, frameInfo.fieldStartIndex + frameInfo.numFields - 1);
    n60 += cycleIndex * 10;

    Frame baseFrame = child->GetFrame(n, env);
    Frame frame60 = child->GetFrame(n60, env);
    Frame mflag = combemaskclip->GetFrame(n, env);

    // �_���ȃu���b�N��bob�t���[������R�s�[
    Frame dst = env->NewVideoFrame(vi);
    MergeBlock<pixel_t>(baseFrame, frame60, mflag, dst, env);

    return dst.frame;
  }

public:
  KPatchCombe(PClip clip24, PClip clip60, PClip fmclip, PClip combemaskclip, PClip containscombeclip, IScriptEnvironment* env)
    : KFMFilterBase(clip24)
    , clip60(clip60)
    , combemaskclip(combemaskclip)
    , containscombeclip(containscombeclip)
    , fmclip(fmclip)
  {
    //
  }

  PVideoFrame __stdcall GetFrame(int n, IScriptEnvironment* env_)
  {
    PNeoEnv env = env_;

    int pixelSize = vi.ComponentSize();
    switch (pixelSize) {
    case 1:
      return GetFrameT<uint8_t>(n, env);
    case 2:
      return GetFrameT<uint16_t>(n, env);
    default:
      env->ThrowError("[KPatchCombe] Unsupported pixel format");
    }

    return PVideoFrame();
  }

  static AVSValue __cdecl Create(AVSValue args, void* user_data, IScriptEnvironment* env)
  {
    return new KPatchCombe(
      args[0].AsClip(),       // clip24
      args[1].AsClip(),       // clip60
      args[2].AsClip(),       // fmclip
      args[3].AsClip(),       // combemaskclip
      args[4].AsClip(),       // containscombeclip
      env
    );
  }
};

enum KFMSWTICH_FLAG {
  FRAME_60 = 1,
	FRAME_24,
  FRAME_UCF,
};

class KFMSwitch : public KFMFilterBase
{
	typedef uint8_t pixel_t;

	PClip clip24;
	PClip fmclip;
  PClip combemaskclip;
  PClip containscombeclip;
  PClip ucfclip;
	float thswitch;
	bool show;
	bool showflag;

	int logUVx;
	int logUVy;
	int nBlkX, nBlkY;

  VideoInfo workvi;

	PulldownPatterns patterns;

	template <typename pixel_t>
	void VisualizeFlag(Frame& dst, Frame& flag, PNeoEnv env)
	{
		// ���茋�ʂ�\��
		int blue[] = { 73, 230, 111 };

		pixel_t* dstY = dst.GetWritePtr<pixel_t>(PLANAR_Y);
		pixel_t* dstU = dst.GetWritePtr<pixel_t>(PLANAR_U);
		pixel_t* dstV = dst.GetWritePtr<pixel_t>(PLANAR_V);
    const uint8_t* flagY = flag.GetReadPtr<uint8_t>(PLANAR_Y);
    const uint8_t* flagC = flag.GetReadPtr<uint8_t>(PLANAR_U);

		int dstPitchY = dst.GetPitch<pixel_t>(PLANAR_Y);
		int dstPitchUV = dst.GetPitch<pixel_t>(PLANAR_U);
    int fpitchY = flag.GetPitch<uint8_t>(PLANAR_Y);
    int fpitchUV = flag.GetPitch<uint8_t>(PLANAR_U);

		// �F��t����
		for (int y = 0; y < vi.height; ++y) {
			for (int x = 0; x < vi.width; ++x) {
        int coefY = flagY[x + y * fpitchY];
				int offY = x + y * dstPitchY;
        dstY[offY] = (blue[0] * coefY + dstY[offY] * (128 - coefY)) >> 7;
        
        int coefC = flagC[(x >> logUVx) + (y >> logUVy) * fpitchUV];
				int offUV = (x >> logUVx) + (y >> logUVy) * dstPitchUV;
				dstU[offUV] = (blue[1] * coefC + dstU[offUV] * (128 - coefC)) >> 7;
				dstV[offUV] = (blue[2] * coefC + dstV[offUV] * (128 - coefC)) >> 7;
			}
		}
	}

	template <typename pixel_t>
	Frame InternalGetFrame(int n60, Frame& fmframe, int& type, PNeoEnv env)
	{
		int cycleIndex = n60 / 10;
		int kfmPattern = fmframe.GetProperty("KFM_Pattern", -1);
    if (kfmPattern == -1) {
      env->ThrowError("[KFMSwitch] Failed to get frame info. Check fmclip");
    }
		float kfmCost = (float)fmframe.GetProperty("KFM_Cost", 1.0);
    Frame baseFrame;

		if (kfmCost > thswitch || PulldownPatterns::Is30p(kfmPattern)) {
			// �R�X�g�������̂�60p�Ɣ��f or 30p�̏ꍇ
      type = FRAME_60;

      if (ucfclip) {
        baseFrame = ucfclip->GetFrame(n60, env);
        auto prop = baseFrame.GetProperty(DECOMB_UCF_FLAG_STR);
        if (prop == nullptr) {
          env->ThrowError("Invalid UCF clip");
        }
        auto flag = (DECOMB_UCF_FLAG)prop->GetInt();
        if (flag == DECOMB_UCF_NEXT || flag == DECOMB_UCF_PREV) {
          // �t���[���u�������ꂽ�ꍇ�́A60p�����}�[�W���������s����
          type = FRAME_UCF;
        }
        else {
          return baseFrame;
        }
      }
      else {
        return child->GetFrame(n60, env);
      }
		}
    else {
      type = FRAME_24;
    }

    // �����ł�type�� 24 or UCF

		// 24p�t���[���ԍ����擾
		Frame24Info frameInfo = patterns.GetFrame60(kfmPattern, n60);
		int n24 = frameInfo.cycleIndex * 4 + frameInfo.frameIndex + frameInfo.fieldShift;

		if (frameInfo.frameIndex < 0) {
			// �O�ɋ󂫂�����̂őO�̃T�C�N��
			n24 = frameInfo.cycleIndex * 4 - 1;
		}
		else if (frameInfo.frameIndex >= 4) {
			// ���̃T�C�N���̃p�^�[�����擾
			Frame nextfmframe = fmclip->GetFrame(cycleIndex + 1, env);
			int nextPattern = nextfmframe.GetProperty("KFM_Pattern", -1);
			int fstart = patterns.GetFrame24(nextPattern, 0).fieldStartIndex;
			if (fstart > 0) {
				// �O�ɋ󂫂�����̂őO�̃T�C�N��
				n24 = frameInfo.cycleIndex * 4 + 3;
			}
			else {
				// �O�ɋ󂫂��Ȃ��̂Ō��̃T�C�N��
				n24 = frameInfo.cycleIndex * 4 + 4;
			}
		}

		Frame frame24 = clip24->GetFrame(n24, env);

    if (type == FRAME_24) {
      baseFrame = frame24;
    }

		{
      Frame containsframe = env->GetFrame(containscombeclip, n24, env->GetDevice(DEV_TYPE_CPU, 0));
      if (*containsframe.GetReadPtr<int>() == 0) {
        // �_���ȃu���b�N�͂Ȃ��̂ł��̂܂ܕԂ�
        return baseFrame;
      }
		}

    Frame frame60 = child->GetFrame(n60, env);
    Frame mflag = combemaskclip->GetFrame(n24, env);

		if (!IS_CUDA && vi.ComponentSize() == 1 && showflag) {
			env->MakeWritable(&baseFrame.frame);
			VisualizeFlag<pixel_t>(baseFrame, mflag, env);
			return baseFrame;
		}

		// �_���ȃu���b�N��bob�t���[������R�s�[
		Frame dst = env->NewVideoFrame(vi);
		MergeBlock<pixel_t>(baseFrame, frame60, mflag, dst, env);

		return dst;
	}

  template <typename pixel_t>
  PVideoFrame GetFrameTop(int n60, PNeoEnv env)
  {
    int cycleIndex = n60 / 10;
    Frame fmframe = env->GetFrame(fmclip, cycleIndex, env->GetDevice(DEV_TYPE_CPU, 0));
    int frameType;

    Frame dst = InternalGetFrame<pixel_t>(n60, fmframe, frameType, env);

    if (show) {
      const std::pair<int, float>* pfm = fmframe.GetReadPtr<std::pair<int, float>>();
      const char* fps = (frameType == FRAME_60) ? "60p" : (frameType == FRAME_24) ? "24p" : "UCF";
      char buf[100]; sprintf(buf, "KFMSwitch: %s pattern:%2d cost:%.1f", fps, pfm->first, pfm->second);
      DrawText<pixel_t>(dst.frame, vi.BitsPerComponent(), 0, 0, buf, env);
      return dst.frame;
    }

    return dst.frame;
  }

public:
	KFMSwitch(PClip clip60, PClip clip24, PClip fmclip, PClip combemaskclip, PClip containscombeclip, PClip ucfclip,
		float thswitch, bool show, bool showflag, IScriptEnvironment* env)
		: KFMFilterBase(clip60)
		, clip24(clip24)
		, fmclip(fmclip)
    , combemaskclip(combemaskclip)
    , containscombeclip(containscombeclip)
    , ucfclip(ucfclip)
		, thswitch(thswitch)
		, show(show)
		, showflag(showflag)
		, logUVx(vi.GetPlaneWidthSubsampling(PLANAR_U))
		, logUVy(vi.GetPlaneHeightSubsampling(PLANAR_U))
	{
		if (vi.width & 7) env->ThrowError("[KFMSwitch]: width must be multiple of 8");
		if (vi.height & 7) env->ThrowError("[KFMSwitch]: height must be multiple of 8");


		nBlkX = nblocks(vi.width, OVERLAP);
		nBlkY = nblocks(vi.height, OVERLAP);

    // check clip device
    if (!(GetDeviceTypes(fmclip) & DEV_TYPE_CPU)) {
      env->ThrowError("[KFMSwitch]: fmclip must be CPU device");
    }
    if (!(GetDeviceTypes(containscombeclip) & DEV_TYPE_CPU)) {
      env->ThrowError("[KFMSwitch]: containscombeclip must be CPU device");
    }

    auto devs = GetDeviceTypes(clip60);
    if (!(GetDeviceTypes(clip24) & devs)) {
      env->ThrowError("[KFMSwitch]: clip24 device unmatch");
    }
    if (!(GetDeviceTypes(combemaskclip) & devs)) {
      env->ThrowError("[KFMSwitch]: combeclip device unmatch");
    }
    if (!(GetDeviceTypes(clip24) & devs)) {
      env->ThrowError("[KFMSwitch]: clip24 device unmatch");
    }
    if (ucfclip && !(GetDeviceTypes(ucfclip) & devs)) {
      env->ThrowError("[KFMSwitch]: ucfclip device unmatch");
    }
	}

	PVideoFrame __stdcall GetFrame(int n60, IScriptEnvironment* env_)
	{
		PNeoEnv env = env_;

		int pixelSize = vi.ComponentSize();
		switch (pixelSize) {
		case 1:
			return GetFrameTop<uint8_t>(n60, env);
		case 2:
      return GetFrameTop<uint16_t>(n60, env);
		default:
			env->ThrowError("[KFMSwitch] Unsupported pixel format");
			break;
		}

		return PVideoFrame();
	}

	static AVSValue __cdecl Create(AVSValue args, void* user_data, IScriptEnvironment* env)
	{
		return new KFMSwitch(
			args[0].AsClip(),           // clip60
			args[1].AsClip(),           // clip24
			args[2].AsClip(),           // fmclip
      args[3].AsClip(),           // combemaskclip
      args[4].AsClip(),           // containscombeclip
      args[5].Defined() ? args[5].AsClip() : nullptr,           // ucfclip
      (float)args[6].AsFloat(0.8f),// thswitch
			args[7].AsBool(false),      // show
			args[8].AsBool(false),      // showflag
			env
			);
	}
};

class KFMPad : public KFMFilterBase
{
  VideoInfo srcvi;

  template <typename pixel_t>
  PVideoFrame GetFrameT(int n, PNeoEnv env)
  {
    Frame src = child->GetFrame(n, env);
    Frame dst = Frame(env->NewVideoFrame(vi), VPAD);

    CopyFrame<pixel_t>(src, dst, env);
    PadFrame<pixel_t>(dst, env);

    return dst.frame;
  }
public:
  KFMPad(PClip src, IScriptEnvironment* env)
    : KFMFilterBase(src)
    , srcvi(vi)
  {
    if (srcvi.width & 3) env->ThrowError("[KFMPad]: width must be multiple of 4");
    if (srcvi.height & 3) env->ThrowError("[KFMPad]: height must be multiple of 4");

    vi.height += VPAD * 2;
  }

  PVideoFrame __stdcall GetFrame(int n, IScriptEnvironment* env_)
  {
    PNeoEnv env = env_;

    int pixelSize = vi.ComponentSize();
    switch (pixelSize) {
    case 1:
      return GetFrameT<uint8_t>(n, env);
    case 2:
      return GetFrameT<uint16_t>(n, env);
    default:
      env->ThrowError("[KFMPad] Unsupported pixel format");
      break;
    }

    return PVideoFrame();
  }

  static AVSValue __cdecl Create(AVSValue args, void* user_data, IScriptEnvironment* env)
  {
    return new KFMPad(
      args[0].AsClip(),       // src
      env
    );
  }
};


class AssumeDevice : public GenericVideoFilter
{
  int devices;
public:
  AssumeDevice(PClip clip, int devices)
    : GenericVideoFilter(clip)
    , devices(devices)
  { }

	int __stdcall SetCacheHints(int cachehints, int frame_range) {
		if (cachehints == CACHE_GET_DEV_TYPE) {
			return devices;
		}
		return 0;
	}

	static AVSValue __cdecl Create(AVSValue args, void* user_data, IScriptEnvironment* env)
	{
		return new AssumeDevice(args[0].AsClip(), args[1].AsInt());
	}
};

void AddFuncFMKernel(IScriptEnvironment* env)
{
  env->AddFunction("KPatchCombe", "ccccc", KPatchCombe::Create, 0);
  env->AddFunction("KFMSwitch", "ccccc[ucfclip]c[thswitch]f[show]b[showflag]b", KFMSwitch::Create, 0);
  env->AddFunction("KFMPad", "c", KFMPad::Create, 0);
	env->AddFunction("AssumeDevice", "ci", AssumeDevice::Create, 0);
}
