#include "hip/hip_runtime.h"

#include <stdint.h>
#include <avisynth.h>

#include <algorithm>
#include <memory>

#include "CommonFunctions.h"
#include "KFM.h"
#include "TextOut.h"

#include "VectorFunctions.cuh"
#include "ReduceKernel.cuh"
#include "KFMFilterBase.cuh"

bool cpu_contains_durty_block(const uint8_t* flagp, int fpitch, int nBlkX, int nBlkY, int* work, int thresh)
{
	for (int by = 0; by < nBlkY; ++by) {
		for (int bx = 0; bx < nBlkX; ++bx) {
			if (flagp[bx + by * fpitch] >= thresh) return true;
		}
	}
	return false;
}

__global__ void kl_init_contains_durty_block(int* work)
{
	*work = 0;
}

__global__ void kl_contains_durty_block(const uint8_t* flagp, int fpitch, int nBlkX, int nBlkY, int* work, int thresh)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < nBlkX && y < nBlkY) {
		if (flagp[x + y * fpitch] >= thresh) {
			*work = 1;
		}
	}
}

void cpu_binary_flag(
	uint8_t* dst, int dpitch, const uint8_t* src, int spitch, 
	int nBlkX, int nBlkY, int thresh)
{
	for (int y = 0; y < nBlkY; ++y) {
		for (int x = 0; x < nBlkX; ++x) {
			dst[x + y * dpitch] = ((src[x + y * spitch] >= thresh) ? 128 : 0);
		}
	}
}

__global__ void kl_binary_flag(
	uint8_t* dst, int dpitch, const uint8_t* src, int spitch, 
	int nBlkX, int nBlkY, int thresh)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < nBlkX && y < nBlkY) {
		dst[x + y * dpitch] = ((src[x + y * spitch] >= thresh) ? 128 : 0);
	}
}

void cpu_bilinear_x8_v(uint8_t* dst, int width, int height, int dpitch, const uint8_t* src, int spitch)
{
	for (int y = 0; y < height; ++y) {
		for (int x = 0; x < width; ++x) {
			int y0 = ((y - 4) >> 3);
			int c0 = ((y0 + 1) << 3) - (y - 4);
			int c1 = 8 - c0;
			auto s0 = src[x + (y0 + 0) * spitch];
			auto s1 = src[x + (y0 + 1) * spitch];
			dst[x + y * dpitch] = (s0 * c0 + s1 * c1 + 4) >> 3;
		}
	}
}

__global__ void kl_bilinear_x8_v(uint8_t* dst, int width, int height, int dpitch, const uint8_t* src, int spitch)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < width && y < height) {
		int y0 = ((y - 4) >> 3);
		int c0 = ((y0 + 1) << 3) - (y - 4);
		int c1 = 8 - c0;
		auto s0 = src[x + (y0 + 0) * spitch];
		auto s1 = src[x + (y0 + 1) * spitch];
		dst[x + y * dpitch] = (s0 * c0 + s1 * c1 + 4) >> 3;
	}
}

void cpu_bilinear_x8_h(uint8_t* dst, int width, int height, int dpitch, const uint8_t* src, int spitch)
{
	for (int y = 0; y < height; ++y) {
		for (int x = 0; x < width; ++x) {
			int x0 = ((x - 4) >> 3);
			int c0 = ((x0 + 1) << 3) - (x - 4);
			int c1 = 8 - c0;
			auto s0 = src[(x0 + 0) + y * spitch];
			auto s1 = src[(x0 + 1) + y * spitch];
			dst[x + y * dpitch] = (s0 * c0 + s1 * c1 + 4) >> 3;
		}
	}
}

__global__ void kl_bilinear_x8_h(uint8_t* dst, int width, int height, int dpitch, const uint8_t* src, int spitch)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < width && y < height) {
		int x0 = ((x - 4) >> 3);
		int c0 = ((x0 + 1) << 3) - (x - 4);
		int c1 = 8 - c0;
		auto s0 = src[(x0 + 0) + y * spitch];
		auto s1 = src[(x0 + 1) + y * spitch];
		dst[x + y * dpitch] = (s0 * c0 + s1 * c1 + 4) >> 3;
	}
}

template <typename vpixel_t, typename fpixel_t>
void cpu_merge(vpixel_t* dst,
	const vpixel_t* src24, const vpixel_t* src60, 
	int width, int height, int pitch, 
	const fpixel_t* flagp, int fpitch,
	int logx, int logy, int nBlkX, int nBlkY)
{
	for (int y = 0; y < height; ++y) {
		for (int x = 0; x < width; ++x) {
			int4 combe = to_int(flagp[(x << logx) + (y << logy) * fpitch]);
			int4 invcombe = VHelper<int4>::make(128) - combe;
			int4 tmp = (combe * to_int(src60[x + y * pitch]) + invcombe * to_int(src24[x + y * pitch]) + 64) >> 7;
			dst[x + y * pitch] = VHelper<vpixel_t>::cast_to(tmp);
		}
	}
}

template <typename vpixel_t, typename fpixel_t>
__global__ void kl_merge(vpixel_t* dst,
	const vpixel_t* src24, const vpixel_t* src60, 
	int width, int height, int pitch,
	const fpixel_t* flagp, int fpitch,
	int logx, int logy, int nBlkX, int nBlkY)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < width && y < height) {
		int4 combe = to_int(flagp[(x << logx) + (y << logy) * fpitch]);
		int4 invcombe = VHelper<int4>::make(128) - combe;
		int4 tmp = (combe * to_int(src60[x + y * pitch]) + invcombe * to_int(src24[x + y * pitch]) + 64) >> 7;
		dst[x + y * pitch] = VHelper<vpixel_t>::cast_to(tmp);
	}
}

enum KFMSWTICH_FLAG {
	FRAME_60 = 1,
	FRAME_24,
};

class KFMSwitch : public KFMFilterBase
{
	typedef uint8_t pixel_t;

	PClip clip24;
	PClip fmclip;
	PClip combeclip;
	float thswitch;
	float thpatch;
	bool show;
	bool showflag;

	int logUVx;
	int logUVy;
	int nBlkX, nBlkY;

	VideoInfo workvi;

	PulldownPatterns patterns;

	bool ContainsDurtyBlock(PVideoFrame& flag, PVideoFrame& work, int thpatch, PNeoEnv env)
	{
		const uint8_t* flagp = reinterpret_cast<const uint8_t*>(flag->GetReadPtr());
		int* pwork = reinterpret_cast<int*>(work->GetWritePtr());
		int pitch = flag->GetPitch();

		if (IS_CUDA) {
			dim3 threads(32, 16);
			dim3 blocks(nblocks(nBlkX, threads.x), nblocks(nBlkY, threads.y));
			kl_init_contains_durty_block << <1, 1 >> > (pwork);
			kl_contains_durty_block << <blocks, threads >> > (flagp, pitch, nBlkX, nBlkY, pwork, thpatch);
			int result;
			CUDA_CHECK(hipMemcpy(&result, pwork, sizeof(int), hipMemcpyDeviceToHost));
			return result != 0;
		}
		else {
			return cpu_contains_durty_block(flagp, pitch, nBlkX, nBlkY, pwork, thpatch);
		}
	}

	void MakeMergeFlag(PVideoFrame& dst, PVideoFrame& src, PVideoFrame& dsttmp, PVideoFrame& srctmp, int thpatch, PNeoEnv env)
	{
		const uint8_t* srcp = reinterpret_cast<const uint8_t*>(src->GetReadPtr());
		uint8_t* dstp = reinterpret_cast<uint8_t*>(dst->GetWritePtr());
		uint8_t* dsttmpp = reinterpret_cast<uint8_t*>(dsttmp->GetWritePtr()) + dsttmp->GetPitch();
		uint8_t* srctmpp = reinterpret_cast<uint8_t*>(srctmp->GetWritePtr());

		// 0��128��2�l�ɂ�����A���`��Ԃŉ摜�T�C�Y�܂Ŋg�� //

		if (IS_CUDA) {
			dim3 threads(32, 8);
			dim3 binary_blocks(nblocks(nBlkX, threads.x), nblocks(nBlkY, threads.y));
			kl_binary_flag << <binary_blocks, threads >> >(
				srctmpp, srctmp->GetPitch(), srcp, src->GetPitch(), nBlkX, nBlkY, thpatch);
			DEBUG_SYNC;
			{
				dim3 threads(32, 1);
				dim3 blocks(nblocks(nBlkX, threads.x));
				kl_padv << <blocks, threads >> > (srctmpp, nBlkX, nBlkY, srctmp->GetPitch(), 1);
				DEBUG_SYNC;
			}
			{
				dim3 threads(1, 32);
				dim3 blocks(1, nblocks(nBlkY, threads.y));
				kl_padh << <blocks, threads >> > (srctmpp, nBlkX, nBlkY + 1 * 2, srctmp->GetPitch(), 1);
				DEBUG_SYNC;
			}
			dim3 h_blocks(nblocks(vi.width, threads.x), nblocks(nBlkY, threads.y));
			kl_bilinear_x8_h << <h_blocks, threads >> >(
				dsttmpp, vi.width, nBlkY + 2, dsttmp->GetPitch(), srctmpp - srctmp->GetPitch(), srctmp->GetPitch());
			DEBUG_SYNC;
			dim3 v_blocks(nblocks(vi.width, threads.x), nblocks(vi.height, threads.y));
			kl_bilinear_x8_v << <v_blocks, threads >> >(
				dstp, vi.width, vi.height, dst->GetPitch(), dsttmpp + dsttmp->GetPitch(), dsttmp->GetPitch());
			DEBUG_SYNC;
		}
		else {
			cpu_binary_flag(srctmpp, srctmp->GetPitch(), srcp, src->GetPitch(), nBlkX, nBlkY, thpatch);
			cpu_padv(srctmpp, nBlkX, nBlkY, srctmp->GetPitch(), 1);
			cpu_padh(srctmpp, nBlkX, nBlkY + 1 * 2, srctmp->GetPitch(), 1);
			// �㉺�p�f�B���O1�s�����܂߂ď���
			cpu_bilinear_x8_h(dsttmpp, vi.width, nBlkY + 2, dsttmp->GetPitch(), srctmpp - srctmp->GetPitch(), srctmp->GetPitch());
			// �\�[�X�̓p�f�B���O1�s�����X�L�b�v���ēn��
			cpu_bilinear_x8_v(dstp, vi.width, vi.height, dst->GetPitch(), dsttmpp + dsttmp->GetPitch(), dsttmp->GetPitch());
		}
	}

	template <typename pixel_t>
	void MergeBlock(PVideoFrame& src24, PVideoFrame& src60, PVideoFrame& flag, PVideoFrame& dst, PNeoEnv env)
	{
		typedef typename VectorType<pixel_t>::type vpixel_t;
		const vpixel_t* src24Y = reinterpret_cast<const vpixel_t*>(src24->GetReadPtr(PLANAR_Y));
		const vpixel_t* src24U = reinterpret_cast<const vpixel_t*>(src24->GetReadPtr(PLANAR_U));
		const vpixel_t* src24V = reinterpret_cast<const vpixel_t*>(src24->GetReadPtr(PLANAR_V));
		const vpixel_t* src60Y = reinterpret_cast<const vpixel_t*>(src60->GetReadPtr(PLANAR_Y));
		const vpixel_t* src60U = reinterpret_cast<const vpixel_t*>(src60->GetReadPtr(PLANAR_U));
		const vpixel_t* src60V = reinterpret_cast<const vpixel_t*>(src60->GetReadPtr(PLANAR_V));
		vpixel_t* dstY = reinterpret_cast<vpixel_t*>(dst->GetWritePtr(PLANAR_Y));
		vpixel_t* dstU = reinterpret_cast<vpixel_t*>(dst->GetWritePtr(PLANAR_U));
		vpixel_t* dstV = reinterpret_cast<vpixel_t*>(dst->GetWritePtr(PLANAR_V));
		const uchar4* flagp = reinterpret_cast<const uchar4*>(flag->GetReadPtr());

		int pitchY = src24->GetPitch(PLANAR_Y) / sizeof(vpixel_t);
		int pitchUV = src24->GetPitch(PLANAR_U) / sizeof(vpixel_t);
		int width4 = vi.width >> 2;
		int width4UV = width4 >> logUVx;
		int heightUV = vi.height >> logUVy;
		int fpitch4 = flag->GetPitch() / sizeof(uchar4);

		if (IS_CUDA) {
			dim3 threads(32, 16);
			dim3 blocks(nblocks(width4, threads.x), nblocks(vi.height, threads.y));
			dim3 blocksUV(nblocks(width4UV, threads.x), nblocks(heightUV, threads.y));
			kl_merge << <blocks, threads >> >(
				dstY, src24Y, src60Y, width4, vi.height, pitchY, flagp, fpitch4, 0, 0, nBlkX, nBlkY);
			DEBUG_SYNC;
			kl_merge << <blocksUV, threads >> >(
				dstU, src24U, src60U, width4UV, heightUV, pitchUV, flagp, fpitch4, logUVx, logUVy, nBlkX, nBlkY);
			DEBUG_SYNC;
			kl_merge << <blocksUV, threads >> >(
				dstV, src24V, src60V, width4UV, heightUV, pitchUV, flagp, fpitch4, logUVx, logUVy, nBlkX, nBlkY);
			DEBUG_SYNC;
		}
		else {
			cpu_merge(dstY, src24Y, src60Y, width4, vi.height, pitchY, flagp, fpitch4, 0, 0, nBlkX, nBlkY);
			cpu_merge(dstU, src24U, src60U, width4UV, heightUV, pitchUV, flagp, fpitch4, logUVx, logUVy, nBlkX, nBlkY);
			cpu_merge(dstV, src24V, src60V, width4UV, heightUV, pitchUV, flagp, fpitch4, logUVx, logUVy, nBlkX, nBlkY);
		}
	}

	template <typename pixel_t>
	void VisualizeFlag(PVideoFrame& dst, PVideoFrame& mf, PNeoEnv env)
	{
		// ���茋�ʂ�\��
		int blue[] = { 73, 230, 111 };

		const uint8_t* mfp = reinterpret_cast<const uint8_t*>(mf->GetReadPtr());
		pixel_t* dstY = reinterpret_cast<pixel_t*>(dst->GetWritePtr(PLANAR_Y));
		pixel_t* dstU = reinterpret_cast<pixel_t*>(dst->GetWritePtr(PLANAR_U));
		pixel_t* dstV = reinterpret_cast<pixel_t*>(dst->GetWritePtr(PLANAR_V));

		int mfpitch = mf->GetPitch(PLANAR_Y) / sizeof(uint8_t);
		int dstPitchY = dst->GetPitch(PLANAR_Y) / sizeof(pixel_t);
		int dstPitchUV = dst->GetPitch(PLANAR_U) / sizeof(pixel_t);

		// �F��t����
		for (int y = 0; y < vi.height; ++y) {
			for (int x = 0; x < vi.width; ++x) {
				int score = mfp[x + y * mfpitch];
				int offY = x + y * dstPitchY;
				int offUV = (x >> logUVx) + (y >> logUVy) * dstPitchUV;
				dstY[offY] = (blue[0] * score + dstY[offY] * (128 - score)) >> 7;
				dstU[offUV] = (blue[1] * score + dstU[offUV] * (128 - score)) >> 7;
				dstV[offUV] = (blue[2] * score + dstV[offUV] * (128 - score)) >> 7;
			}
		}
	}

	template <typename pixel_t>
	PVideoFrame InternalGetFrame(int n60, PVideoFrame& fmframe, int& type, PNeoEnv env)
	{
		int cycleIndex = n60 / 10;
		int kfmPattern = (int)fmframe->GetProperty("KFM_Pattern")->GetInt();
		float kfmCost = (float)fmframe->GetProperty("KFM_Cost")->GetFloat();

		if (kfmCost > thswitch || PulldownPatterns::Is30p(kfmPattern)) {
			// �R�X�g�������̂�60p�Ɣ��f or 30p�̏ꍇ
			PVideoFrame frame60 = child->GetFrame(n60, env);
			type = FRAME_60;
			return frame60;
		}

		type = FRAME_24;

		// 24p�t���[���ԍ����擾
		Frame24Info frameInfo = patterns.GetFrame60(kfmPattern, n60);
		int n24 = frameInfo.cycleIndex * 4 + frameInfo.frameIndex;

		if (frameInfo.frameIndex < 0) {
			// �O�ɋ󂫂�����̂őO�̃T�C�N��
			n24 = frameInfo.cycleIndex * 4 - 1;
		}
		else if (frameInfo.frameIndex >= 4) {
			// ���̃T�C�N���̃p�^�[�����擾
			PVideoFrame nextfmframe = fmclip->GetFrame(cycleIndex + 1, env);
			int nextPattern = (int)nextfmframe->GetProperty("KFM_Pattern")->GetInt();
			int fstart = patterns.GetFrame24(nextPattern, 0).fieldStartIndex;
			if (fstart > 0) {
				// �O�ɋ󂫂�����̂őO�̃T�C�N��
				n24 = frameInfo.cycleIndex * 4 + 3;
			}
			else {
				// �O�ɋ󂫂��Ȃ��̂Ō��̃T�C�N��
				n24 = frameInfo.cycleIndex * 4 + 4;
			}
		}

		PVideoFrame frame24 = clip24->GetFrame(n24, env);
		PVideoFrame flag = combeclip->GetFrame(n24, env)->GetProperty(COMBE_FLAG_STR)->GetFrame();

		{
			PVideoFrame work = env->NewVideoFrame(workvi);
			if (ContainsDurtyBlock(flag, work, (int)thpatch, env) == false) {
				// �_���ȃu���b�N�͂Ȃ��̂ł��̂܂ܕԂ�
				return frame24;
			}
		}

		PVideoFrame frame60 = child->GetFrame(n60, env);

		VideoInfo mfvi = vi;
		mfvi.pixel_type = VideoInfo::CS_Y8;
		PVideoFrame mflag = env->NewVideoFrame(mfvi);

		{
			// �}�[�W�p�t���O�쐬
			PVideoFrame mflagtmp = env->NewVideoFrame(mfvi);
			PVideoFrame flagtmp = NewSwitchFlagFrame(vi, env->GetProperty(AEP_FRAME_ALIGN), 2, env);
			MakeMergeFlag(mflag, flag, mflagtmp, flagtmp, (int)thpatch, env);
		}

		if (!IS_CUDA && vi.ComponentSize() == 1 && showflag) {
			env->MakeWritable(&frame24);
			VisualizeFlag<pixel_t>(frame24, mflag, env);
			return frame24;
		}

		// �_���ȃu���b�N��60p�t���[������R�s�[
		PVideoFrame dst = env->NewVideoFrame(vi);
		MergeBlock<pixel_t>(frame24, frame60, mflag, dst, env);

		return dst;
	}

	void DrawInfo(PVideoFrame& dst, const char* fps, int pattern, float score, IScriptEnvironment* env) {
		env->MakeWritable(&dst);

		char buf[100]; sprintf(buf, "KFMSwitch: %s pattern:%2d cost:%.1f", fps, pattern, score);
		DrawText(dst, true, 0, 0, buf);
	}

public:
	KFMSwitch(PClip clip60, PClip clip24, PClip fmclip, PClip combeclip,
		float thswitch, float thpatch, bool show, bool showflag, IScriptEnvironment* env)
		: KFMFilterBase(clip60)
		, clip24(clip24)
		, fmclip(fmclip)
		, combeclip(combeclip)
		, thswitch(thswitch)
		, thpatch(thpatch)
		, show(show)
		, showflag(showflag)
		, logUVx(vi.GetPlaneWidthSubsampling(PLANAR_U))
		, logUVy(vi.GetPlaneHeightSubsampling(PLANAR_U))
	{
		if (vi.width & 7) env->ThrowError("[KFMSwitch]: width must be multiple of 8");
		if (vi.height & 7) env->ThrowError("[KFMSwitch]: height must be multiple of 8");

		nBlkX = nblocks(vi.width, OVERLAP);
		nBlkY = nblocks(vi.height, OVERLAP);

		int work_bytes = sizeof(int);
		workvi.pixel_type = VideoInfo::CS_BGR32;
		workvi.width = 4;
		workvi.height = nblocks(work_bytes, workvi.width * 4);
	}

	PVideoFrame __stdcall GetFrame(int n60, IScriptEnvironment* env_)
	{
		PNeoEnv env = env_;

		int cycleIndex = n60 / 10;
		PVideoFrame fmframe = fmclip->GetFrame(cycleIndex, env);
		int frameType;

		PVideoFrame dst;
		int pixelSize = vi.ComponentSize();
		switch (pixelSize) {
		case 1:
			dst = InternalGetFrame<uint8_t>(n60, fmframe, frameType, env);
			break;
		case 2:
			dst = InternalGetFrame<uint16_t>(n60, fmframe, frameType, env);
			break;
		default:
			env->ThrowError("[KFMSwitch] Unsupported pixel format");
			break;
		}

		if (!IS_CUDA && pixelSize == 1 && show) {
			const std::pair<int, float>* pfm = (std::pair<int, float>*)fmframe->GetReadPtr();
			const char* fps = (frameType == FRAME_60) ? "60p" : "24p";
			DrawInfo(dst, fps, pfm->first, pfm->second, env);
		}

		return dst;
	}

	static AVSValue __cdecl Create(AVSValue args, void* user_data, IScriptEnvironment* env)
	{
		return new KFMSwitch(
			args[0].AsClip(),           // clip60
			args[1].AsClip(),           // clip24

			args[2].AsClip(),           // fmclip
			args[3].AsClip(),           // combeclip
			(float)args[4].AsFloat(0.8f),// thswitch
			(float)args[5].AsFloat(40.0f),// thpatch
			args[6].AsBool(false),      // show
			args[7].AsBool(false),      // showflag
			env
			);
	}
};

class AssertOnCUDA : public GenericVideoFilter
{
public:
	AssertOnCUDA(PClip clip) : GenericVideoFilter(clip) { }

	int __stdcall SetCacheHints(int cachehints, int frame_range) {
		if (cachehints == CACHE_GET_DEV_TYPE) {
			return DEV_TYPE_CUDA;
		}
		return 0;
	}

	static AVSValue __cdecl Create(AVSValue args, void* user_data, IScriptEnvironment* env)
	{
		return new AssertOnCUDA(args[0].AsClip());
	}
};

void AddFuncFMKernel(IScriptEnvironment* env)
{
	env->AddFunction("KFMSwitch", "cccc[thswitch]f[thpatch]f[show]b[showflag]b", KFMSwitch::Create, 0);
	env->AddFunction("AssertOnCUDA", "c", AssertOnCUDA::Create, 0);
}
