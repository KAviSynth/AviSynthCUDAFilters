#include "hip/hip_runtime.h"

#include <stdint.h>
#include <avisynth.h>

#include <algorithm>
#include "CommonFunctions.h"
#include "KFM.h"
#include "VectorFunctions.cuh"
#include "KFMFilterBase.cuh"


int scaleParam(float thresh, int pixelBits)
{
  return (int)(thresh * (1 << (pixelBits - 8)) + 0.5f);
}

int Get8BitType(VideoInfo& vi) {
  if (vi.Is420()) return VideoInfo::CS_YV12;
  else if (vi.Is422()) return VideoInfo::CS_YV16;
  else if (vi.Is444()) return VideoInfo::CS_YV24;
  // ����ȊO�͒m���
  return VideoInfo::CS_BGR24;
}

PVideoFrame NewSwitchFlagFrame(VideoInfo vi, int hpad, int vpad, PNeoEnv env)
{
  typedef typename VectorType<uint8_t>::type vpixel_t;

  VideoInfo blockpadvi = vi;
  blockpadvi.width = nblocks(vi.width, OVERLAP) + hpad * 2;
  blockpadvi.height = nblocks(vi.height, OVERLAP) + vpad * 2;
  blockpadvi.pixel_type = VideoInfo::CS_Y8;
  PVideoFrame frame = env->NewVideoFrame(blockpadvi);

  // �[��������
  vpixel_t* flagp = reinterpret_cast<vpixel_t*>(frame->GetWritePtr());
  int pitch = frame->GetPitch() / sizeof(vpixel_t);
  int width = frame->GetPitch() / sizeof(vpixel_t);
  if (IS_CUDA) {
    dim3 threads(32, 8);
    dim3 blocks(nblocks(width, threads.x), nblocks(blockpadvi.height, threads.y));
    kl_fill<vpixel_t, 0> << <blocks, threads >> >(flagp, width, blockpadvi.height, pitch);
  }
  else {
    cpu_fill<vpixel_t, 0>(flagp, width, blockpadvi.height, pitch);
  }

  return env->SubframePlanar(frame,
    hpad * sizeof(uint8_t) + frame->GetPitch(PLANAR_Y) * vpad,
    frame->GetPitch(PLANAR_Y),
    frame->GetRowSize(PLANAR_Y) - hpad * 2 * sizeof(uint8_t),
    frame->GetHeight(PLANAR_Y) - vpad * 2,
    0, 0, 0);
}

template <typename pixel_t, int fill_v>
void cpu_fill(pixel_t* dst, int width, int height, int pitch)
{
  for (int y = 0; y < height; ++y) {
    for (int x = 0; x < width; ++x) {
      dst[x + y * pitch] = VHelper<pixel_t>::make(fill_v);
    }
  }
}

template <typename pixel_t, int fill_v>
__global__ void kl_fill(pixel_t* dst, int width, int height, int pitch)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < width && y < height) {
    dst[x + y * pitch] = VHelper<pixel_t>::make(fill_v);
  }
}

template <typename pixel_t>
void cpu_copy(pixel_t* dst, const pixel_t* __restrict__ src, int width, int height, int pitch)
{
  for (int y = 0; y < height; ++y) {
    for (int x = 0; x < width; ++x) {
      dst[x + y * pitch] = src[x + y * pitch];
    }
  }
}

template <typename pixel_t>
__global__ void kl_copy(pixel_t* dst, const pixel_t* __restrict__ src, int width, int height, int pitch)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < width && y < height) {
    dst[x + y * pitch] = src[x + y * pitch];
  }
}

template <typename pixel_t>
void cpu_average(pixel_t* dst, const pixel_t* __restrict__ src0, const pixel_t* __restrict__ src1, int width, int height, int pitch)
{
  for (int y = 0; y < height; ++y) {
    for (int x = 0; x < width; ++x) {
      auto tmp = (to_int(src0[x + y * pitch]) + to_int(src1[x + y * pitch])) >> 1;
      dst[x + y * pitch] = VHelper<pixel_t>::cast_to(tmp);
    }
  }
}

template void cpu_average(uchar4* dst, const uchar4* __restrict__ src0, const uchar4* __restrict__ src1, int width, int height, int pitch);
template void cpu_average(ushort4* dst, const ushort4* __restrict__ src0, const ushort4* __restrict__ src1, int width, int height, int pitch);

template <typename pixel_t>
__global__ void kl_average(pixel_t* dst, const pixel_t* __restrict__ src0, const pixel_t* __restrict__ src1, int width, int height, int pitch)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < width && y < height) {
    auto tmp = (to_int(src0[x + y * pitch]) + to_int(src1[x + y * pitch])) >> 1;
    dst[x + y * pitch] = VHelper<pixel_t>::cast_to(tmp);
  }
}

template __global__ void kl_average(uchar4* dst, const uchar4* __restrict__ src0, const uchar4* __restrict__ src1, int width, int height, int pitch);
template __global__ void kl_average(ushort4* dst, const ushort4* __restrict__ src0, const ushort4* __restrict__ src1, int width, int height, int pitch);

template <typename pixel_t>
void cpu_padv(pixel_t* dst, int width, int height, int pitch, int vpad)
{
  for (int y = 0; y < vpad; ++y) {
    for (int x = 0; x < width; ++x) {
      dst[x + (-y - 1) * pitch] = dst[x + (y)* pitch];
      dst[x + (height + y) * pitch] = dst[x + (height - y - 1)* pitch];
    }
  }
}

template void cpu_padv(uint8_t* dst, int width, int height, int pitch, int vpad);

template <typename pixel_t>
__global__ void kl_padv(pixel_t* dst, int width, int height, int pitch, int vpad)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y;

  if (x < width) {
    dst[x + (-y - 1) * pitch] = dst[x + (y)* pitch];
    dst[x + (height + y) * pitch] = dst[x + (height - y - 1)* pitch];
  }
}

template __global__ void kl_padv(uint8_t* dst, int width, int height, int pitch, int vpad);

template <typename pixel_t>
void cpu_padh(pixel_t* dst, int width, int height, int pitch, int hpad)
{
  for (int y = 0; y < height; ++y) {
    for (int x = 0; x < hpad; ++x) {
      dst[(-x - 1) + y * pitch] = dst[(x)+y * pitch];
      dst[(width + x) + y * pitch] = dst[(width - x - 1) + y * pitch];
    }
  }
}

template void cpu_padh(uint8_t* dst, int width, int height, int pitch, int hpad);

template <typename pixel_t>
__global__ void kl_padh(pixel_t* dst, int width, int height, int pitch, int hpad)
{
  int x = threadIdx.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (y < height) {
    dst[(-x - 1) + y * pitch] = dst[(x)+y * pitch];
    dst[(width + x) + y * pitch] = dst[(width - x - 1) + y * pitch];
  }
}

template __global__ void kl_padh(uint8_t* dst, int width, int height, int pitch, int hpad);

template <typename pixel_t>
void cpu_copy_border(pixel_t* dst,
  const pixel_t* src, int width, int height, int pitch, int vborder)
{
  for (int y = 0; y < vborder; ++y) {
    for (int x = 0; x < width; ++x) {
      dst[x + y * pitch] = src[x + y * pitch];
      dst[x + (height - y - 1) * pitch] = src[x + (height - y - 1) * pitch];
    }
  }
}

template void cpu_copy_border(uint8_t* dst,
  const uint8_t* src, int width, int height, int pitch, int vborder);

template <typename pixel_t>
__global__ void kl_copy_border(pixel_t* dst,
  const pixel_t* __restrict__ src, int width, int height, int pitch, int vborder)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y;

  if (x < width) {
    dst[x + y * pitch] = src[x + y * pitch];
    dst[x + (height - y - 1) * pitch] = src[x + (height - y - 1) * pitch];
  }
}

__device__ __host__ uint8_t MakeDiffFlag(int t, int diff, int threshM, int threshS, int threshLS) {
  uint8_t flag = 0;
  if (t > threshS) flag |= SHIMA;
  if (t > threshLS) flag |= LSHIMA;
  if (diff > threshM) flag |= MOVE;
  return flag;
}

// sref��base-1���C��
template <typename vpixel_t>
void cpu_analyze_frame(uchar4* dst, int dstPitch,
  const vpixel_t* base, const vpixel_t* sref, const vpixel_t* mref,
  int width, int height, int pitch, int threshM, int threshS, int threshLS)
{
  for (int y = 0; y < height; ++y) {
    for (int x = 0; x < width; ++x) {
      vpixel_t a = base[x + (y - 1) * pitch];
      vpixel_t b = sref[x + y * pitch];
      vpixel_t c = base[x + y * pitch];
      vpixel_t d = sref[x + (y + 1) * pitch];
      vpixel_t e = base[x + (y + 1) * pitch];
      int4 t = CalcCombe(to_int(a), to_int(b), to_int(c), to_int(d), to_int(e));
      int4 diff = absdiff(mref[x + y * pitch], c);
      uchar4 flags = {
        MakeDiffFlag(t.x, diff.x, threshM, threshS, threshLS),
        MakeDiffFlag(t.y, diff.y, threshM, threshS, threshLS),
        MakeDiffFlag(t.z, diff.z, threshM, threshS, threshLS),
        MakeDiffFlag(t.w, diff.w, threshM, threshS, threshLS),
      };
      // �t���O�i�[
      dst[x + y * dstPitch] = flags;
    }
  }
}

template <typename vpixel_t>
__global__ void kl_analyze_frame(uchar4* dst, int dstPitch,
  const vpixel_t* __restrict__ base,
  const vpixel_t* __restrict__ sref,
  const vpixel_t* __restrict__ mref,
  int width, int height, int pitch, int threshM, int threshS, int threshLS)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < width && y < height) {
    vpixel_t a = base[x + (y - 1) * pitch];
    vpixel_t b = sref[x + y * pitch];
    vpixel_t c = base[x + y * pitch];
    vpixel_t d = sref[x + (y + 1) * pitch];
    vpixel_t e = base[x + (y + 1) * pitch];
    int4 t = CalcCombe(to_int(a), to_int(b), to_int(c), to_int(d), to_int(e));
    int4 diff = absdiff(mref[x + y * pitch], c);
    uchar4 flags = {
      MakeDiffFlag(t.x, diff.x, threshM, threshS, threshLS),
      MakeDiffFlag(t.y, diff.y, threshM, threshS, threshLS),
      MakeDiffFlag(t.z, diff.z, threshM, threshS, threshLS),
      MakeDiffFlag(t.w, diff.w, threshM, threshS, threshLS),
    };
    // �t���O�i�[
    dst[x + y * dstPitch] = flags;
  }
}

void cpu_merge_uvflags(uint8_t* fY,
  const uint8_t* fU, const uint8_t* fV,
  int width, int height, int pitchY, int pitchUV, int logUVx, int logUVy)
{
  for (int y = 0; y < height; ++y) {
    for (int x = 0; x < width; ++x) {
      int offUV = (x >> logUVx) + (y >> logUVy) * pitchUV;
      int flagUV = fU[offUV] | fV[offUV];
      fY[x + y * pitchY] |= (flagUV << 4);
    }
  }
}

__global__ void kl_merge_uvflags(uint8_t* fY,
  const uint8_t* __restrict__ fU, const uint8_t* __restrict__ fV,
  int width, int height, int pitchY, int pitchUV, int logUVx, int logUVy)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < width && y < height) {
    int offUV = (x >> logUVx) + (y >> logUVy) * pitchUV;
    int flagUV = fU[offUV] | fV[offUV];
    fY[x + y * pitchY] |= (flagUV << 4);
  }
}

template <typename pixel_t>
void cpu_merge_uvcoefs(pixel_t* fY,
  const pixel_t* fU, const pixel_t* fV,
  int width, int height, int pitchY, int pitchUV, int logUVx, int logUVy)
{
  for (int y = 0; y < height; ++y) {
    for (int x = 0; x < width; ++x) {
      int offUV = (x >> logUVx) + (y >> logUVy) * pitchUV;
      fY[x + y * pitchY] = max(fY[x + y * pitchY], max(fU[offUV], fV[offUV]));
    }
  }
}

template <typename pixel_t>
__global__ void kl_merge_uvcoefs(pixel_t* fY,
  const pixel_t* __restrict__ fU, const pixel_t* __restrict__ fV,
  int width, int height, int pitchY, int pitchUV, int logUVx, int logUVy)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < width && y < height) {
    int offUV = (x >> logUVx) + (y >> logUVy) * pitchUV;
    fY[x + y * pitchY] = max(fY[x + y * pitchY], max(fU[offUV], fV[offUV]));
  }
}

template <typename vpixel_t>
void cpu_and_coefs(vpixel_t* dstp, const vpixel_t* diffp,
  int width, int height, int pitch, float invcombe, float invdiff)
{
  for (int y = 0; y < height; ++y) {
    for (int x = 0; x < width; ++x) {
      float4 combe = clamp(to_float(dstp[x + y * pitch]) * invcombe + (-1.0f), -0.5f, 0.5f);
      float4 diff = clamp(to_float(diffp[x + y * pitch]) * (-invdiff) + 1.0f, -0.5f, 0.5f);
      float4 tmp = max(combe + diff, 0.0f) * 128.0f + 0.5f;
      dstp[x + y * pitch] = VHelper<vpixel_t>::cast_to(tmp);
    }
  }
}

template void cpu_and_coefs(uchar4* dstp, const uchar4* diffp,
  int width, int height, int pitch, float invcombe, float invdiff);
template void cpu_and_coefs(ushort4* dstp, const ushort4* diffp,
  int width, int height, int pitch, float invcombe, float invdiff);

template <typename pixel_t>
void cpu_apply_uvcoefs_420(
  const pixel_t* fY, pixel_t* fU, pixel_t* fV,
  int widthUV, int heightUV, int pitchY, int pitchUV)
{
  for (int y = 0; y < heightUV; ++y) {
    for (int x = 0; x < widthUV; ++x) {
      int v =
        fY[(x * 2 + 0) + (y * 2 + 0) * pitchY] + fY[(x * 2 + 1) + (y * 2 + 0) * pitchY] +
        fY[(x * 2 + 0) + (y * 2 + 1) * pitchY] + fY[(x * 2 + 1) + (y * 2 + 1) * pitchY];
      fU[x + y * pitchUV] = fV[x + y * pitchUV] = (v + 2) >> 2;
    }
  }
}

template <typename pixel_t>
__global__ void kl_apply_uvcoefs_420(
  const pixel_t* __restrict__ fY, pixel_t* fU, pixel_t* fV,
  int widthUV, int heightUV, int pitchY, int pitchUV)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < widthUV && y < heightUV) {
    int v =
      fY[(x * 2 + 0) + (y * 2 + 0) * pitchY] + fY[(x * 2 + 1) + (y * 2 + 0) * pitchY] +
      fY[(x * 2 + 0) + (y * 2 + 1) * pitchY] + fY[(x * 2 + 1) + (y * 2 + 1) * pitchY];
    fU[x + y * pitchUV] = fV[x + y * pitchUV] = (v + 2) >> 2;
  }
}

template <typename vpixel_t>
void cpu_extend_coef(vpixel_t* dst, const vpixel_t* src, int width, int height, int pitch)
{
  for (int y = 0; y < height; ++y) {
    for (int x = 0; x < width; ++x) {
      int4 tmp = max(to_int(src[x + (y - 1) * pitch]), max(to_int(src[x + y * pitch]), to_int(src[x + (y + 1) * pitch])));
      dst[x + y * pitch] = VHelper<vpixel_t>::cast_to(tmp);
    }
  }
}

template <typename vpixel_t>
__global__ void kl_extend_coef(vpixel_t* dst, const vpixel_t* __restrict__ src, int width, int height, int pitch)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < width && y < height) {
    int4 tmp = max(to_int(src[x + (y - 1) * pitch]), max(to_int(src[x + y * pitch]), to_int(src[x + (y + 1) * pitch])));
    dst[x + y * pitch] = VHelper<vpixel_t>::cast_to(tmp);
  }
}

template <typename vpixel_t>
void cpu_calc_combe(vpixel_t* dst, const vpixel_t* src, int width, int height, int pitch)
{
  for (int y = 0; y < height; ++y) {
    for (int x = 0; x < width; ++x) {
      int4 combe = CalcCombe(
        to_int(src[x + (y - 2) * pitch]),
        to_int(src[x + (y - 1) * pitch]),
        to_int(src[x + (y + 0) * pitch]),
        to_int(src[x + (y + 1) * pitch]),
        to_int(src[x + (y + 2) * pitch]));

      int4 tmp = clamp(combe >> 2, 0, 255);
      dst[x + y * pitch] = VHelper<vpixel_t>::cast_to(tmp);
    }
  }
}

template <typename vpixel_t>
__global__ void kl_calc_combe(vpixel_t* dst, const vpixel_t* __restrict__ src, int width, int height, int pitch)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < width && y < height) {
    int4 combe = CalcCombe(
      to_int(src[x + (y - 2) * pitch]),
      to_int(src[x + (y - 1) * pitch]),
      to_int(src[x + (y + 0) * pitch]),
      to_int(src[x + (y + 1) * pitch]),
      to_int(src[x + (y + 2) * pitch]));

    int4 tmp = clamp(combe >> 2, 0, 255);
    dst[x + y * pitch] = VHelper<vpixel_t>::cast_to(tmp);
  }
}


template <typename pixel_t>
void KFMFilterBase::CopyFrame(PVideoFrame& src, PVideoFrame& dst, PNeoEnv env)
{
  typedef typename VectorType<pixel_t>::type vpixel_t;
  const vpixel_t* srcY = reinterpret_cast<const vpixel_t*>(src->GetReadPtr(PLANAR_Y));
  const vpixel_t* srcU = reinterpret_cast<const vpixel_t*>(src->GetReadPtr(PLANAR_U));
  const vpixel_t* srcV = reinterpret_cast<const vpixel_t*>(src->GetReadPtr(PLANAR_V));
  vpixel_t* dstY = reinterpret_cast<vpixel_t*>(dst->GetWritePtr(PLANAR_Y));
  vpixel_t* dstU = reinterpret_cast<vpixel_t*>(dst->GetWritePtr(PLANAR_U));
  vpixel_t* dstV = reinterpret_cast<vpixel_t*>(dst->GetWritePtr(PLANAR_V));

  int pitchY = src->GetPitch(PLANAR_Y) / sizeof(vpixel_t);
  int pitchUV = src->GetPitch(PLANAR_U) / sizeof(vpixel_t);
  int width4 = srcvi.width >> 2;
  int width4UV = width4 >> logUVx;
  int heightUV = srcvi.height >> logUVy;

  if (IS_CUDA) {
    dim3 threads(32, 16);
    dim3 blocks(nblocks(width4, threads.x), nblocks(srcvi.height, threads.y));
    dim3 blocksUV(nblocks(width4UV, threads.x), nblocks(heightUV, threads.y));
    kl_copy << <blocks, threads >> >(dstY, srcY, width4, srcvi.height, pitchY);
    DEBUG_SYNC;
    kl_copy << <blocksUV, threads >> >(dstU, srcU, width4UV, heightUV, pitchUV);
    DEBUG_SYNC;
    kl_copy << <blocksUV, threads >> >(dstV, srcV, width4UV, heightUV, pitchUV);
    DEBUG_SYNC;
  }
  else {
    cpu_copy<vpixel_t>(dstY, srcY, width4, srcvi.height, pitchY);
    cpu_copy<vpixel_t>(dstU, srcU, width4UV, heightUV, pitchUV);
    cpu_copy<vpixel_t>(dstV, srcV, width4UV, heightUV, pitchUV);
  }
}

template void KFMFilterBase::CopyFrame<uint8_t>(PVideoFrame& src, PVideoFrame& dst, PNeoEnv env);
template void KFMFilterBase::CopyFrame<uint16_t>(PVideoFrame& src, PVideoFrame& dst, PNeoEnv env);


template <typename pixel_t>
void KFMFilterBase::PadFrame(PVideoFrame& dst, PNeoEnv env)
{
  typedef typename VectorType<pixel_t>::type vpixel_t;
  vpixel_t* dstY = reinterpret_cast<vpixel_t*>(dst->GetWritePtr(PLANAR_Y));
  vpixel_t* dstU = reinterpret_cast<vpixel_t*>(dst->GetWritePtr(PLANAR_U));
  vpixel_t* dstV = reinterpret_cast<vpixel_t*>(dst->GetWritePtr(PLANAR_V));

  int pitchY = dst->GetPitch(PLANAR_Y) / sizeof(vpixel_t);
  int pitchUV = dst->GetPitch(PLANAR_U) / sizeof(vpixel_t);
  int width4 = srcvi.width >> 2;
  int width4UV = width4 >> logUVx;
  int heightUV = srcvi.height >> logUVy;
  int vpadUV = VPAD >> logUVy;

  if (IS_CUDA) {
    dim3 threads(32, VPAD);
    dim3 blocks(nblocks(width4, threads.x));
    dim3 threadsUV(32, vpadUV);
    dim3 blocksUV(nblocks(width4UV, threads.x));
    kl_padv << <blocks, threads >> >(dstY, width4, srcvi.height, pitchY, VPAD);
    DEBUG_SYNC;
    kl_padv << <blocksUV, threadsUV >> >(dstU, width4UV, heightUV, pitchUV, vpadUV);
    DEBUG_SYNC;
    kl_padv << <blocksUV, threadsUV >> >(dstV, width4UV, heightUV, pitchUV, vpadUV);
    DEBUG_SYNC;
  }
  else {
    cpu_padv<vpixel_t>(dstY, width4, srcvi.height, pitchY, VPAD);
    cpu_padv<vpixel_t>(dstU, width4UV, heightUV, pitchUV, vpadUV);
    cpu_padv<vpixel_t>(dstV, width4UV, heightUV, pitchUV, vpadUV);
  }
}

template void KFMFilterBase::PadFrame<uint8_t>(PVideoFrame& dst, PNeoEnv env);
template void KFMFilterBase::PadFrame<uint16_t>(PVideoFrame& dst, PNeoEnv env);

template <typename vpixel_t>
void KFMFilterBase::LaunchAnalyzeFrame(uchar4* dst, int dstPitch,
  const vpixel_t* base, const vpixel_t* sref, const vpixel_t* mref,
  int width, int height, int pitch, int threshM, int threshS, int threshLS,
  PNeoEnv env)
{
  if (IS_CUDA) {
    dim3 threads(32, 16);
    dim3 blocks(nblocks(width, threads.x), nblocks(height, threads.y));
    kl_analyze_frame << <blocks, threads >> >(
      dst, dstPitch, base, sref, mref, width, height, pitch, threshM, threshS, threshLS);
  }
  else {
    cpu_analyze_frame(
      dst, dstPitch, base, sref, mref, width, height, pitch, threshM, threshS, threshLS);
  }
}

template void KFMFilterBase::LaunchAnalyzeFrame(uchar4* dst, int dstPitch,
  const uchar4* base, const uchar4* sref, const uchar4* mref,
  int width, int height, int pitch, int threshM, int threshS, int threshLS,
  PNeoEnv env);
template void KFMFilterBase::LaunchAnalyzeFrame(uchar4* dst, int dstPitch,
  const ushort4* base, const ushort4* sref, const ushort4* mref,
  int width, int height, int pitch, int threshM, int threshS, int threshLS,
  PNeoEnv env);

template <typename pixel_t>
void KFMFilterBase::AnalyzeFrame(PVideoFrame& f0, PVideoFrame& f1, PVideoFrame& flag,
  const FrameAnalyzeParam* prmY, const FrameAnalyzeParam* prmC, PNeoEnv env)
{
  typedef typename VectorType<pixel_t>::type vpixel_t;

  int planes[] = { PLANAR_Y, PLANAR_U, PLANAR_V };

  // �e�v���[���𔻒�
  for (int pi = 0; pi < 3; ++pi) {
    int p = planes[pi];

    const vpixel_t* f0p = reinterpret_cast<const vpixel_t*>(f0->GetReadPtr(p));
    const vpixel_t* f1p = reinterpret_cast<const vpixel_t*>(f1->GetReadPtr(p));
    uchar4* flagp = reinterpret_cast<uchar4*>(flag->GetWritePtr(p));
    int pitch = f0->GetPitch(p) / sizeof(vpixel_t);
    int dstPitch = flag->GetPitch(p) / sizeof(uchar4);

    int width4 = srcvi.width >> 2;
    int height = srcvi.height;
    if (pi > 0) {
      width4 >>= logUVx;
      height >>= logUVy;
    }

    auto prm = (pi == 0) ? prmY : prmC;

    // top
    LaunchAnalyzeFrame(
      flagp, dstPitch * 2,
      f0p,
      f0p - pitch,
      f1p,
      width4, height / 2, pitch * 2,
      prm->threshM, prm->threshS, prm->threshLS, env);

    // bottom
    LaunchAnalyzeFrame(
      flagp + dstPitch, dstPitch * 2,
      f0p + pitch,
      f1p,
      f1p + pitch,
      width4, height / 2, pitch * 2,
      prm->threshM, prm->threshS, prm->threshLS, env);
  }
}

template void KFMFilterBase::AnalyzeFrame<uint8_t>(PVideoFrame& f0, PVideoFrame& f1, PVideoFrame& flag,
  const FrameAnalyzeParam* prmY, const FrameAnalyzeParam* prmC, PNeoEnv env);
template void KFMFilterBase::AnalyzeFrame<uint16_t>(PVideoFrame& f0, PVideoFrame& f1, PVideoFrame& flag,
  const FrameAnalyzeParam* prmY, const FrameAnalyzeParam* prmC, PNeoEnv env);

void KFMFilterBase::MergeUVFlags(PVideoFrame& flag, PNeoEnv env)
{
  uint8_t* fY = reinterpret_cast<uint8_t*>(flag->GetWritePtr(PLANAR_Y));
  uint8_t* fU = reinterpret_cast<uint8_t*>(flag->GetWritePtr(PLANAR_U));
  uint8_t* fV = reinterpret_cast<uint8_t*>(flag->GetWritePtr(PLANAR_V));
  int pitchY = flag->GetPitch(PLANAR_Y) / sizeof(uint8_t);
  int pitchUV = flag->GetPitch(PLANAR_U) / sizeof(uint8_t);

  if (IS_CUDA) {
    dim3 threads(32, 16);
    dim3 blocks(nblocks(srcvi.width, threads.x), nblocks(srcvi.height, threads.y));
    kl_merge_uvflags << <blocks, threads >> >(fY,
      fU, fV, srcvi.width, srcvi.height, pitchY, pitchUV, logUVx, logUVy);
    DEBUG_SYNC;
  }
  else {
    cpu_merge_uvflags(fY,
      fU, fV, srcvi.width, srcvi.height, pitchY, pitchUV, logUVx, logUVy);
  }
}

template <typename pixel_t>
void KFMFilterBase::MergeUVCoefs(PVideoFrame& flag, PNeoEnv env)
{
  pixel_t* fY = reinterpret_cast<pixel_t*>(flag->GetWritePtr(PLANAR_Y));
  pixel_t* fU = reinterpret_cast<pixel_t*>(flag->GetWritePtr(PLANAR_U));
  pixel_t* fV = reinterpret_cast<pixel_t*>(flag->GetWritePtr(PLANAR_V));
  int pitchY = flag->GetPitch(PLANAR_Y) / sizeof(pixel_t);
  int pitchUV = flag->GetPitch(PLANAR_U) / sizeof(pixel_t);

  if (IS_CUDA) {
    dim3 threads(32, 16);
    dim3 blocks(nblocks(vi.width, threads.x), nblocks(vi.height, threads.y));
    kl_merge_uvcoefs << <blocks, threads >> >(fY,
      fU, fV, vi.width, vi.height, pitchY, pitchUV, logUVx, logUVy);
    DEBUG_SYNC;
  }
  else {
    cpu_merge_uvcoefs(fY,
      fU, fV, vi.width, vi.height, pitchY, pitchUV, logUVx, logUVy);
  }
}

template void KFMFilterBase::MergeUVCoefs<uint8_t>(PVideoFrame& flag, PNeoEnv env);
template void KFMFilterBase::MergeUVCoefs<uint16_t>(PVideoFrame& flag, PNeoEnv env);

template <typename pixel_t>
void KFMFilterBase::ApplyUVCoefs(PVideoFrame& flag, PNeoEnv env)
{
  pixel_t* fY = reinterpret_cast<pixel_t*>(flag->GetWritePtr(PLANAR_Y));
  pixel_t* fU = reinterpret_cast<pixel_t*>(flag->GetWritePtr(PLANAR_U));
  pixel_t* fV = reinterpret_cast<pixel_t*>(flag->GetWritePtr(PLANAR_V));
  int pitchY = flag->GetPitch(PLANAR_Y) / sizeof(pixel_t);
  int pitchUV = flag->GetPitch(PLANAR_U) / sizeof(pixel_t);
  int widthUV = vi.width >> logUVx;
  int heightUV = vi.height >> logUVy;

  if (IS_CUDA) {
    dim3 threads(32, 16);
    dim3 blocks(nblocks(widthUV, threads.x), nblocks(heightUV, threads.y));
    kl_apply_uvcoefs_420 << <blocks, threads >> >(fY,
      fU, fV, widthUV, heightUV, pitchY, pitchUV);
    DEBUG_SYNC;
  }
  else {
    cpu_apply_uvcoefs_420(fY, fU, fV, widthUV, heightUV, pitchY, pitchUV);
  }
}

template void KFMFilterBase::ApplyUVCoefs<uint8_t>(PVideoFrame& flag, PNeoEnv env);
template void KFMFilterBase::ApplyUVCoefs<uint16_t>(PVideoFrame& flag, PNeoEnv env);

template <typename pixel_t>
void KFMFilterBase::ExtendCoefs(PVideoFrame& src, PVideoFrame& dst, PNeoEnv env)
{
  typedef typename VectorType<pixel_t>::type vpixel_t;
  const vpixel_t* srcY = reinterpret_cast<const vpixel_t*>(src->GetReadPtr(PLANAR_Y));
  vpixel_t* dstY = reinterpret_cast<vpixel_t*>(dst->GetWritePtr(PLANAR_Y));

  int pitchY = src->GetPitch(PLANAR_Y) / sizeof(vpixel_t);
  int width4 = vi.width >> 2;

  if (IS_CUDA) {
    dim3 threads(32, 16);
    dim3 blocks(nblocks(width4, threads.x), nblocks(vi.height, threads.y));
    kl_extend_coef << <blocks, threads >> >(
      dstY + pitchY, srcY + pitchY, width4, vi.height - 2, pitchY);
    DEBUG_SYNC;
    dim3 threadsB(32, 1);
    dim3 blocksB(nblocks(width4, threads.x));
    kl_copy_border << <blocksB, threadsB >> > (
      dstY, srcY, width4, vi.height, pitchY, 1);
    DEBUG_SYNC;
  }
  else {
    cpu_extend_coef(dstY + pitchY, srcY + pitchY, width4, vi.height - 2, pitchY);
    cpu_copy_border(dstY, srcY, width4, vi.height, pitchY, 1);
  }
}

template void KFMFilterBase::ExtendCoefs<uint8_t>(PVideoFrame& src, PVideoFrame& dst, PNeoEnv env);
template void KFMFilterBase::ExtendCoefs<uint16_t>(PVideoFrame& src, PVideoFrame& dst, PNeoEnv env);

template <typename pixel_t>
void KFMFilterBase::CompareFields(PVideoFrame& src, PVideoFrame& flag, PNeoEnv env)
{
  typedef typename VectorType<pixel_t>::type vpixel_t;
  const vpixel_t* srcY = reinterpret_cast<const vpixel_t*>(src->GetReadPtr(PLANAR_Y));
  const vpixel_t* srcU = reinterpret_cast<const vpixel_t*>(src->GetReadPtr(PLANAR_U));
  const vpixel_t* srcV = reinterpret_cast<const vpixel_t*>(src->GetReadPtr(PLANAR_V));
  vpixel_t* dstY = reinterpret_cast<vpixel_t*>(flag->GetWritePtr(PLANAR_Y));
  vpixel_t* dstU = reinterpret_cast<vpixel_t*>(flag->GetWritePtr(PLANAR_U));
  vpixel_t* dstV = reinterpret_cast<vpixel_t*>(flag->GetWritePtr(PLANAR_V));

  int pitchY = src->GetPitch(PLANAR_Y) / sizeof(vpixel_t);
  int pitchUV = src->GetPitch(PLANAR_U) / sizeof(vpixel_t);
  int width4 = vi.width >> 2;
  int width4UV = width4 >> logUVx;
  int heightUV = vi.height >> logUVy;

  if (IS_CUDA) {
    dim3 threads(32, 16);
    dim3 blocks(nblocks(width4, threads.x), nblocks(vi.height, threads.y));
    dim3 blocksUV(nblocks(width4UV, threads.x), nblocks(heightUV, threads.y));
    kl_calc_combe << <blocks, threads >> >(dstY, srcY, width4, vi.height, pitchY);
    DEBUG_SYNC;
    kl_calc_combe << <blocksUV, threads >> >(dstU, srcU, width4UV, heightUV, pitchUV);
    DEBUG_SYNC;
    kl_calc_combe << <blocksUV, threads >> >(dstV, srcV, width4UV, heightUV, pitchUV);
    DEBUG_SYNC;
  }
  else {
    cpu_calc_combe(dstY, srcY, width4, vi.height, pitchY);
    cpu_calc_combe(dstU, srcU, width4UV, heightUV, pitchUV);
    cpu_calc_combe(dstV, srcV, width4UV, heightUV, pitchUV);
  }
}

template void KFMFilterBase::CompareFields<uint8_t>(PVideoFrame& src, PVideoFrame& flag, PNeoEnv env);
template void KFMFilterBase::CompareFields<uint16_t>(PVideoFrame& src, PVideoFrame& flag, PNeoEnv env);

PVideoFrame KFMFilterBase::OffsetPadFrame(const PVideoFrame& frame, PNeoEnv env)
{
  int vpad = VPAD;
  int vpadUV = VPAD >> logUVy;

  return env->SubframePlanar(frame,
    frame->GetPitch(PLANAR_Y) * vpad, frame->GetPitch(PLANAR_Y), frame->GetRowSize(PLANAR_Y), frame->GetHeight(PLANAR_Y) - vpad * 2,
    frame->GetPitch(PLANAR_U) * vpadUV, frame->GetPitch(PLANAR_U) * vpadUV, frame->GetPitch(PLANAR_U));
}

KFMFilterBase::KFMFilterBase(PClip _child)
  : GenericVideoFilter(_child)
  , srcvi(vi)
  , logUVx(vi.GetPlaneWidthSubsampling(PLANAR_U))
  , logUVy(vi.GetPlaneHeightSubsampling(PLANAR_U))
{ }

int __stdcall KFMFilterBase::SetCacheHints(int cachehints, int frame_range) {
  if (cachehints == CACHE_GET_DEV_TYPE) {
    return GetDeviceTypes(child) &
      (DEV_TYPE_CPU | DEV_TYPE_CUDA);
  }
  return 0;
}
