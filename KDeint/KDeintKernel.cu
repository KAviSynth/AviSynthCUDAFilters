#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS
#include <windows.h>
#include "avisynth.h"

#include <algorithm>
#include <memory>

#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>

#include "CommonFunctions.h"
#include "KDeintKernel.h"

/////////////////////////////////////////////////////////////////////////////
// COPY
/////////////////////////////////////////////////////////////////////////////

template <typename pixel_t>
__global__ void kl_copy(
  pixel_t* dst, int dst_pitch, const pixel_t* src, int src_pitch, int width, int height)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < width && y < height) {
    dst[x + y * dst_pitch] = src[x + y * src_pitch];
  }
}

template <typename pixel_t>
void KDeintKernel::Copy(
  pixel_t* dst, int dst_pitch, const pixel_t* src, int src_pitch, int width, int height)
{
  dim3 threads(32, 16);
  dim3 blocks(nblocks(width, threads.x), nblocks(height, threads.y));
  kl_copy<pixel_t> << <blocks, threads, 0, stream >> > (
    dst, dst_pitch, src, src_pitch, width, height);
  DebugSync();
}

template void KDeintKernel::Copy<uint8_t>(
  uint8_t* dst, int dst_pitch, const uint8_t* src, int src_pitch, int width, int height);
template void KDeintKernel::Copy<uint16_t>(
  uint16_t* dst, int dst_pitch, const uint16_t* src, int src_pitch, int width, int height);
template void KDeintKernel::Copy<int16_t>(
  int16_t* dst, int dst_pitch, const int16_t* src, int src_pitch, int width, int height);
template void KDeintKernel::Copy<int32_t>(
  int32_t* dst, int dst_pitch, const int32_t* src, int src_pitch, int width, int height);


/////////////////////////////////////////////////////////////////////////////
// PadFrame
/////////////////////////////////////////////////////////////////////////////

// width �� Pad ���܂܂Ȃ�����
// block(2, -), threads(hPad, -)
template <typename pixel_t>
__global__ void kl_pad_frame_h(pixel_t* ptr, int pitch, int hPad, int width, int height)
{
  bool isLeft = (blockIdx.x == 0);
  int x = threadIdx.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (y < height) {
    if (isLeft) {
      ptr[x + y * pitch] = ptr[hPad + y * pitch];
    }
    else {
      ptr[(hPad + width + x) + y * pitch] = ptr[(hPad + width) + y * pitch];
    }
  }
}

// height �� Pad ���܂܂Ȃ�����
// block(-, 2), threads(-, vPad)
template <typename pixel_t>
__global__ void kl_pad_frame_v(pixel_t* ptr, int pitch, int vPad, int width, int height)
{
  bool isTop = (blockIdx.y == 0);
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y;

  if (x < width) {
    if (isTop) {
      ptr[x + y * pitch] = ptr[x + vPad * pitch];
    }
    else {
      ptr[x + (vPad + height + y) * pitch] = ptr[x + (vPad + height) * pitch];
    }
  }
}

template<typename pixel_t>
void KDeintKernel::PadFrame(pixel_t *ptr, int pitch, int hPad, int vPad, int width, int height)
{
  { // H����
    dim3 threads(hPad, 32);
    dim3 blocks(2, nblocks(height, threads.y));
    kl_pad_frame_h<pixel_t> << <blocks, threads, 0, stream >> > (
      ptr + vPad * pitch, pitch, hPad, width, height);
    DebugSync();
  }
  { // V�����i���ł�Pad���ꂽH���������܂ށj
    dim3 threads(32, vPad);
    dim3 blocks(nblocks(width + hPad * 2, threads.x), 2);
    kl_pad_frame_v<pixel_t> << <blocks, threads, 0, stream >> > (
      ptr, pitch, vPad, width + hPad * 2, height);
    DebugSync();
  }
}

template void KDeintKernel::PadFrame<uint8_t>(
  uint8_t *ptr, int pitch, int hPad, int vPad, int width, int height);
template void KDeintKernel::PadFrame<uint16_t>(
  uint16_t *ptr, int pitch, int hPad, int vPad, int width, int height);


/////////////////////////////////////////////////////////////////////////////
// Wiener
/////////////////////////////////////////////////////////////////////////////

// so called Wiener interpolation. (sharp, similar to Lanczos ?)
// invarint simplified, 6 taps. Weights: (1, -5, 20, 20, -5, 1)/32 - added by Fizick
template<typename pixel_t>
__global__ void kl_vertical_wiener(pixel_t *pDst, const pixel_t *pSrc, int nDstPitch,
  int nSrcPitch, int nWidth, int nHeight, int max_pixel_value)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < nWidth) {
    if (y < 2) {
      pDst[x + y * nDstPitch] = (pSrc[x + y * nSrcPitch] + pSrc[x + (y + 1) * nSrcPitch] + 1) >> 1;
    }
    else if (y < nHeight - 4) {
      pDst[x + y * nDstPitch] = min(max_pixel_value, max(0,
        (pSrc[x + (y - 2) * nSrcPitch]
          + (-(pSrc[x + (y - 1) * nSrcPitch]) + (pSrc[x + y * nSrcPitch] << 2) +
          (pSrc[x + (y + 1) * nSrcPitch] << 2) - (pSrc[x + (y + 2) * nSrcPitch])) * 5
          + (pSrc[x + (y + 3) * nSrcPitch]) + 16) >> 5));
    }
    else if (y < nHeight - 1) {
      pDst[x + y * nDstPitch] = (pSrc[x + y * nSrcPitch] + pSrc[x + (y + 1) * nSrcPitch] + 1) >> 1;
    }
    else if (y < nHeight) {
      // last row
      pDst[x + y * nDstPitch] = pSrc[x + y * nSrcPitch];
    }
  }
}

template<typename pixel_t>
void KDeintKernel::VerticalWiener(
  pixel_t *pDst, const pixel_t *pSrc, int nDstPitch,
  int nSrcPitch, int nWidth, int nHeight, int bits_per_pixel)
{
  const int max_pixel_value = sizeof(pixel_t) == 1 ? 255 : (1 << bits_per_pixel) - 1;

  dim3 threads(32, 16);
  dim3 blocks(nblocks(nWidth, threads.x), nblocks(nHeight, threads.y));
  kl_vertical_wiener<pixel_t> << <blocks, threads, 0, stream >> > (
    pDst, pSrc, nDstPitch, nSrcPitch, nWidth, nHeight, max_pixel_value);
  DebugSync();
}

template<typename pixel_t>
__global__ void kl_horizontal_wiener(pixel_t *pDst, const pixel_t *pSrc, int nDstPitch,
  int nSrcPitch, int nWidth, int nHeight, int max_pixel_value)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (y < nHeight) {
    if (x < 2) {
      pDst[x + y * nDstPitch] = (pSrc[x + y * nSrcPitch] + pSrc[(x + 1) + y * nSrcPitch] + 1) >> 1;
    }
    else if (x < nWidth - 4) {
      pDst[x + y * nDstPitch] = min(max_pixel_value, max(0,
        (pSrc[(x - 2) + y * nSrcPitch]
          + (-(pSrc[(x - 1) + y * nSrcPitch]) + (pSrc[x + y * nSrcPitch] << 2) +
          (pSrc[(x + 1) + y * nSrcPitch] << 2) - (pSrc[(x + 2) + y * nSrcPitch])) * 5
          + (pSrc[(x + 3) + y * nSrcPitch]) + 16) >> 5));
    }
    else if (x < nWidth - 1) {
      pDst[x + y * nDstPitch] = (pSrc[x + y * nSrcPitch] + pSrc[(x + 1) + y * nSrcPitch] + 1) >> 1;
    }
    else if (x < nWidth) {
      // last column
      pDst[x + y * nDstPitch] = pSrc[x + y * nSrcPitch];
    }
  }
}

template<typename pixel_t>
void KDeintKernel::HorizontalWiener(
  pixel_t *pDst, const pixel_t *pSrc, int nDstPitch,
  int nSrcPitch, int nWidth, int nHeight, int bits_per_pixel)
{
  const int max_pixel_value = sizeof(pixel_t) == 1 ? 255 : (1 << bits_per_pixel) - 1;

  dim3 threads(32, 16);
  dim3 blocks(nblocks(nWidth, threads.x), nblocks(nHeight, threads.y));
  kl_horizontal_wiener<pixel_t> << <blocks, threads, 0, stream >> > (
    pDst, pSrc, nDstPitch, nSrcPitch, nWidth, nHeight, max_pixel_value);
  DebugSync();
}


template void KDeintKernel::VerticalWiener<uint8_t>(
  uint8_t *pDst, const uint8_t *pSrc, int nDstPitch,
  int nSrcPitch, int nWidth, int nHeight, int bits_per_pixel);
template void KDeintKernel::VerticalWiener<uint16_t>(
  uint16_t *pDst, const uint16_t *pSrc, int nDstPitch,
  int nSrcPitch, int nWidth, int nHeight, int bits_per_pixel);
template void KDeintKernel::HorizontalWiener<uint8_t>(
  uint8_t *pDst, const uint8_t *pSrc, int nDstPitch,
  int nSrcPitch, int nWidth, int nHeight, int bits_per_pixel);
template void KDeintKernel::HorizontalWiener<uint16_t>(
  uint16_t *pDst, const uint16_t *pSrc, int nDstPitch,
  int nSrcPitch, int nWidth, int nHeight, int bits_per_pixel);


/////////////////////////////////////////////////////////////////////////////
// RB2BilinearFilter
/////////////////////////////////////////////////////////////////////////////

enum {
  RB2B_BILINEAR_W = 32,
  RB2B_BILINEAR_H = 16,
};

// BilinearFiltered with 1/8, 3/8, 3/8, 1/8 filter for smoothing and anti-aliasing - Fizick
// threads=(RB2B_BILINEAR_W,RB2B_BILINEAR_H)
// nblocks=(nblocks(nWidth*2, RB2B_BILINEAR_W - 2),nblocks(nHeight,RB2B_BILINEAR_H))
template<typename pixel_t>
__global__ void kl_RB2B_bilinear_filtered(
  pixel_t *pDst, const pixel_t *pSrc, int nDstPitch, int nSrcPitch, int nWidth, int nHeight)
{
  __shared__ pixel_t tmp[RB2B_BILINEAR_H][RB2B_BILINEAR_W];

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Vertical�����s
  // Horizontal�ŎQ�Ƃ��邽�ߗ��[1�񂸂]���Ɏ��s
  int x = tx - 1 + blockIdx.x * (RB2B_BILINEAR_W - 2);
  int y = ty + blockIdx.y * RB2B_BILINEAR_H;
  int y2 = y * 2;

  if (x >= 0 && x < nWidth * 2) {
    if (y < 1) {
      tmp[ty][tx] = (pSrc[x + y2 * nSrcPitch] + pSrc[x + (y2 + 1) * nSrcPitch] + 1) / 2;
    }
    else if (y < nHeight - 1) {
      tmp[ty][tx] = (pSrc[x + (y2 - 1) * nSrcPitch]
        + pSrc[x + y2 * nSrcPitch] * 3
        + pSrc[x + (y2 + 1) * nSrcPitch] * 3
        + pSrc[x + (y2 + 2) * nSrcPitch] + 4) / 8;
    }
    else if (y < nHeight) {
      tmp[ty][tx] = (pSrc[x + y2 * nSrcPitch] + pSrc[x + (y2 + 1) * nSrcPitch] + 1) / 2;
    }
  }

  __syncthreads();

  // Horizontal�����s
  x = tx + blockIdx.x * ((RB2B_BILINEAR_W - 2) / 2);
  int tx2 = tx * 2;

  if (tx < ((RB2B_BILINEAR_W - 2) / 2) && y < nHeight) {
    // tmp��[0][1]�����_�ł��邱�Ƃɒ���
    if (x < 1) {
      pDst[x + y * nDstPitch] = (tmp[ty][tx2 + 1] + tmp[ty][tx2 + 2] + 1) / 2;
    }
    else if (x < nWidth - 1) {
      pDst[x + y * nDstPitch] = (tmp[ty][tx2]
        + tmp[ty][tx2 + 1] * 3
        + tmp[ty][tx2 + 2] * 3
        + tmp[ty][tx2 + 3] + 4) / 8;
    }
    else if (x < nWidth) {
      pDst[x + y * nDstPitch] = (tmp[ty][tx2 + 1] + tmp[ty][tx2 + 2] + 1) / 2;
    }
  }
}

template<typename pixel_t>
void KDeintKernel::RB2BilinearFiltered(
  pixel_t *pDst, const pixel_t *pSrc, int nDstPitch, int nSrcPitch, int nWidth, int nHeight)
{
  dim3 threads(RB2B_BILINEAR_W, RB2B_BILINEAR_H);
  dim3 blocks(nblocks(nWidth*2, RB2B_BILINEAR_W - 2), nblocks(nHeight, RB2B_BILINEAR_H));
  kl_RB2B_bilinear_filtered<pixel_t> << <blocks, threads, 0, stream >> > (
    pDst, pSrc, nDstPitch, nSrcPitch, nWidth, nHeight);
  DebugSync();
}

template void KDeintKernel::RB2BilinearFiltered<uint8_t>(
  uint8_t *pDst, const uint8_t *pSrc, int nDstPitch, int nSrcPitch, int nWidth, int nHeight);
template void KDeintKernel::RB2BilinearFiltered<uint16_t>(
  uint16_t *pDst, const uint16_t *pSrc, int nDstPitch, int nSrcPitch, int nWidth, int nHeight);



/////////////////////////////////////////////////////////////////////////////
// SearchMV
/////////////////////////////////////////////////////////////////////////////


typedef int sad_t; // ���float�ɂ���

enum {
  SRCH_DIMX = 128
};

struct SearchBlock {
  // [0-3]: nDxMax, nDyMax, nDxMin, nDyMin �iMax��Max-1�ɂ��Ă����j
  // [4-9]: Left predictor, Up predictor, bottom-right predictor(from coarse level)
  // �����ȂƂ���͍��Ȃ��悤�ɂ���i�Œ�ł��ǂꂩ�P�͗L���Ȃ̂Ŗ����Ȃ�Ƃ���͂��̃C���f�b�N�X�Ŗ��߂�j
  // [10-11]: predictor �� x, y
  int data[12];
  // [0-3]: penaltyZero, penaltyGlobal, 1(penaltyPredictor), penaltyNew
  // [4]: lambda
  sad_t dataf[5];
};

#define CLIP_RECT data
#define REF_VECTOR_INDEX (&data[4])
#define PRED_X data[10]
#define PRED_Y data[11]
#define PENALTIES dataf
#define PENALTY_NEW dataf[3]
#define LAMBDA dataf[4]

#define LARGE_COST INT_MAX

struct CostResult {
  sad_t cost;
  short2 xy;
};

__device__ void dev_clip_mv(short2& v, const int* rect)
{
  v.x = (v.x > rect[0]) ? rect[0] : (v.x < rect[2]) ? rect[2] : v.x;
  v.y = (v.y > rect[1]) ? rect[1] : (v.y < rect[3]) ? rect[3] : v.y;
}

__device__ bool dev_check_mv(int x, int y, const int* rect)
{
  return (x <= rect[0]) & (y <= rect[1]) & (x >= rect[2]) & (y >= rect[3]);
}

__device__ int dev_max(int a, int b, int c) {
  int ab = (a > b) ? a : b;
  return (ab > c) ? ab : c;
}

__device__ int dev_min(int a, int b, int c) {
  int ab = (a < b) ? a : b;
  return (ab < c) ? ab : c;
}

__device__ int dev_sq_norm(int ax, int ay, int bx, int by) {
  return (ax - bx) * (ax - bx) + (ay - by) * (ay - by);
}

// pRef �� �u���b�N�I�t�Z�b�g����\�߈ړ������Ă������|�C���^
// vx,vy �� �T�u�s�N�Z�����܂߂��x�N�g��
template <typename pixel_t, int NPEL>
__device__ const pixel_t* dev_get_ref_block(const pixel_t* pRef, int nPitch, int nImgPitch, int vx, int vy)
{
  if (NPEL != 1) {
    int sx = vx % NPEL;
    int sy = vy % NPEL;
    int si = sx + sy * NPEL;
    int x = vx / NPEL;
    int y = vy / NPEL;
    return &pRef[x + y * nPitch + si * nImgPitch];
  }
  else {
    return &pRef[vx + vy * nPitch];
  }
}

__device__ int dev_reduce_sad(int sad, int tid)
{
  // warp shuffle��reduce
  sad += __shfl_down(sad, 8);
  sad += __shfl_down(sad, 4);
  sad += __shfl_down(sad, 2);
  sad += __shfl_down(sad, 1);
  return sad;
}

template <typename pixel_t, int BLK_SIZE>
__device__ sad_t dev_calc_sad(
  int wi,
  const pixel_t* pSrcY, const pixel_t* pSrcU, const pixel_t* pSrcV,
  const pixel_t* pRefY, const pixel_t* pRefU, const pixel_t* pRefV,
  int nPitchY, int nPitchU, int nPitchV)
{
  int sad = 0;
  if (BLK_SIZE == 16) {
    // �u���b�N�T�C�Y���X���b�h���ƈ�v
    int yx = wi;
    for (int yy = 0; yy < BLK_SIZE; ++yy) { // 16�񃋁[�v
      sad = __sad(pSrcY[yx + yy * BLK_SIZE], pRefY[yx + yy * nPitchY], sad);
    }
    // UV��8x8
    int uvx = wi % 8;
    int uvy = wi / 8;
    for (int t = 0; t < 4; ++t, uvy += 2) { // 4�񃋁[�v
      sad = __sad(pSrcU[uvx + uvy * BLK_SIZE], pRefU[uvx + uvy * nPitchU], sad);
      sad = __sad(pSrcV[uvx + uvy * BLK_SIZE], pRefV[uvx + uvy * nPitchV], sad);
    }
  }
  else if (BLK_SIZE == 32) {
    // 32x32
    int yx = wi;
    for (int yy = 0; yy < BLK_SIZE; ++yy) { // 32�񃋁[�v
      sad = __sad(pSrcY[yx + yy * BLK_SIZE], pRefY[yx + yy * nPitchY], sad);
      sad = __sad(pSrcY[yx + 16 + yy * BLK_SIZE], pRefY[yx + 16 + yy * nPitchY], sad);
    }
    // �u���b�N�T�C�Y���X���b�h���ƈ�v
    int uvx = wi;
    for (int uvy = 0; uvy < BLK_SIZE; ++uvy) { // 16�񃋁[�v
      sad = __sad(pSrcU[uvx + uvy * BLK_SIZE], pRefU[uvx + uvy * nPitchU], sad);
      sad = __sad(pSrcV[uvx + uvy * BLK_SIZE], pRefV[uvx + uvy * nPitchV], sad);
    }
  }
  return dev_reduce_sad(sad, wi);
}

// MAX - (MAX/4) <= (���ʂ̌�) <= MAX �ł��邱��
// �X���b�h���� (���ʂ̌�) - MAX/2
template <int MAX>
__device__ void dev_reduce_result(CostResult* tmp_, int tid)
{
  volatile CostResult* tmp = (volatile CostResult*)tmp_;
  if(MAX >= 16) tmp[tid] = (tmp[tid].cost < tmp[tid + 8].cost) ? tmp[tid] : tmp[tid + 8];
  tmp[tid] = (tmp[tid].cost < tmp[tid + 4].cost) ? tmp[tid] : tmp[tid + 4];
  tmp[tid] = (tmp[tid].cost < tmp[tid + 2].cost) ? tmp[tid] : tmp[tid + 2];
  tmp[tid] = (tmp[tid].cost < tmp[tid + 1].cost) ? tmp[tid] : tmp[tid + 1];
}

// __syncthreads()���Ăяo���Ă���̂őS���ŌĂ�
template <typename pixel_t, int BLK_SIZE, int NPEL>
__device__ void dev_expanding_search_1(
  int tx, int wi, int bx, int cx, int cy,
  const int* data, const sad_t* dataf,
  CostResult& bestResult,
  const pixel_t* pSrcY, const pixel_t* pSrcU, const pixel_t* pSrcV,
  const pixel_t* __restrict__ pRefBY, const pixel_t* __restrict__ pRefBU, const pixel_t* __restrict__ pRefBV,
  int nPitchY, int nPitchU, int nPitchV,
  int nImgPitchY, int nImgPitchU, int nImgPitchV)
{
  int2 area[] = {
    { -1, -1 },
    { 0, -1 },
    { 1, -1 },
    { -1, 0 },
    { 1, 0 },
    { -1, 1 },
    { 0, 1 },
    { 1, 1 }
  };

  __shared__ bool isVectorOK[8];
  __shared__ CostResult result[8];
  __shared__ const pixel_t* pRefY[8];
  __shared__ const pixel_t* pRefU[8];
  __shared__ const pixel_t* pRefV[8];

  if (tx < 8) {
    int x = result[tx].xy.x = cx + area[tx].x;
    int y = result[tx].xy.y = cy + area[tx].y;
    bool ok = dev_check_mv(x, y, CLIP_RECT);
    int cost = (LAMBDA * dev_sq_norm(x, y, PRED_X, PRED_Y)) >> 8;

    // no additional SAD calculations if partial sum is already above minCost
    if (cost >= bestResult.cost) {
      ok = false;
    }

    isVectorOK[tx] = ok;
    result[tx].cost = ok ? cost : LARGE_COST;

    pRefY[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBY, nPitchY, nImgPitchY, x, y);
    pRefU[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBU, nPitchU, nImgPitchU, x / 2, y / 2);
    pRefV[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBV, nPitchV, nImgPitchV, x / 2, y / 2);
  }

  __syncthreads();

  if (isVectorOK[bx]) {
    sad_t sad = dev_calc_sad<pixel_t, BLK_SIZE>(wi, pSrcY, pSrcU, pSrcV, pRefY[bx], pRefU[bx], pRefV[bx], nPitchY, nPitchU, nPitchV);
    if (wi == 0) {
      result[bx].cost += (sad * PENALTY_NEW) >> 8;
    }
  }

  __syncthreads();

  // ���ʏW��
  if (tx < 4) { // reduce��8-4=4�X���b�h�ŌĂ�
    dev_reduce_result<8>(result, tx);

    if (tx == 0) { // tx == 0�͍Ō�̃f�[�^����������ł���̂ŃA�N�Z�XOK
      if (result[0].cost < bestResult.cost) {
        bestResult = result[0];
      }
    }
  }
}

// __syncthreads()���Ăяo���Ă���̂őS���ŌĂ�
template <typename pixel_t, int BLK_SIZE, int NPEL>
__device__ void dev_expanding_search_2(
  int tx, int wi, int bx, int cx, int cy,
  const int* data, const sad_t* dataf,
  CostResult& bestResult,
  const pixel_t* pSrcY, const pixel_t* pSrcU, const pixel_t* pSrcV,
  const pixel_t* __restrict__ pRefBY, const pixel_t* __restrict__ pRefBU, const pixel_t* __restrict__ pRefBV,
  int nPitchY, int nPitchU, int nPitchV,
  int nImgPitchY, int nImgPitchU, int nImgPitchV)
{
  int2 area[] = {
    { -2, -2 },
    { -1, -2 },
    { 0, -2 },
    { 1, -2 },
    { 2, -2 },

    { -2, -1 },
    { 2, -1 },
    { -2, 0 },
    { 2, 0 },
    { -2, 1 },
    { 2, 1 },

    { -2, 2 },
    { -1, 2 },
    { 0, 2 },
    { 1, 2 },
    { 2, 2 }
  };

  __shared__ bool isVectorOK[16];
  __shared__ CostResult result[16];
  __shared__ const pixel_t* pRefY[16];
  __shared__ const pixel_t* pRefU[16];
  __shared__ const pixel_t* pRefV[16];

  if (tx < 16) {
    int x = result[tx].xy.x = cx + area[tx].x;
    int y = result[tx].xy.y = cy + area[tx].y;
    bool ok = dev_check_mv(x, y, CLIP_RECT);
    int cost = (LAMBDA * dev_sq_norm(x, y, PRED_X, PRED_Y)) >> 8;

    // no additional SAD calculations if partial sum is already above minCost
    if (cost >= bestResult.cost) {
      ok = false;
    }

    isVectorOK[tx] = ok;
    result[tx].cost = ok ? cost : LARGE_COST;

    pRefY[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBY, nPitchY, nImgPitchY, x, y);
    pRefU[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBU, nPitchU, nImgPitchU, x / 2, y / 2);
    pRefV[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBV, nPitchV, nImgPitchV, x / 2, y / 2);
  }

  __syncthreads();

  if (isVectorOK[bx]) {
    sad_t sad = dev_calc_sad<pixel_t, BLK_SIZE>(wi, pSrcY, pSrcU, pSrcV, pRefY[bx], pRefU[bx], pRefV[bx], nPitchY, nPitchU, nPitchV);
    if (wi == 0) {
      result[bx].cost += (sad * PENALTY_NEW) >> 8;
    }
  }
  int bx2 = bx + 8;
  if (isVectorOK[bx2]) {
    sad_t sad = dev_calc_sad<pixel_t, BLK_SIZE>(wi, pSrcY, pSrcU, pSrcV, pRefY[bx2], pRefU[bx2], pRefV[bx2], nPitchY, nPitchU, nPitchV);
    if (wi == 0) {
      result[bx2].cost += (sad * PENALTY_NEW) >> 8;
    }
  }

  __syncthreads();

  // ���ʏW��
  if (tx < 8) { // reduce��16-8=8�X���b�h�ŌĂ�
    dev_reduce_result<16>(result, tx);

    if (tx == 0) { // tx == 0�͍Ō�̃f�[�^����������ł���̂ŃA�N�Z�XOK
      if (result[0].cost < bestResult.cost) {
        bestResult = result[0];
      }
    }
  }
}

// __syncthreads()���Ăяo���Ă���̂őS���ŌĂ�
template <typename pixel_t, int BLK_SIZE, int NPEL>
__device__ void dev_hex2_search_1(
  int tx, int wi, int bx, int cx, int cy,
  const int* data, const sad_t* dataf,
  CostResult& bestResult,
  const pixel_t* pSrcY, const pixel_t* pSrcU, const pixel_t* pSrcV,
  const pixel_t* __restrict__ pRefBY, const pixel_t* __restrict__ pRefBU, const pixel_t* __restrict__ pRefBV,
  int nPitchY, int nPitchU, int nPitchV,
  int nImgPitchY, int nImgPitchU, int nImgPitchV)
{
  int2 area[] = { { -1,-2 },{ -2,0 },{ -1,2 },{ 1,2 },{ 2,0 },{ 1,-2 },{ -1,-2 },{ -2,0 } };

  __shared__ bool isVectorOK[8];
  __shared__ CostResult result[8];
  __shared__ const pixel_t* pRefY[8];
  __shared__ const pixel_t* pRefU[8];
  __shared__ const pixel_t* pRefV[8];

  isVectorOK[tx] = false;

  if (tx < 6) {
    int x = result[tx].xy.x = cx + area[tx].x;
    int y = result[tx].xy.y = cy + area[tx].y;
    bool ok = dev_check_mv(x, y, CLIP_RECT);
    int cost = (LAMBDA * dev_sq_norm(x, y, PRED_X, PRED_Y)) >> 8;

    // no additional SAD calculations if partial sum is already above minCost
    if (cost >= bestResult.cost) {
      ok = false;
    }

    isVectorOK[tx] = ok;
    result[tx].cost = ok ? cost : LARGE_COST;

    pRefY[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBY, nPitchY, nImgPitchY, x, y);
    pRefU[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBU, nPitchU, nImgPitchU, x / 2, y / 2);
    pRefV[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBV, nPitchV, nImgPitchV, x / 2, y / 2);
  }

  __syncthreads();

  if (isVectorOK[bx]) {
    sad_t sad = dev_calc_sad<pixel_t, BLK_SIZE>(wi, pSrcY, pSrcU, pSrcV, pRefY[bx], pRefU[bx], pRefV[bx], nPitchY, nPitchU, nPitchV);
    if (wi == 0) {
      result[bx].cost += (sad * PENALTY_NEW) >> 8;
    }
  }

  __syncthreads();

  // ���ʏW��
  if (tx < 2) { // reduce��6-4=2�X���b�h�ŌĂ�
    dev_reduce_result<8>(result, tx);

    if (tx == 0) { // tx == 0�͍Ō�̃f�[�^����������ł���̂ŃA�N�Z�XOK
      if (result[0].cost < bestResult.cost) {
        bestResult = result[0];
      }
    }
  }
}

// SRCH_DIMX % BLK_SIZE == 0������
template <typename pixel_t, int BLK_SIZE>
__device__ void dev_read_pixels(int tx, const pixel_t* src, int nPitch, int offx, int offy, pixel_t *dst)
{
  int y = tx / BLK_SIZE;
  int x = tx % BLK_SIZE;
  for (; y < BLK_SIZE; y += SRCH_DIMX / BLK_SIZE) {
    dst[x + y * BLK_SIZE] = src[(x + offx) + (y + offy) * nPitch];
  }
}

template <typename pixel_t, int BLK_SIZE, int SEARCH, int NPEL>
__global__ void kl_search(
  int nBlkX, int nBlkY, const SearchBlock* __restrict__ blocks,
  short2* vectors, // [x,y]
  int nHPad, int nVPad,
  const pixel_t* __restrict__ pSrcY, const pixel_t* __restrict__ pSrcU, const pixel_t* __restrict__ pSrcV,
  const pixel_t* __restrict__ pRefY, const pixel_t* __restrict__ pRefU, const pixel_t* __restrict__ pRefV,
  int nPitchY, int nPitchU, int nPitchV,
  int nImgPitchY, int nImgPitchU, int nImgPitchV
)
{
  enum {
    BLK_SIZE_UV = BLK_SIZE / 2,
    BLK_STEP = BLK_SIZE / 2,
  };

  const int tx = threadIdx.x;
  const int wi = tx % 16;
  const int bx = tx / 16;

  for (int blkx = blockIdx.x; blkx < nBlkX; blkx += blockDim.x) {
    for (int blky = 0; blky < nBlkY; ++blky) {

      // src��shared memory�ɓ]��
      int offx = nHPad + blkx * BLK_STEP;
      int offy = nVPad + blky * BLK_STEP;

      __shared__ pixel_t srcY[BLK_SIZE * BLK_SIZE];
      __shared__ pixel_t srcU[BLK_SIZE_UV * BLK_SIZE_UV];
      __shared__ pixel_t srcV[BLK_SIZE_UV * BLK_SIZE_UV];

      dev_read_pixels<pixel_t, BLK_SIZE>(tx, pSrcY, nPitchY, offx, offy, srcY);
      dev_read_pixels<pixel_t, BLK_SIZE_UV>(tx, pSrcU, nPitchU, offx / 2, offy / 2, srcU);
      dev_read_pixels<pixel_t, BLK_SIZE_UV>(tx, pSrcV, nPitchV, offx / 2, offy / 2, srcV);

      __shared__ const pixel_t* pRefBY;
      __shared__ const pixel_t* pRefBU;
      __shared__ const pixel_t* pRefBV;

      if (tx == 0) {
        pRefBY = &pRefY[offx + offy * nPitchY];
        pRefBU = &pRefU[offx / 2 + offy / 2 * nPitchU];
        pRefBV = &pRefV[offx / 2 + offy / 2 * nPitchV];
      }

      // �p�����[�^�Ȃǂ̃f�[�^��shared memory�Ɋi�[
      __shared__ int data[12];
      __shared__ sad_t dataf[5];

      if (tx < 12) {
        int blkIdx = blky*nBlkX + blkx;
        data[tx] = blocks[blkIdx].data[tx];
        if (tx < 5) {
          dataf[tx] = blocks[blkIdx].dataf[tx];
        }
      }

      __syncthreads();

      // FetchPredictors
      __shared__ CostResult result[8];
      __shared__ const pixel_t* pRefY[8];
      __shared__ const pixel_t* pRefU[8];
      __shared__ const pixel_t* pRefV[8];

      if (tx < 6) {
        __shared__ volatile short pred[7][2]; // x, y

        // zero, global, predictor, predictors[1]�`[3]���擾
        short2 vec = vectors[REF_VECTOR_INDEX[tx]];
        dev_clip_mv(vec, CLIP_RECT);
        pred[tx][0] = vec.x;
        pred[tx][1] = vec.y;
        // memfence
        if (tx < 2) {
          // Median predictor
          // �v�Z�����������̂ŏ��������E�E�E
          int a = pred[3][tx];
          int b = pred[4][tx];
          int c = pred[5][tx];
          int max_ = dev_max(a, b, c);
          int min_ = dev_min(a, b, c);
          int med_ = a + b + c - max_ - min_;
          pred[6][tx] = med_;
        }
        // memfence
        int x = result[tx].xy.x = pred[tx][0];
        int y = result[tx].xy.y = pred[tx][1];
        result[tx].cost = (LAMBDA * dev_sq_norm(x, y, PRED_X, PRED_Y)) >> 8;

        pRefY[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBY, nPitchY, nImgPitchY, x, y);
        pRefU[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBU, nPitchU, nImgPitchU, x / 2, y / 2);
        pRefV[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBV, nPitchV, nImgPitchV, x / 2, y / 2);
      }

      __syncthreads();

      // �܂���7�ӏ����v�Z
      if (bx < 7) {
        sad_t sad = dev_calc_sad<pixel_t, BLK_SIZE>(wi, srcY, srcU, srcV, pRefY[bx], pRefU[bx], pRefV[bx], nPitchY, nPitchU, nPitchV);
        if (wi == 0) {
          if (bx < 3) {
            // pzero, pglobal, 1
            result[bx].cost = (sad * PENALTIES[bx]) >> 8;
          }
          else {
            result[bx].cost += sad;
          }
        }
        // �Ƃ肠������r��cost�����ł��̂�SAD�͗v��Ȃ�
        // SAD�͒T�����I�������Čv�Z����
      }

      __syncthreads();

      // ���ʏW��
      if (tx < 3) { // 7-4=3�X���b�h�ŌĂ�
        dev_reduce_result<8>(result, tx);
      }

      __syncthreads();

      // Refine
      if (SEARCH == 1) {
        // EXHAUSTIVE
        int bmx = result[0].xy.x;
        int bmy = result[0].xy.y;
        dev_expanding_search_1<pixel_t, BLK_SIZE, NPEL>(
          tx, wi, bx, bmx, bmy, data, dataf, result[0],
          srcY, srcU, srcV, pRefBY, pRefBU, pRefBV,
          nPitchY, nPitchU, nPitchV, nImgPitchY, nImgPitchU, nImgPitchV);
        dev_expanding_search_2<pixel_t, BLK_SIZE, NPEL>(
          tx, wi, bx, bmx, bmy, data, dataf, result[0],
          srcY, srcU, srcV, pRefBY, pRefBU, pRefBV,
          nPitchY, nPitchU, nPitchV, nImgPitchY, nImgPitchU, nImgPitchV);
      }
      else if (SEARCH == 2) {
        // HEX2SEARCH
        dev_hex2_search_1<pixel_t, BLK_SIZE, NPEL>(
          tx, wi, bx, result[0].xy.x, result[0].xy.y, data, dataf, result[0],
          srcY, srcU, srcV, pRefBY, pRefBU, pRefBV,
          nPitchY, nPitchU, nPitchV, nImgPitchY, nImgPitchU, nImgPitchV);
        dev_expanding_search_1<pixel_t, BLK_SIZE, NPEL>(
          tx, wi, bx, result[0].xy.x, result[0].xy.y, data, dataf, result[0],
          srcY, srcU, srcV, pRefBY, pRefBU, pRefBV,
          nPitchY, nPitchU, nPitchV, nImgPitchY, nImgPitchU, nImgPitchV);
      }

      // ���ʏ�������
      if (tx == 0) {
        vectors[blky*nBlkX + blkx] = result[0].xy;
      }

      // ���L�������ی�
      __syncthreads();
    }
  }
}



