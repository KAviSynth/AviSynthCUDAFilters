#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS
#include <windows.h>
#include "avisynth.h"

#include <algorithm>
#include <memory>

#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>

#include "CommonFunctions.h"
#include "KDeintKernel.h"

#include "ReduceKernel.cuh"

/////////////////////////////////////////////////////////////////////////////
// COPY
/////////////////////////////////////////////////////////////////////////////

template <typename pixel_t>
__global__ void kl_copy(
  pixel_t* dst, int dst_pitch, const pixel_t* src, int src_pitch, int width, int height)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < width && y < height) {
    dst[x + y * dst_pitch] = src[x + y * src_pitch];
  }
}

template <typename pixel_t>
void KDeintKernel::Copy(
  pixel_t* dst, int dst_pitch, const pixel_t* src, int src_pitch, int width, int height)
{
  dim3 threads(32, 16);
  dim3 blocks(nblocks(width, threads.x), nblocks(height, threads.y));
  kl_copy<pixel_t> << <blocks, threads, 0, stream >> > (
    dst, dst_pitch, src, src_pitch, width, height);
  DebugSync();
}

template void KDeintKernel::Copy<uint8_t>(
  uint8_t* dst, int dst_pitch, const uint8_t* src, int src_pitch, int width, int height);
template void KDeintKernel::Copy<uint16_t>(
  uint16_t* dst, int dst_pitch, const uint16_t* src, int src_pitch, int width, int height);
template void KDeintKernel::Copy<int16_t>(
  int16_t* dst, int dst_pitch, const int16_t* src, int src_pitch, int width, int height);
template void KDeintKernel::Copy<int32_t>(
  int32_t* dst, int dst_pitch, const int32_t* src, int src_pitch, int width, int height);


/////////////////////////////////////////////////////////////////////////////
// PadFrame
/////////////////////////////////////////////////////////////////////////////

// width �� Pad ���܂܂Ȃ�����
// block(2, -), threads(hPad, -)
template <typename pixel_t>
__global__ void kl_pad_frame_h(pixel_t* ptr, int pitch, int hPad, int width, int height)
{
  bool isLeft = (blockIdx.x == 0);
  int x = threadIdx.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (y < height) {
    if (isLeft) {
      ptr[x + y * pitch] = ptr[hPad + y * pitch];
    }
    else {
      ptr[(hPad + width + x) + y * pitch] = ptr[(hPad + width) + y * pitch];
    }
  }
}

// height �� Pad ���܂܂Ȃ�����
// block(-, 2), threads(-, vPad)
template <typename pixel_t>
__global__ void kl_pad_frame_v(pixel_t* ptr, int pitch, int vPad, int width, int height)
{
  bool isTop = (blockIdx.y == 0);
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y;

  if (x < width) {
    if (isTop) {
      ptr[x + y * pitch] = ptr[x + vPad * pitch];
    }
    else {
      ptr[x + (vPad + height + y) * pitch] = ptr[x + (vPad + height) * pitch];
    }
  }
}

template<typename pixel_t>
void KDeintKernel::PadFrame(pixel_t *ptr, int pitch, int hPad, int vPad, int width, int height)
{
  { // H����
    dim3 threads(hPad, 32);
    dim3 blocks(2, nblocks(height, threads.y));
    kl_pad_frame_h<pixel_t> << <blocks, threads, 0, stream >> > (
      ptr + vPad * pitch, pitch, hPad, width, height);
    DebugSync();
  }
  { // V�����i���ł�Pad���ꂽH���������܂ށj
    dim3 threads(32, vPad);
    dim3 blocks(nblocks(width + hPad * 2, threads.x), 2);
    kl_pad_frame_v<pixel_t> << <blocks, threads, 0, stream >> > (
      ptr, pitch, vPad, width + hPad * 2, height);
    DebugSync();
  }
}

template void KDeintKernel::PadFrame<uint8_t>(
  uint8_t *ptr, int pitch, int hPad, int vPad, int width, int height);
template void KDeintKernel::PadFrame<uint16_t>(
  uint16_t *ptr, int pitch, int hPad, int vPad, int width, int height);


/////////////////////////////////////////////////////////////////////////////
// Wiener
/////////////////////////////////////////////////////////////////////////////

// so called Wiener interpolation. (sharp, similar to Lanczos ?)
// invarint simplified, 6 taps. Weights: (1, -5, 20, 20, -5, 1)/32 - added by Fizick
template<typename pixel_t>
__global__ void kl_vertical_wiener(pixel_t *pDst, const pixel_t *pSrc, int nDstPitch,
  int nSrcPitch, int nWidth, int nHeight, int max_pixel_value)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < nWidth) {
    if (y < 2) {
      pDst[x + y * nDstPitch] = (pSrc[x + y * nSrcPitch] + pSrc[x + (y + 1) * nSrcPitch] + 1) >> 1;
    }
    else if (y < nHeight - 4) {
      pDst[x + y * nDstPitch] = min(max_pixel_value, max(0,
        (pSrc[x + (y - 2) * nSrcPitch]
          + (-(pSrc[x + (y - 1) * nSrcPitch]) + (pSrc[x + y * nSrcPitch] << 2) +
          (pSrc[x + (y + 1) * nSrcPitch] << 2) - (pSrc[x + (y + 2) * nSrcPitch])) * 5
          + (pSrc[x + (y + 3) * nSrcPitch]) + 16) >> 5));
    }
    else if (y < nHeight - 1) {
      pDst[x + y * nDstPitch] = (pSrc[x + y * nSrcPitch] + pSrc[x + (y + 1) * nSrcPitch] + 1) >> 1;
    }
    else if (y < nHeight) {
      // last row
      pDst[x + y * nDstPitch] = pSrc[x + y * nSrcPitch];
    }
  }
}

template<typename pixel_t>
void KDeintKernel::VerticalWiener(
  pixel_t *pDst, const pixel_t *pSrc, int nDstPitch,
  int nSrcPitch, int nWidth, int nHeight, int bits_per_pixel)
{
  const int max_pixel_value = sizeof(pixel_t) == 1 ? 255 : (1 << bits_per_pixel) - 1;

  dim3 threads(32, 16);
  dim3 blocks(nblocks(nWidth, threads.x), nblocks(nHeight, threads.y));
  kl_vertical_wiener<pixel_t> << <blocks, threads, 0, stream >> > (
    pDst, pSrc, nDstPitch, nSrcPitch, nWidth, nHeight, max_pixel_value);
  DebugSync();
}

template<typename pixel_t>
__global__ void kl_horizontal_wiener(pixel_t *pDst, const pixel_t *pSrc, int nDstPitch,
  int nSrcPitch, int nWidth, int nHeight, int max_pixel_value)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (y < nHeight) {
    if (x < 2) {
      pDst[x + y * nDstPitch] = (pSrc[x + y * nSrcPitch] + pSrc[(x + 1) + y * nSrcPitch] + 1) >> 1;
    }
    else if (x < nWidth - 4) {
      pDst[x + y * nDstPitch] = min(max_pixel_value, max(0,
        (pSrc[(x - 2) + y * nSrcPitch]
          + (-(pSrc[(x - 1) + y * nSrcPitch]) + (pSrc[x + y * nSrcPitch] << 2) +
          (pSrc[(x + 1) + y * nSrcPitch] << 2) - (pSrc[(x + 2) + y * nSrcPitch])) * 5
          + (pSrc[(x + 3) + y * nSrcPitch]) + 16) >> 5));
    }
    else if (x < nWidth - 1) {
      pDst[x + y * nDstPitch] = (pSrc[x + y * nSrcPitch] + pSrc[(x + 1) + y * nSrcPitch] + 1) >> 1;
    }
    else if (x < nWidth) {
      // last column
      pDst[x + y * nDstPitch] = pSrc[x + y * nSrcPitch];
    }
  }
}

template<typename pixel_t>
void KDeintKernel::HorizontalWiener(
  pixel_t *pDst, const pixel_t *pSrc, int nDstPitch,
  int nSrcPitch, int nWidth, int nHeight, int bits_per_pixel)
{
  const int max_pixel_value = sizeof(pixel_t) == 1 ? 255 : (1 << bits_per_pixel) - 1;

  dim3 threads(32, 16);
  dim3 blocks(nblocks(nWidth, threads.x), nblocks(nHeight, threads.y));
  kl_horizontal_wiener<pixel_t> << <blocks, threads, 0, stream >> > (
    pDst, pSrc, nDstPitch, nSrcPitch, nWidth, nHeight, max_pixel_value);
  DebugSync();
}


template void KDeintKernel::VerticalWiener<uint8_t>(
  uint8_t *pDst, const uint8_t *pSrc, int nDstPitch,
  int nSrcPitch, int nWidth, int nHeight, int bits_per_pixel);
template void KDeintKernel::VerticalWiener<uint16_t>(
  uint16_t *pDst, const uint16_t *pSrc, int nDstPitch,
  int nSrcPitch, int nWidth, int nHeight, int bits_per_pixel);
template void KDeintKernel::HorizontalWiener<uint8_t>(
  uint8_t *pDst, const uint8_t *pSrc, int nDstPitch,
  int nSrcPitch, int nWidth, int nHeight, int bits_per_pixel);
template void KDeintKernel::HorizontalWiener<uint16_t>(
  uint16_t *pDst, const uint16_t *pSrc, int nDstPitch,
  int nSrcPitch, int nWidth, int nHeight, int bits_per_pixel);


/////////////////////////////////////////////////////////////////////////////
// RB2BilinearFilter
/////////////////////////////////////////////////////////////////////////////

enum {
  RB2B_BILINEAR_W = 32,
  RB2B_BILINEAR_H = 16,
};

// BilinearFiltered with 1/8, 3/8, 3/8, 1/8 filter for smoothing and anti-aliasing - Fizick
// threads=(RB2B_BILINEAR_W,RB2B_BILINEAR_H)
// nblocks=(nblocks(nWidth*2, RB2B_BILINEAR_W - 2),nblocks(nHeight,RB2B_BILINEAR_H))
template<typename pixel_t>
__global__ void kl_RB2B_bilinear_filtered(
  pixel_t *pDst, const pixel_t *pSrc, int nDstPitch, int nSrcPitch, int nWidth, int nHeight)
{
  __shared__ pixel_t tmp[RB2B_BILINEAR_H][RB2B_BILINEAR_W];

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Vertical�����s
  // Horizontal�ŎQ�Ƃ��邽�ߗ��[1�񂸂]���Ɏ��s
  int x = tx - 1 + blockIdx.x * (RB2B_BILINEAR_W - 2);
  int y = ty + blockIdx.y * RB2B_BILINEAR_H;
  int y2 = y * 2;

  if (x >= 0 && x < nWidth * 2) {
    if (y < 1) {
      tmp[ty][tx] = (pSrc[x + y2 * nSrcPitch] + pSrc[x + (y2 + 1) * nSrcPitch] + 1) / 2;
    }
    else if (y < nHeight - 1) {
      tmp[ty][tx] = (pSrc[x + (y2 - 1) * nSrcPitch]
        + pSrc[x + y2 * nSrcPitch] * 3
        + pSrc[x + (y2 + 1) * nSrcPitch] * 3
        + pSrc[x + (y2 + 2) * nSrcPitch] + 4) / 8;
    }
    else if (y < nHeight) {
      tmp[ty][tx] = (pSrc[x + y2 * nSrcPitch] + pSrc[x + (y2 + 1) * nSrcPitch] + 1) / 2;
    }
  }

  __syncthreads();

  // Horizontal�����s
  x = tx + blockIdx.x * ((RB2B_BILINEAR_W - 2) / 2);
  int tx2 = tx * 2;

  if (tx < ((RB2B_BILINEAR_W - 2) / 2) && y < nHeight) {
    // tmp��[0][1]�����_�ł��邱�Ƃɒ���
    if (x < 1) {
      pDst[x + y * nDstPitch] = (tmp[ty][tx2 + 1] + tmp[ty][tx2 + 2] + 1) / 2;
    }
    else if (x < nWidth - 1) {
      pDst[x + y * nDstPitch] = (tmp[ty][tx2]
        + tmp[ty][tx2 + 1] * 3
        + tmp[ty][tx2 + 2] * 3
        + tmp[ty][tx2 + 3] + 4) / 8;
    }
    else if (x < nWidth) {
      pDst[x + y * nDstPitch] = (tmp[ty][tx2 + 1] + tmp[ty][tx2 + 2] + 1) / 2;
    }
  }
}

template<typename pixel_t>
void KDeintKernel::RB2BilinearFiltered(
  pixel_t *pDst, const pixel_t *pSrc, int nDstPitch, int nSrcPitch, int nWidth, int nHeight)
{
  dim3 threads(RB2B_BILINEAR_W, RB2B_BILINEAR_H);
  dim3 blocks(nblocks(nWidth*2, RB2B_BILINEAR_W - 2), nblocks(nHeight, RB2B_BILINEAR_H));
  kl_RB2B_bilinear_filtered<pixel_t> << <blocks, threads, 0, stream >> > (
    pDst, pSrc, nDstPitch, nSrcPitch, nWidth, nHeight);
  DebugSync();
}

template void KDeintKernel::RB2BilinearFiltered<uint8_t>(
  uint8_t *pDst, const uint8_t *pSrc, int nDstPitch, int nSrcPitch, int nWidth, int nHeight);
template void KDeintKernel::RB2BilinearFiltered<uint16_t>(
  uint16_t *pDst, const uint16_t *pSrc, int nDstPitch, int nSrcPitch, int nWidth, int nHeight);



/////////////////////////////////////////////////////////////////////////////
// SearchMV
/////////////////////////////////////////////////////////////////////////////


typedef int sad_t; // ���float�ɂ���

enum {
  SRCH_DIMX = 128
};

struct SearchBlock {
  // [0-3]: nDxMax, nDyMax, nDxMin, nDyMin �iMax��Max-1�ɂ��Ă����j
  // [4-9]: Left predictor, Up predictor, bottom-right predictor(from coarse level)
  // �����ȂƂ���͍��Ȃ��悤�ɂ���i�Œ�ł��ǂꂩ�P�͗L���Ȃ̂Ŗ����Ȃ�Ƃ���͂��̃C���f�b�N�X�Ŗ��߂�j
  // [10-11]: predictor �� x, y
  int data[12];
  // [0-3]: penaltyZero, penaltyGlobal, 1(penaltyPredictor), penaltyNew
  // [4]: lambda
  sad_t dataf[5];
};

#define CLIP_RECT data
#define REF_VECTOR_INDEX (&data[4])
#define PRED_X data[10]
#define PRED_Y data[11]
#define PENALTIES dataf
#define PENALTY_NEW dataf[3]
#define LAMBDA dataf[4]

#define LARGE_COST INT_MAX

struct CostResult {
  sad_t cost;
  short2 xy;
};

__device__ void dev_clip_mv(short2& v, const int* rect)
{
  v.x = (v.x > rect[0]) ? rect[0] : (v.x < rect[2]) ? rect[2] : v.x;
  v.y = (v.y > rect[1]) ? rect[1] : (v.y < rect[3]) ? rect[3] : v.y;
}

__device__ bool dev_check_mv(int x, int y, const int* rect)
{
  return (x <= rect[0]) & (y <= rect[1]) & (x >= rect[2]) & (y >= rect[3]);
}

__device__ int dev_max(int a, int b, int c) {
  int ab = (a > b) ? a : b;
  return (ab > c) ? ab : c;
}

__device__ int dev_min(int a, int b, int c) {
  int ab = (a < b) ? a : b;
  return (ab < c) ? ab : c;
}

__device__ int dev_sq_norm(int ax, int ay, int bx, int by) {
  return (ax - bx) * (ax - bx) + (ay - by) * (ay - by);
}

// pRef �� �u���b�N�I�t�Z�b�g����\�߈ړ������Ă������|�C���^
// vx,vy �� �T�u�s�N�Z�����܂߂��x�N�g��
template <typename pixel_t, int NPEL>
__device__ const pixel_t* dev_get_ref_block(const pixel_t* pRef, int nPitch, int nImgPitch, int vx, int vy)
{
  if (NPEL != 1) {
    int sx = vx % NPEL;
    int sy = vy % NPEL;
    int si = sx + sy * NPEL;
    int x = vx / NPEL;
    int y = vy / NPEL;
    return &pRef[x + y * nPitch + si * nImgPitch];
  }
  else {
    return &pRef[vx + vy * nPitch];
  }
}

template <typename pixel_t, int BLK_SIZE>
__device__ sad_t dev_calc_sad(
  int wi,
  const pixel_t* pSrcY, const pixel_t* pSrcU, const pixel_t* pSrcV,
  const pixel_t* pRefY, const pixel_t* pRefU, const pixel_t* pRefV,
  int nPitchY, int nPitchU, int nPitchV)
{
  int sad = 0;
  if (BLK_SIZE == 16) {
    // �u���b�N�T�C�Y���X���b�h���ƈ�v
    int yx = wi;
    for (int yy = 0; yy < BLK_SIZE; ++yy) { // 16�񃋁[�v
      sad = __sad(pSrcY[yx + yy * BLK_SIZE], pRefY[yx + yy * nPitchY], sad);
    }
    // UV��8x8
    int uvx = wi % 8;
    int uvy = wi / 8;
    for (int t = 0; t < 4; ++t, uvy += 2) { // 4�񃋁[�v
      sad = __sad(pSrcU[uvx + uvy * BLK_SIZE], pRefU[uvx + uvy * nPitchU], sad);
      sad = __sad(pSrcV[uvx + uvy * BLK_SIZE], pRefV[uvx + uvy * nPitchV], sad);
    }
  }
  else if (BLK_SIZE == 32) {
    // 32x32
    int yx = wi;
    for (int yy = 0; yy < BLK_SIZE; ++yy) { // 32�񃋁[�v
      sad = __sad(pSrcY[yx + yy * BLK_SIZE], pRefY[yx + yy * nPitchY], sad);
      sad = __sad(pSrcY[yx + 16 + yy * BLK_SIZE], pRefY[yx + 16 + yy * nPitchY], sad);
    }
    // �u���b�N�T�C�Y���X���b�h���ƈ�v
    int uvx = wi;
    for (int uvy = 0; uvy < BLK_SIZE; ++uvy) { // 16�񃋁[�v
      sad = __sad(pSrcU[uvx + uvy * BLK_SIZE], pRefU[uvx + uvy * nPitchU], sad);
      sad = __sad(pSrcV[uvx + uvy * BLK_SIZE], pRefV[uvx + uvy * nPitchV], sad);
    }
  }
  return dev_reduce_warp<int, 16, AddReducer>(sad, wi);
}

// MAX - (MAX/4) <= (���ʂ̌�) <= MAX �ł��邱��
// �X���b�h���� (���ʂ̌�) - MAX/2
template <int MAX>
__device__ void dev_reduce_result(CostResult* tmp_, int tid)
{
  volatile CostResult* tmp = (volatile CostResult*)tmp_;
  if(MAX >= 16) tmp[tid] = (tmp[tid].cost < tmp[tid + 8].cost) ? tmp[tid] : tmp[tid + 8];
  tmp[tid] = (tmp[tid].cost < tmp[tid + 4].cost) ? tmp[tid] : tmp[tid + 4];
  tmp[tid] = (tmp[tid].cost < tmp[tid + 2].cost) ? tmp[tid] : tmp[tid + 2];
  tmp[tid] = (tmp[tid].cost < tmp[tid + 1].cost) ? tmp[tid] : tmp[tid + 1];
}

// __syncthreads()���Ăяo���Ă���̂őS���ŌĂ�
template <typename pixel_t, int BLK_SIZE, int NPEL>
__device__ void dev_expanding_search_1(
  int tx, int wi, int bx, int cx, int cy,
  const int* data, const sad_t* dataf,
  CostResult& bestResult,
  const pixel_t* pSrcY, const pixel_t* pSrcU, const pixel_t* pSrcV,
  const pixel_t* __restrict__ pRefBY, const pixel_t* __restrict__ pRefBU, const pixel_t* __restrict__ pRefBV,
  int nPitchY, int nPitchU, int nPitchV,
  int nImgPitchY, int nImgPitchU, int nImgPitchV)
{
  int2 area[] = {
    { -1, -1 },
    { 0, -1 },
    { 1, -1 },
    { -1, 0 },
    { 1, 0 },
    { -1, 1 },
    { 0, 1 },
    { 1, 1 }
  };

  __shared__ bool isVectorOK[8];
  __shared__ CostResult result[8];
  __shared__ const pixel_t* pRefY[8];
  __shared__ const pixel_t* pRefU[8];
  __shared__ const pixel_t* pRefV[8];

  if (tx < 8) {
    int x = result[tx].xy.x = cx + area[tx].x;
    int y = result[tx].xy.y = cy + area[tx].y;
    bool ok = dev_check_mv(x, y, CLIP_RECT);
    int cost = (LAMBDA * dev_sq_norm(x, y, PRED_X, PRED_Y)) >> 8;

    // no additional SAD calculations if partial sum is already above minCost
    if (cost >= bestResult.cost) {
      ok = false;
    }

    isVectorOK[tx] = ok;
    result[tx].cost = ok ? cost : LARGE_COST;

    pRefY[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBY, nPitchY, nImgPitchY, x, y);
    pRefU[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBU, nPitchU, nImgPitchU, x / 2, y / 2);
    pRefV[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBV, nPitchV, nImgPitchV, x / 2, y / 2);
  }

  __syncthreads();

  if (isVectorOK[bx]) {
    sad_t sad = dev_calc_sad<pixel_t, BLK_SIZE>(wi, pSrcY, pSrcU, pSrcV, pRefY[bx], pRefU[bx], pRefV[bx], nPitchY, nPitchU, nPitchV);
    if (wi == 0) {
      result[bx].cost += (sad * PENALTY_NEW) >> 8;
    }
  }

  __syncthreads();

  // ���ʏW��
  if (tx < 4) { // reduce��8-4=4�X���b�h�ŌĂ�
    dev_reduce_result<8>(result, tx);

    if (tx == 0) { // tx == 0�͍Ō�̃f�[�^����������ł���̂ŃA�N�Z�XOK
      if (result[0].cost < bestResult.cost) {
        bestResult = result[0];
      }
    }
  }
}

// __syncthreads()���Ăяo���Ă���̂őS���ŌĂ�
template <typename pixel_t, int BLK_SIZE, int NPEL>
__device__ void dev_expanding_search_2(
  int tx, int wi, int bx, int cx, int cy,
  const int* data, const sad_t* dataf,
  CostResult& bestResult,
  const pixel_t* pSrcY, const pixel_t* pSrcU, const pixel_t* pSrcV,
  const pixel_t* __restrict__ pRefBY, const pixel_t* __restrict__ pRefBU, const pixel_t* __restrict__ pRefBV,
  int nPitchY, int nPitchU, int nPitchV,
  int nImgPitchY, int nImgPitchU, int nImgPitchV)
{
  int2 area[] = {
    { -2, -2 },
    { -1, -2 },
    { 0, -2 },
    { 1, -2 },
    { 2, -2 },

    { -2, -1 },
    { 2, -1 },
    { -2, 0 },
    { 2, 0 },
    { -2, 1 },
    { 2, 1 },

    { -2, 2 },
    { -1, 2 },
    { 0, 2 },
    { 1, 2 },
    { 2, 2 }
  };

  __shared__ bool isVectorOK[16];
  __shared__ CostResult result[16];
  __shared__ const pixel_t* pRefY[16];
  __shared__ const pixel_t* pRefU[16];
  __shared__ const pixel_t* pRefV[16];

  if (tx < 16) {
    int x = result[tx].xy.x = cx + area[tx].x;
    int y = result[tx].xy.y = cy + area[tx].y;
    bool ok = dev_check_mv(x, y, CLIP_RECT);
    int cost = (LAMBDA * dev_sq_norm(x, y, PRED_X, PRED_Y)) >> 8;

    // no additional SAD calculations if partial sum is already above minCost
    if (cost >= bestResult.cost) {
      ok = false;
    }

    isVectorOK[tx] = ok;
    result[tx].cost = ok ? cost : LARGE_COST;

    pRefY[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBY, nPitchY, nImgPitchY, x, y);
    pRefU[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBU, nPitchU, nImgPitchU, x / 2, y / 2);
    pRefV[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBV, nPitchV, nImgPitchV, x / 2, y / 2);
  }

  __syncthreads();

  if (isVectorOK[bx]) {
    sad_t sad = dev_calc_sad<pixel_t, BLK_SIZE>(wi, pSrcY, pSrcU, pSrcV, pRefY[bx], pRefU[bx], pRefV[bx], nPitchY, nPitchU, nPitchV);
    if (wi == 0) {
      result[bx].cost += (sad * PENALTY_NEW) >> 8;
    }
  }
  int bx2 = bx + 8;
  if (isVectorOK[bx2]) {
    sad_t sad = dev_calc_sad<pixel_t, BLK_SIZE>(wi, pSrcY, pSrcU, pSrcV, pRefY[bx2], pRefU[bx2], pRefV[bx2], nPitchY, nPitchU, nPitchV);
    if (wi == 0) {
      result[bx2].cost += (sad * PENALTY_NEW) >> 8;
    }
  }

  __syncthreads();

  // ���ʏW��
  if (tx < 8) { // reduce��16-8=8�X���b�h�ŌĂ�
    dev_reduce_result<16>(result, tx);

    if (tx == 0) { // tx == 0�͍Ō�̃f�[�^����������ł���̂ŃA�N�Z�XOK
      if (result[0].cost < bestResult.cost) {
        bestResult = result[0];
      }
    }
  }
}

// __syncthreads()���Ăяo���Ă���̂őS���ŌĂ�
template <typename pixel_t, int BLK_SIZE, int NPEL>
__device__ void dev_hex2_search_1(
  int tx, int wi, int bx, int cx, int cy,
  const int* data, const sad_t* dataf,
  CostResult& bestResult,
  const pixel_t* pSrcY, const pixel_t* pSrcU, const pixel_t* pSrcV,
  const pixel_t* __restrict__ pRefBY, const pixel_t* __restrict__ pRefBU, const pixel_t* __restrict__ pRefBV,
  int nPitchY, int nPitchU, int nPitchV,
  int nImgPitchY, int nImgPitchU, int nImgPitchV)
{
  int2 area[] = { { -1,-2 },{ -2,0 },{ -1,2 },{ 1,2 },{ 2,0 },{ 1,-2 },{ -1,-2 },{ -2,0 } };

  __shared__ bool isVectorOK[8];
  __shared__ CostResult result[8];
  __shared__ const pixel_t* pRefY[8];
  __shared__ const pixel_t* pRefU[8];
  __shared__ const pixel_t* pRefV[8];

  isVectorOK[tx] = false;

  if (tx < 6) {
    int x = result[tx].xy.x = cx + area[tx].x;
    int y = result[tx].xy.y = cy + area[tx].y;
    bool ok = dev_check_mv(x, y, CLIP_RECT);
    int cost = (LAMBDA * dev_sq_norm(x, y, PRED_X, PRED_Y)) >> 8;

    // no additional SAD calculations if partial sum is already above minCost
    if (cost >= bestResult.cost) {
      ok = false;
    }

    isVectorOK[tx] = ok;
    result[tx].cost = ok ? cost : LARGE_COST;

    pRefY[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBY, nPitchY, nImgPitchY, x, y);
    pRefU[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBU, nPitchU, nImgPitchU, x / 2, y / 2);
    pRefV[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBV, nPitchV, nImgPitchV, x / 2, y / 2);
  }

  __syncthreads();

  if (isVectorOK[bx]) {
    sad_t sad = dev_calc_sad<pixel_t, BLK_SIZE>(wi, pSrcY, pSrcU, pSrcV, pRefY[bx], pRefU[bx], pRefV[bx], nPitchY, nPitchU, nPitchV);
    if (wi == 0) {
      result[bx].cost += (sad * PENALTY_NEW) >> 8;
    }
  }

  __syncthreads();

  // ���ʏW��
  if (tx < 2) { // reduce��6-4=2�X���b�h�ŌĂ�
    dev_reduce_result<8>(result, tx);

    if (tx == 0) { // tx == 0�͍Ō�̃f�[�^����������ł���̂ŃA�N�Z�XOK
      if (result[0].cost < bestResult.cost) {
        bestResult = result[0];
      }
    }
  }
}

// SRCH_DIMX % BLK_SIZE == 0������
template <typename pixel_t, int BLK_SIZE>
__device__ void dev_read_pixels(int tx, const pixel_t* src, int nPitch, int offx, int offy, pixel_t *dst)
{
  int y = tx / BLK_SIZE;
  int x = tx % BLK_SIZE;
  if (BLK_SIZE == 8) {
    if (y < 8) {
      dst[x + y * BLK_SIZE] = src[(x + offx) + (y + offy) * nPitch];
    }
  }
  else if (BLK_SIZE == 16) {
    dst[x + y * BLK_SIZE] = src[(x + offx) + (y + offy) * nPitch];
    y += 8;
    dst[x + y * BLK_SIZE] = src[(x + offx) + (y + offy) * nPitch];
  }
  else if (BLK_SIZE == 32) {
    for (; y < BLK_SIZE; y += SRCH_DIMX / BLK_SIZE) {
      dst[x + y * BLK_SIZE] = src[(x + offx) + (y + offy) * nPitch];
    }
  }
}

template <typename pixel_t, int BLK_SIZE, int SEARCH, int NPEL>
__global__ void kl_search(
  int nBlkX, int nBlkY, const SearchBlock* __restrict__ blocks,
  short2* vectors, // [x,y]
  int nPad,
  const pixel_t* __restrict__ pSrcY, const pixel_t* __restrict__ pSrcU, const pixel_t* __restrict__ pSrcV,
  const pixel_t* __restrict__ pRefY, const pixel_t* __restrict__ pRefU, const pixel_t* __restrict__ pRefV,
  int nPitchY, int nPitchUV,
  int nImgPitchY, int nImgPitchUV
)
{
  enum {
    BLK_SIZE_UV = BLK_SIZE / 2,
    BLK_STEP = BLK_SIZE / 2,
  };

  const int tx = threadIdx.x;
  const int wi = tx % 16;
  const int bx = tx / 16;

  for (int blkx = blockIdx.x; blkx < nBlkX; blkx += blockDim.x) {
    for (int blky = 0; blky < nBlkY; ++blky) {

      // src��shared memory�ɓ]��
      int offx = nPad + blkx * BLK_STEP;
      int offy = nPad + blky * BLK_STEP;

      __shared__ pixel_t srcY[BLK_SIZE * BLK_SIZE];
      __shared__ pixel_t srcU[BLK_SIZE_UV * BLK_SIZE_UV];
      __shared__ pixel_t srcV[BLK_SIZE_UV * BLK_SIZE_UV];

      dev_read_pixels<pixel_t, BLK_SIZE>(tx, pSrcY, nPitchY, offx, offy, srcY);
      dev_read_pixels<pixel_t, BLK_SIZE_UV>(tx, pSrcU, nPitchUV, offx / 2, offy / 2, srcU);
      dev_read_pixels<pixel_t, BLK_SIZE_UV>(tx, pSrcV, nPitchUV, offx / 2, offy / 2, srcV);

      __shared__ const pixel_t* pRefBY;
      __shared__ const pixel_t* pRefBU;
      __shared__ const pixel_t* pRefBV;

      if (tx == 0) {
        pRefBY = &pRefY[offx + offy * nPitchY];
        pRefBU = &pRefU[offx / 2 + offy / 2 * nPitchU];
        pRefBV = &pRefV[offx / 2 + offy / 2 * nPitchV];
      }

      // �p�����[�^�Ȃǂ̃f�[�^��shared memory�Ɋi�[
      __shared__ int data[12];
      __shared__ sad_t dataf[5];

      if (tx < 12) {
        int blkIdx = blky*nBlkX + blkx;
        data[tx] = blocks[blkIdx].data[tx];
        if (tx < 5) {
          dataf[tx] = blocks[blkIdx].dataf[tx];
        }
      }

      __syncthreads();

      // FetchPredictors
      __shared__ CostResult result[8];
      __shared__ const pixel_t* pRefY[8];
      __shared__ const pixel_t* pRefU[8];
      __shared__ const pixel_t* pRefV[8];

      if (tx < 6) {
        __shared__ volatile short pred[7][2]; // x, y

        // zero, global, predictor, predictors[1]�`[3]���擾
        short2 vec = vectors[REF_VECTOR_INDEX[tx]];
        dev_clip_mv(vec, CLIP_RECT);
        pred[tx][0] = vec.x;
        pred[tx][1] = vec.y;
        // memfence
        if (tx < 2) {
          // Median predictor
          // �v�Z�����������̂ŏ��������E�E�E
          int a = pred[3][tx];
          int b = pred[4][tx];
          int c = pred[5][tx];
          int max_ = dev_max(a, b, c);
          int min_ = dev_min(a, b, c);
          int med_ = a + b + c - max_ - min_;
          pred[6][tx] = med_;
        }
        // memfence
        int x = result[tx].xy.x = pred[tx][0];
        int y = result[tx].xy.y = pred[tx][1];
        result[tx].cost = (LAMBDA * dev_sq_norm(x, y, PRED_X, PRED_Y)) >> 8;

        pRefY[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBY, nPitchY, nImgPitchY, x, y);
        pRefU[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBU, nPitchUV, nImgPitchUV, x / 2, y / 2);
        pRefV[tx] = dev_get_ref_block<pixel_t, NPEL>(pRefBV, nPitchUV, nImgPitchUV, x / 2, y / 2);
      }

      __syncthreads();

      // �܂���7�ӏ����v�Z
      if (bx < 7) {
        sad_t sad = dev_calc_sad<pixel_t, BLK_SIZE>(wi, srcY, srcU, srcV, pRefY[bx], pRefU[bx], pRefV[bx], nPitchY, nPitchUV, nPitchUV);
        if (wi == 0) {
          if (bx < 3) {
            // pzero, pglobal, 1
            result[bx].cost = (sad * PENALTIES[bx]) >> 8;
          }
          else {
            result[bx].cost += sad;
          }
        }
        // �Ƃ肠������r��cost�����ł��̂�SAD�͗v��Ȃ�
        // SAD�͒T�����I�������Čv�Z����
      }

      __syncthreads();

      // ���ʏW��
      if (tx < 3) { // 7-4=3�X���b�h�ŌĂ�
        dev_reduce_result<8>(result, tx);
      }

      __syncthreads();

      // Refine
      if (SEARCH == 1) {
        // EXHAUSTIVE
        int bmx = result[0].xy.x;
        int bmy = result[0].xy.y;
        dev_expanding_search_1<pixel_t, BLK_SIZE, NPEL>(
          tx, wi, bx, bmx, bmy, data, dataf, result[0],
          srcY, srcU, srcV, pRefBY, pRefBU, pRefBV,
          nPitchY, nPitchUV, nPitchUV, nImgPitchY, nImgPitchUV, nImgPitchUV);
        dev_expanding_search_2<pixel_t, BLK_SIZE, NPEL>(
          tx, wi, bx, bmx, bmy, data, dataf, result[0],
          srcY, srcU, srcV, pRefBY, pRefBU, pRefBV,
          nPitchY, nPitchUV, nPitchUV, nImgPitchY, nImgPitchUV, nImgPitchUV);
      }
      else if (SEARCH == 2) {
        // HEX2SEARCH
        dev_hex2_search_1<pixel_t, BLK_SIZE, NPEL>(
          tx, wi, bx, result[0].xy.x, result[0].xy.y, data, dataf, result[0],
          srcY, srcU, srcV, pRefBY, pRefBU, pRefBV,
          nPitchY, nPitchUV, nPitchUV, nImgPitchY, nImgPitchUV, nImgPitchUV);
        dev_expanding_search_1<pixel_t, BLK_SIZE, NPEL>(
          tx, wi, bx, result[0].xy.x, result[0].xy.y, data, dataf, result[0],
          srcY, srcU, srcV, pRefBY, pRefBU, pRefBV,
          nPitchY, nPitchUV, nPitchUV, nImgPitchY, nImgPitchUV, nImgPitchUV);
      }

      // ���ʏ�������
      if (tx == 0) {
        vectors[blky*nBlkX + blkx] = result[0].xy;
      }

      // ���L�������ی�
      __syncthreads();
    }
  }
}

// threads=128,
template <typename pixel_t, int BLK_SIZE, int NPEL>
__global__ void kl_calc_all_sad(
  int nBlkX, int nBlkY,
  const short2* vectors, // [x,y]
  int* dst_sad,
  int nPad,
  const pixel_t* __restrict__ pSrcY, const pixel_t* __restrict__ pSrcU, const pixel_t* __restrict__ pSrcV,
  const pixel_t* __restrict__ pRefY, const pixel_t* __restrict__ pRefU, const pixel_t* __restrict__ pRefV,
  int nPitchY, int nPitchUV,
  int nImgPitchY, int nImgPitchUV)
{
  enum {
    BLK_SIZE_UV = BLK_SIZE / 2,
    BLK_STEP = BLK_SIZE / 2,
  };

  int tid = threadIdx.x;
  int x = blockIdx.x * blockDim.x;
  int y = blockIdx.y * blockDim.y;

  int offx = nPad + blkx * BLK_STEP;
  int offy = nPad + blky * BLK_STEP;

  __shared__ const pixel_t* pRefBY;
  __shared__ const pixel_t* pRefBU;
  __shared__ const pixel_t* pRefBV;
  __shared__ const pixel_t* pSrcBY;
  __shared__ const pixel_t* pSrcBU;
  __shared__ const pixel_t* pSrcBV;

  if (tid == 0) {
    pRefBY = &pRefY[offx + offy * nPitchY];
    pRefBU = &pRefU[offx / 2 + offy / 2 * nPitchU];
    pRefBV = &pRefV[offx / 2 + offy / 2 * nPitchV];
    pSrcBY = &pSrcY[offx + offy * nPitchY];
    pSrcBU = &pSrcU[offx / 2 + offy / 2 * nPitchU];
    pSrcBV = &pSrcV[offx / 2 + offy / 2 * nPitchV];

    short2 xy = vectors[x + y * nBlkX];

    pRefBY = dev_get_ref_block<pixel_t, NPEL>(pRefBY, nPitchY, nImgPitchY, xy.x, xy.y);
    pRefBU = dev_get_ref_block<pixel_t, NPEL>(pRefBU, nPitchUV, nImgPitchUV, xy.x / 2, xy.y / 2);
    pRefBV = dev_get_ref_block<pixel_t, NPEL>(pRefBV, nPitchUV, nImgPitchUV, xy.x / 2, xy.y / 2);
  }

  __syncthreads();

  int sad = 0;
  if (BLK_SIZE == 16) {
    // 16x16
    int yx = tid % 16;
    int yy = tid / 16;
    for (int t = 0; t < 2; ++t, yy += 8) { // 2�񃋁[�v
      sad = __sad(pSrcBY[yx + yy * nPitchY], pRefBY[yx + yy * nPitchY], sad);
    }
    // UV��8x8
    int uvx = tid % 8;
    int uvy = tid / 8;
    if (uvy > 8) {
      uvy -= 8;
      sad = __sad(pSrcBU[uvx + uvy * nPitchUV], pRefBU[uvx + uvy * nPitchUV], sad);
    }
    else {
      sad = __sad(pSrcBV[uvx + uvy * nPitchUV], pRefBV[uvx + uvy * nPitchUV], sad);
    }
  }
  else if (BLK_SIZE == 32) {
    // 32x32
    int yx = tid % 32;
    int yy = tid / 32;
    for (int t = 0; t < 8; ++t, yy += 4) { // 8�񃋁[�v
      sad = __sad(pSrcBY[yx + yy * nPitchY], pRefBY[yx + yy * nPitchY], sad);
    }
    // �u���b�N�T�C�Y���X���b�h���ƈ�v
    int uvx = tid % 16;
    int uvy = tid / 16;
    for (int t = 0; t < 2; ++t, uvy += 8) { // 2�񃋁[�v
      sad = __sad(pSrcBU[uvx + uvy * nPitchUV], pRefBU[uvx + uvy * nPitchUV], sad);
      sad = __sad(pSrcBV[uvx + uvy * nPitchUV], pRefBV[uvx + uvy * nPitchUV], sad);
    }
  }
  sad = dev_reduce<128>(sad, tid);
  
  if (tx == 0) {
    dst_sad[x + y * nBlkX] = sad;
  }
}

__global__ void kl_prepare_search(
  int nBlkX, int nBlkY, int nBlkSize, int nLogScale,
  sad_t nCurrentLambda, sad_t lsad,
  sad_t penaltyZero, sad_t penaltyGlobal, sad_t penaltyNew,
  int nPel, int nPad, int nBlkSizeOvr, int nExtendedWidth, int nExptendedHeight,
  const int* sads, SearchBlock* dst_blocks)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < nBlkX && y < nBlkY) {
    //
    int blkIdx = x + y*nBlkX;
    int sad = sads[blkIdx];
    SearchBlock *data = &dst_blocks[blkIdx];

    int x = nPad + nBlkSizeOvr * x;
    int y = nPad + nBlkSizeOvr * y;
    //
    int nPaddingScaled = nPad >> nLogScale;

    int nDxMax = nPel * (nExtendedWidth - x - nBlkSize - nPad + nPaddingScaled) - 1;
    int nDyMax = nPel * (nExptendedHeight - y - nBlkSize - nPad + nPaddingScaled) - 1;
    int nDxMin = -nPel * (x - nPad + nPaddingScaled);
    int nDyMin = -nPel * (y - nPad + nPaddingScaled);

    data->data[0] = nDxMax;
    data->data[1] = nDyMax;
    data->data[2] = nDxMin;
    data->data[3] = nDyMin;

    int p1 = -2; // -2��zero�x�N�^
    // Left (or right) predictor
    if (x >= 2)
    {
      p1 = blkIdx - (1 + (x & 1));
    }

    int p2 = -2;
    // Up predictor
    if (y > 0)
    {
      p2 = blkIdx - nBlkX;
    }
    else {
      // median��left��I�΂��邽��
      p2 = p1;
    }

    int p3 = -2;
    // bottom-right pridictor (from coarse level)
    if ((y < nBlkY - 1) && (x < nBlkX - 1))
    {
      p3 = blkIdx + nBlkX + 1;
    }

    data->data[4] = -2;    // zero
    data->data[5] = -1;    // global
    data->data[6] = blkIdx;// predictor
    data->data[7] = p1;    //  predictors[1]
    data->data[8] = p2;    //  predictors[2]
    data->data[9] = p3;    //  predictors[3]

    data->dataf[0] = penaltyZero;
    data->dataf[1] = penaltyGlobal;
    data->dataf[2] = 1;
    data->dataf[3] = penaltyNew;

    sad_t lambda = nCurrentLambda * lsad / (lsad + (sad >> 1)) * lsad / (lsad + (sad >> 1));
    data->dataf[4] = lambda;
  }
}

// threads=(1024), blocks=(2)
__global__ void kl_most_freq_mv(short2* vectors, int nVec, short2* globalMVec)
{
  enum {
    DIMX = 1024,
    // level==1���ő�Ȃ̂ŁA���̃T�C�Y��8K���炢�܂őΉ�
    FREQ_SIZE = DIMX*8,
    HALF_SIZE = FREQ_SIZE / 2
  };

  int tid = threadIdx.x;

  union SharedBuffer {
    int freq_arr[FREQ_SIZE]; //32KB
    struct {
      int red_cnt[DIMX];
      int red_idx[DIMX];
    };
  };
  __shared__ SharedBuffer b;

  for (int i = 0; i < FREQ_SIZE/ DIMX; i += DIMX) {
    b.freq_arr[tid + i * DIMX] = 0;
  }
  __syncthreads();

  if (blockIdx.x == 0) {
    // x
    for (int i = tid; i < nVec; i += DIMX) {
      atomicAdd(&b.freq_arr[vectors[i].x + HALF_SIZE], 1);
    }
  }
  else {
    // y
    for (int i = tid; i < nVec; i += DIMX) {
      atomicAdd(&b.freq_arr[vectors[i].y + HALF_SIZE], 1);
    }
  }
  __syncthreads();

  int maxcnt = 0;
  int index = 0;
  for (int i = 0; i < FREQ_SIZE / DIMX; i += DIMX) {
    if (b.freq_arr[tid + i * DIMX] > maxcnt) {
      maxcnt = b.freq_arr[tid + i * DIMX];
      index = tid + i * DIMX;
    }
  }
  __syncthreads();

  dev_reduce2<int, int, DIMX, CountIndexReducer>(tid, maxcnt, index, b.red_cnt, b.red_idx);

  if (tid == 0) {
    if (blockIdx.x == 0) {
      // x
      globalMVec->x = index - HALF_SIZE;
    }
    else {
      // y
      globalMVec->y = index - HALF_SIZE;
    }
  }
}

__global__ void kl_mean_global_mv(short2* vectors, int nVec, short2* globalMVec)
{
  enum {
    DIMX = 1024,
  };

  int tid = threadIdx.x;
  int medianx = globalMVec->x;
  int mediany = globalMVec->y;

  int meanvx = 0;
  int meanvy = 0;
  int num = 0;

  for (int i = tid; i < nVec; i += DIMX) {
    if (__sad(vectors[i].x, medianx, 0) < 6
      && __sad(vectors[i].y, mediany, 0) < 6)
    {
      meanvx += vectors[i].x;
      meanvy += vectors[i].y;
      num += 1;
    }
  }

  __shared__ int red_vx[DIMX];
  __shared__ int red_vy[DIMX];
  __shared__ int red_num[DIMX];

  red_vx[tid] = meanvx;
  red_vy[tid] = meanvy;
  red_num[tid] = num;

  __syncthreads();
  if (tid < 512) {
    red_vx[tid] += red_vx[tid + 512];
    red_vy[tid] += red_vy[tid + 512];
    red_num[tid] += red_num[tid + 512];
  }
  __syncthreads();
  if (tid < 256) {
    red_vx[tid] += red_vx[tid + 256];
    red_vy[tid] += red_vy[tid + 256];
    red_num[tid] += red_num[tid + 256];
  }
  __syncthreads();
  if (tid < 128) {
    red_vx[tid] += red_vx[tid + 128];
    red_vy[tid] += red_vy[tid + 128];
    red_num[tid] += red_num[tid + 128];
  }
  __syncthreads();
  if (tid < 64) {
    red_vx[tid] += red_vx[tid + 64];
    red_vy[tid] += red_vy[tid + 64];
    red_num[tid] += red_num[tid + 64];
  }
  __syncthreads();
  if (tid < 32) {
    red_vx[tid] += red_vx[tid + 32];
    red_vy[tid] += red_vy[tid + 32];
    red_num[tid] += red_num[tid + 32];
  }
  __syncthreads();
  meanvx = red_vx[tid];
  meanvy = red_vy[tid];
  num = red_num[tid];
  if (tid < 32) {
    meanvx += __shfl_down(meanvx, 16);
    meanvy += __shfl_down(meanvy, 16);
    num += __shfl_down(num, 16);
    meanvx += __shfl_down(meanvx, 8);
    meanvy += __shfl_down(meanvy, 8);
    num += __shfl_down(num, 8);
    meanvx += __shfl_down(meanvx, 4);
    meanvy += __shfl_down(meanvy, 4);
    num += __shfl_down(num, 4);
    meanvx += __shfl_down(meanvx, 2);
    meanvy += __shfl_down(meanvy, 2);
    num += __shfl_down(num, 2);
    meanvx += __shfl_down(meanvx, 1);
    meanvy += __shfl_down(meanvy, 1);
    num += __shfl_down(num, 1);

    if (tid == 0) {
      globalMVec->x = 2 * meanvx / num;
      globalMVec->y = 2 * meanvy / num;
    }
  }
}

// normFactor = 3 - nLogPel + pob.nLogPel
// normov = (nBlkSizeX - nOverlapX)*(nBlkSizeY - nOverlapY)
// aoddx = (nBlkSizeX * 3 - nOverlapX * 2)
// aevenx = (nBlkSizeX * 3 - nOverlapX * 4);
// aoddy = (nBlkSizeY * 3 - nOverlapY * 2);
// aeveny = (nBlkSizeY * 3 - nOverlapY * 4);
// atotalx = (nBlkSizeX - nOverlapX) * 4
// atotaly = (nBlkSizeY - nOverlapY) * 4
__global__ void kl_interpolate_prediction(
  const short2* src_vector, const int* src_sad,
  short2* dst_vector, int* dst_sad,
  int nSrcBlkX, int nSrcBlkY, int nDstBlkX, int nDstBlkY,
  int normFactor, int normov, int atotal, int aodd, int aeven)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < nDstBlkX && y < nDstBlkY) {
    short2 v1, v2, v3, v4;
    int sad1, sad2, sad3, sad4;
    int i = x;
    int j = y;
    if (i >= 2 * nSrcBlkX)
    {
      i = 2 * nSrcBlkX - 1;
    }
    if (j >= 2 * nSrcBlkY)
    {
      j = 2 * nSrcBlkY - 1;
    }
    int offy = -1 + 2 * (j % 2);
    int offx = -1 + 2 * (i % 2);
    int iper2 = i / 2;
    int jper2 = j / 2;

    if ((i == 0) || (i >= 2 * nSrcBlkX - 1))
    {
      if ((j == 0) || (j >= 2 * nSrcBlkY - 1))
      {
        v1 = v2 = v3 = v4 = src_vector[iper2 + (jper2)* nSrcBlkX];
        sad1 = sad2 = sad3 = sad4 = src_sad[iper2 + (jper2)* nSrcBlkX];
      }
      else
      {
        v1 = v2 = src_vector[iper2 + (jper2)* nSrcBlkX];
        sad1 = sad2 = src_sad[iper2 + (jper2)* nSrcBlkX];
        v3 = v4 = src_vector[iper2 + (jper2 + offy) * nSrcBlkX];
        sad3 = sad4 = src_sad[iper2 + (jper2 + offy) * nSrcBlkX];
      }
    }
    else if ((j == 0) || (j >= 2 * nSrcBlkY - 1))
    {
      v1 = v2 = src_vector[iper2 + (jper2)* nSrcBlkX];
      sad1 = sad2 = src_sad[iper2 + (jper2)* nSrcBlkX];
      v3 = v4 = src_vector[iper2 + offx + (jper2)* nSrcBlkX];
      sad3 = sad4 = src_sad[iper2 + offx + (jper2)* nSrcBlkX];
    }
    else
    {
      v1 = src_vector[iper2 + (jper2)* nSrcBlkX];
      sad1 = src_sad[iper2 + (jper2)* nSrcBlkX];
      v2 = src_vector[iper2 + offx + (jper2)* nSrcBlkX];
      sad2 = src_sad[iper2 + offx + (jper2)* nSrcBlkX];
      v3 = src_vector[iper2 + (jper2 + offy) * nSrcBlkX];
      sad3 = src_sad[iper2 + (jper2 + offy) * nSrcBlkX];
      v4 = src_vector[iper2 + offx + (jper2 + offy) * nSrcBlkX];
      sad4 = src_sad[iper2 + offx + (jper2 + offy) * nSrcBlkX];
    }

    int	ax1 = (offx > 0) ? aodd : aeven;
    int ax2 = atotal - ax1;
    int ay1 = (offy > 0) ? aodd : aeven;
    int ay2 = atotal - ay1;
    int a11 = ax1*ay1, a12 = ax1*ay2, a21 = ax2*ay1, a22 = ax2*ay2;
    int vx = (a11*v1.x + a21*v2.x + a12*v3.x + a22*v4.x) / normov;
    int vy = (a11*v1.y + a21*v2.y + a12*v3.y + a22*v4.y) / normov;
    
    sad_t tmp_sad = ((sad_t)a11*sad1 + (sad_t)a21*sad2 + (sad_t)a12*sad3 + (sad_t)a22*sad4) / normov;

    if (normFactor > 0) {
      vx >>= normFactor;
      vy >>= normFactor;
    }
    else {
      vx <<= -normFactor;
      vy <<= -normFactor;
    }

    int index = x + y * nDstBlkX;
    short2 v = { vx,vy };
    dst_vector[index] = v;
    dst_sad[index] = (int)(tmp_sad >> 4);
  }
}

__global__ void kl_load_mv(
	const VECTOR* in,
	short2* vectors, // [x,y]
	int* sads,
	int nBlk)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;

	if (x < nBlk) {
		VECTOR vin = in[x];
		short2 v = { vin.x, vin, y };
		vectors[x] = v;
		sads[x] = vin.sad;
	}
}

__global__ void kl_store_mv(
	VECTOR* dst,
	short2* vectors, // [x,y]
	int* sads,
	int nBlk)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;

	if (x < nBlk) {
		short2 v = vectors[x];
		VECTOR vout = { v.x, v.y, sads[x] };
		vectors[x] = vout;
	}
}

__global__ void kl_write_default_mv(VECTOR* dst, int nBlkCount, int verybigSAD)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;

  if (x < nBlkCount) {
		dst[x].x = 0;
		dst[x].y = 0;
		dst[x].x = verybigSAD;
  }
}


